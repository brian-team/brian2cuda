#include<stdlib.h>
#include "brianlib/cuda_utils.h"
#include "objects.h"
#include<ctime>

#include "code_objects/neurongroup_1_stateupdater_codeobject.h"
#include "code_objects/neurongroup_1_thresholder_codeobject.h"
#include "code_objects/neurongroup_stateupdater_codeobject.h"
#include "code_objects/neurongroup_thresholder_codeobject.h"
#include "code_objects/spikegeneratorgroup_codeobject.h"
#include "code_objects/spikemonitor_1_codeobject.h"
#include "code_objects/spikemonitor_2_codeobject.h"
#include "code_objects/spikemonitor_codeobject.h"
#include "code_objects/synapses_1_group_variable_set_conditional_codeobject.h"
#include "code_objects/synapses_1_group_variable_set_conditional_codeobject_1.h"
#include "code_objects/synapses_1_post_codeobject.h"
#include "code_objects/synapses_1_post_initialise_queue.h"
#include "code_objects/synapses_1_post_push_spikes.h"
#include "code_objects/synapses_1_pre_codeobject.h"
#include "code_objects/synapses_1_pre_initialise_queue.h"
#include "code_objects/synapses_1_pre_push_spikes.h"
#include "code_objects/synapses_1_synapses_create_generator_codeobject.h"
#include "code_objects/synapses_2_pre_codeobject.h"
#include "code_objects/synapses_2_pre_initialise_queue.h"
#include "code_objects/synapses_2_pre_push_spikes.h"
#include "code_objects/synapses_2_synapses_create_generator_codeobject.h"
#include "code_objects/synapses_group_variable_set_conditional_codeobject.h"
#include "code_objects/synapses_pre_codeobject.h"
#include "code_objects/synapses_pre_initialise_queue.h"
#include "code_objects/synapses_pre_push_spikes.h"
#include "code_objects/synapses_synapses_create_generator_codeobject.h"


void brian_start()
{
    _init_arrays();
    _load_arrays();
    srand(time(NULL));

    // Initialize clocks (link timestep and dt to the respective arrays)
    brian::defaultclock.timestep = brian::_array_defaultclock_timestep;
    brian::defaultclock.dt = brian::_array_defaultclock_dt;
    brian::defaultclock.t = brian::_array_defaultclock_t;
}

void brian_end()
{
    _write_arrays();
    _dealloc_arrays();
}


