#include "hip/hip_runtime.h"

#include "objects.h"
#include "rand.h"
#include "synapses_classes.h"
#include "brianlib/clocks.h"
#include "brianlib/cuda_utils.h"
#include "network.h"
#include <hiprand.h>
#include <ctime>
#include <hiprand/hiprand_kernel.h>

// XXX: for some documentation on random number generation, check out our wiki:
//      https://github.com/brian-team/brian2cuda/wiki/Random-number-generation

using namespace brian;

// TODO make this a class member function
// TODO don't call one kernel per codeobject but instead on kernel which takes
//      care of all codeobjects, preferably called with as many threads/blocks
//      as necessary for all states and initializing in parallel with warp
//      level divergence [needs changing set_curand_device_api_states()]
namespace {

    __global__ void init_curand_states(int N, int sequence_offset)
    {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (idx < N)
        {
            // Each thread gets the same seed, a different sequence number and
            // no offset
            // TODO: different seed and 0 sequence number is much faster, with
            // less security for independent sequences, add option as
            // preference!
            //hiprand_init(curand_seed + idx, 0, 0,
            hiprand_init(
                    *d_curand_seed,          // seed
                    sequence_offset + idx,   // sequence number
                    0,                       // offset
                    &d_curand_states[idx]);
        }
    }
}


// need a function pointer for Network::add(), can't pass a pointer to a class
// method, which is of different type
void _run_random_number_buffer()
{
    // random_number_buffer is a RandomNumberBuffer instance, declared in objects.cu
    random_number_buffer.next_time_step();
}


void RandomNumberBuffer::init()
{
    // check that we have enough memory available
    size_t free_byte;
    size_t total_byte;
    CUDA_SAFE_CALL(
            hipMemGetInfo(&free_byte, &total_byte)
            );
    // TODO: This assumes all random number have randomNumber_t type, but poisson
    //       objects have different type
    size_t num_free_floats = free_byte / sizeof(randomNumber_t);

    if (run_counter == 0)
    {
        // number of time steps each codeobject is executed during current Network::run() call
        // XXX: we are assuming here that this function is only run in the first time step of a Network::run()


        // now check if the total number of generated floats fit into available memory
        int total_num_generated_floats = 0;
        if (num_free_floats < total_num_generated_floats)
        {
            // TODO: find a way to deal with this? E.g. looping over buffers sorted
            // by buffer size and reducing them until it fits.
            printf("MEMORY ERROR: Trying to generate more random numbers than fit "
                   "into available memory. Please report this as an issue on "
                   "GitHub: https://github.com/brian-team/brian2cuda/issues/new");
            _dealloc_arrays();
            exit(1);
        }

    } // if (run_counter == 0)

    // init hiprand states only in first run
    if (run_counter == 0)
    {

        // Update hiprand device api states once before anything is run. At this
        // point all N's (also from probabilistically generated synapses) are
        // known. This might update the number of needed hiprand states.
        ensure_enough_curand_states();
    }

}


void RandomNumberBuffer::allocate_device_curand_states()
{
    // allocate globabl memory for hiprand device api states
    CUDA_SAFE_CALL(
            hipMalloc((void**)&dev_curand_states,
                sizeof(hiprandState) * num_curand_states)
            );
    CUDA_SAFE_CALL(
            hipMemcpyToSymbol(HIP_SYMBOL(d_curand_states),
                &dev_curand_states, sizeof(hiprandState*))
            );
}



void RandomNumberBuffer::update_needed_number_curand_states()
{
    // Find the maximum number of threads generating random numbers in parallel
    // using the cuRAND device API. For synapses objects, the number of
    // synapses might not be known yet. This is the case when the first random
    // seed is set and for any seed() call before the synapses creation.
    num_threads_curand_init = max_threads_per_block;
    num_blocks_curand_init = num_curand_states / max_threads_per_block + 1;
    if (num_curand_states < num_threads_curand_init)
        num_threads_curand_init = num_curand_states;
}


void RandomNumberBuffer::set_curand_device_api_states(bool reset_seed)
{
    int sequence_offset = 0;
    int num_curand_states_old = num_curand_states;
    // Whenever hiprand states are set, check if enough states where
    // initialized. This will generate states the first time the seed is set.
    // But it can be that the seed is set before all network objects' N are
    // available (e.g. synapses not created yet) and before the network is
    // run. In such a case, once the network is run, missing hiprand states are
    // generated here. If the seed was not reset inbetween, the pervious states
    // should not be reinitialized (achieved by the `sequence_offset`
    // parameter). If the seed was reset, then all states should be
    // reinitialized.
    update_needed_number_curand_states();

    // number of hiprand states that need to be initialized
    int num_curand_states_to_init;

    if (reset_seed)
    {
        // initialize all hiprand states
        num_curand_states_to_init = num_curand_states;
        sequence_offset = 0;
    }
    else
    {
        // don't initialize existing hiprand states, only the new ones
        num_curand_states_to_init = num_curand_states - num_curand_states_old;
        sequence_offset = num_curand_states_old;
    }

    if (num_curand_states_old < num_curand_states)
    {
        // copy hiprand states to new array of updated size
        hiprandState* dev_curand_states_old = dev_curand_states;
        // allocate memory for new number of hiprand states
        allocate_device_curand_states();

        if ((!reset_seed) && (num_curand_states_old > 0))
        {
            // copy old states to new memory address on device
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_curand_states, dev_curand_states_old,
                        sizeof(hiprandState) * num_curand_states_old,
                        hipMemcpyDeviceToDevice)
                    );
        }
    }

    if (num_curand_states_to_init > 0)
    {
        init_curand_states<<<num_blocks_curand_init, num_threads_curand_init>>>(
                num_curand_states_to_init,
                sequence_offset);
    }
}


void RandomNumberBuffer::ensure_enough_curand_states()
{
    // Separate public function needed for synapses codeobjects that are run
    // only once before the network
    // The N of synapses will not be known when setting the seed and needs to
    // be updated before using random numbers per synapse. This occurs e.g.
    // when initializing synaptic variables (synapses_group_conditional_....)
    bool reset_seed = false;
    set_curand_device_api_states(reset_seed);
}


void RandomNumberBuffer::run_finished()
{
    needs_init = true;
    run_counter += 1;
}


void RandomNumberBuffer::set_seed(unsigned long long seed)
{
    CUDA_SAFE_CALL(
            hiprandSetPseudoRandomGeneratorSeed(curand_generator, seed)
            );

    // generator offset needs to be reset to its default (=0)
    CUDA_SAFE_CALL(
            hiprandSetGeneratorOffset(curand_generator, 0ULL)
            );

    // set seed for hiprand device api calls
    // don't set the same seed for host api and device api random states, just in case
    unsigned long long curand_seed = seed + 1;
    CUDA_SAFE_CALL(
            hipMemcpy(dev_curand_seed, &curand_seed,
                sizeof(unsigned long long), hipMemcpyHostToDevice)
            );

    bool reset_seed = true;
    set_curand_device_api_states(reset_seed);
    // We set all device api states for codeobjects run outside the network
    // since we don't know when they will be used.
    //set_curand_device_api_states_for_separate_calls();
    // Curand device api states for binomials during network runs will be set
    // only for the current run in init(), once the network starts.
}


void RandomNumberBuffer::refill_uniform_numbers(
        randomNumber_t* dev_rand_allocator,
        randomNumber_t* &dev_rand,
        int num_per_gen_rand,
        int &idx_rand)
{
    // generate uniform distributed random numbers and reset buffer index

    hiprandGenerateUniformDouble(curand_generator, dev_rand_allocator, num_per_gen_rand);
    // before: XXX dev_rand = &dev_rand_allocator[0];
    dev_rand = dev_rand_allocator;
    idx_rand = 1;
}


void RandomNumberBuffer::refill_normal_numbers(
        randomNumber_t* dev_randn_allocator,
        randomNumber_t* &dev_randn,
        int num_per_gen_randn,
        int &idx_randn)
{
    // generate normal distributed random numbers and reset buffer index

    hiprandGenerateNormalDouble(curand_generator, dev_randn_allocator, num_per_gen_randn, 0, 1);
    // before: XXX dev_randn = &dev_randn_allocator[0];
    dev_randn = dev_randn_allocator;
    idx_randn = 1;
}


void RandomNumberBuffer::refill_poisson_numbers(
        double lambda,
        unsigned int* dev_poisson_allocator,
        unsigned int* &dev_poisson,
        int num_per_gen_poisson,
        int &idx_poisson)
{
    // generate poisson distributed random numbers and reset buffer index

    printf("num_per_gen_poisson %d, lambda %f\n", num_per_gen_poisson, lambda);
    CUDA_SAFE_CALL(
            hiprandGeneratePoisson(curand_generator, dev_poisson_allocator, num_per_gen_poisson, lambda)
            );
    dev_poisson = dev_poisson_allocator;
    idx_poisson = 1;
}

void RandomNumberBuffer::next_time_step()
{
    // init buffers at fist time step of each run call
    if (needs_init)
    {
        // free device memory for random numbers used during last run call
        if (run_counter > 0)
        {
        }

        // init random number buffers
        init();
        needs_init = false;
    }

    if (run_counter == 0)
    {
    }// run_counter == 0
}
