#include "hip/hip_runtime.h"
#include "objects.h"
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/unique.h>
#include <iostream>
#include <ctime>
#include <limits.h>
#include <tuple>
#include <string>
#include <iomanip>
#include <vector>
#include "code_objects/synapses_1_pre_initialise_queue.h"
#include "brianlib/cuda_utils.h"

// Makro for file and line information in _cudaSafeCall
#define COPY_HOST_ARRAY_TO_DEVICE_SYMBOL(a, b, c, d) \
    _copyHostArrayToDeviceSymbol(a, b, c, d, __FILE__, __LINE__)

namespace {
    // vector_t<T> is an alias for thrust:host_vector<T>
    template <typename T> using vector_t = thrust::host_vector<T>;
    // tuple type typedef
    typedef std::tuple<std::string, size_t, int> tuple_t;

    std::vector<tuple_t> memory_recorder;

    // Functions for online update of mean and std
    // for a new value newValue, compute the new count, new mean, the new M2.
    // mean accumulates the mean of the entire dataset
    // M2 aggregates the squared distance from the mean
    // count aggregates the number of samples seen so far
    inline void updateMeanStd(int &count, double &mean, double& M2, double newValue){
        count += 1;
        double delta = newValue - mean;
        mean += delta / count;
        double delta2 = newValue - mean;
        M2 += delta * delta2;
    }

    // get std from aggregated M2 value
    double getStd(int count, double M2){
        if (count < 2){
            return NAN;
        }
        double variance = M2 / (count - 1);
        double stdValue = sqrt(variance);
        return stdValue;
    }

    // Copy the data from a host array to global device memory and copy the
    // symbol to a global device variable.
    // host_array: host array with data to copy
    // device_symbol: global __device__ variable of same type as `host_array`
    // num_elements: number of elements in host_array to copy
    // NOTE: T can be a pointer variable itself (when copying 2D arrays)
    template <typename T>
    inline void _copyHostArrayToDeviceSymbol(const T *host_array, T *&device_symbol,
            int num_elements, const char* name, const char* file,
            const int line){
        T *d_ptr_tmp;
        size_t bytes = sizeof(T) * num_elements;
        // allocate device memory
        _cudaSafeCall(
                hipMalloc((void**)&d_ptr_tmp, bytes),
                file, line, "hipMalloc");
        // copy data from host array to device
        _cudaSafeCall(
                hipMemcpy(d_ptr_tmp, host_array, bytes, hipMemcpyHostToDevice),
                file, line, "hipMemcpy");
        // copy the device data pointer to the global device symbol
        _cudaSafeCall(
                hipMemcpyToSymbol(HIP_SYMBOL(device_symbol), &d_ptr_tmp, sizeof(T*)),
                file, line, "hipMemcpyToSymbol");
        memory_recorder.push_back(std::make_tuple(name, bytes, num_elements));
    }
}


__global__ void _run_synapses_1_pre_initialise_queue_kernel(
    int _source_N,
    int _num_blocks,
    int _num_threads,
    double _dt,
    int _syn_N,
    int num_queues,
    bool new_mode)
{
    using namespace brian;

    int tid = threadIdx.x;

    synapses_1_pre.queue->prepare(
        tid,
        _num_threads,
        _num_blocks,
        0,
        _source_N,
        _syn_N,
        num_queues,
        synapses_1_pre_num_synapses_by_pre,
        synapses_1_pre_num_synapses_by_bundle,
        synapses_1_pre_num_unique_delays_by_pre,
        synapses_1_pre_unique_delays,
        synapses_1_pre_global_bundle_id_start_by_pre,
        synapses_1_pre_synapses_offset_by_bundle,
        synapses_1_pre_synapse_ids,
        synapses_1_pre_synapse_ids_by_pre,
        synapses_1_pre_unique_delays_offset_by_pre,
        synapses_1_pre_unique_delay_start_idcs);
    synapses_1_pre.no_or_const_delay_mode = new_mode;
}


void _run_synapses_1_pre_initialise_queue()
{
    using namespace brian;

    std::clock_t start_timer = std::clock();
    const double to_MB = 1.0 / (1024.0 * 1024.0);

    CUDA_CHECK_MEMORY();
    size_t used_device_memory_start = used_device_memory;

    ///// HOST_CONSTANTS ///////////
    const int _numN = 1;
		double* const _array_synapses_1_delay = thrust::raw_pointer_cast(&_dynamic_array_synapses_1_delay[0]);
		const int _numdelay = _dynamic_array_synapses_1_delay.size();

    ///// pointers_lines /////
        
    int32_t*   _ptr_array_synapses_1_N = _array_synapses_1_N;
    double*   _ptr_array_defaultclock_dt = _array_defaultclock_dt;
    double*   _ptr_array_synapses_1_delay = _array_synapses_1_delay;


    int64_t syn_N_check = _ptr_array_synapses_1_N[0];

    if (syn_N_check == 0){
        return;
    }
    else if (syn_N_check > INT_MAX){
        printf("ERROR: There are more Synapses (%lu) than an int can "
               "hold on this system (%u).\n", syn_N_check, INT_MAX);
    }
    // total number of synapses
    int syn_N = (int)syn_N_check;

    // simulation time step
    double dt = _ptr_array_defaultclock_dt[0];
    // number of neurons in source group
    int source_N = 2500;
    // number of neurons in target group
    int target_N = 100;

    // TODO: for multiple SynapticPathways for the same Synapses object (on_pre and on_post) the following copy is identical in both pathways initialise templates

    //////////////////////
    // Scalar variables //
    //////////////////////

    // total number of (preID, postBlock) pairs
    int num_pre_post_blocks = num_parallel_blocks * source_N;
    // size of the connectivity matrix (equal number of synapses)
    int size_connectivity_matrix = 0;

    // statistics of number of synapses per (preID, postBlock) pair
    int sum_num_elements = 0;
    int count_num_elements = 0;
    double mean_num_elements = 0;
    double M2_num_elements = 0;


    ////////////////////////////////////////////////////////
    // Create array and vector variables (in host memory) //
    ////////////////////////////////////////////////////////

    /* VARIABLE NAMING:
     * Not scalar variables are named after TYPE_NAME_STRUCTURE, with:
     * STRUCTURE: the first array dimensions structure (`by_pre`, `by_bundle` or none)
     *   `by_pre`: Array (host pointer type) of size `num_pre_post_blocks`,
     *             which is the number of (preID, postBlock) pairs.
     *   `by_bundle`: thrust::host_vector, size of total number of bundles,
     *                which is one for each delay in each (preID, postBlock) pair.
     *                Different (preID, postBlock) pairs can have different sets
     *                of delay values -> each bundle gets a global bundleID
     *   none: If no STRUCTURE given, it's a one dim array storing everything
     * TYPE: data type in STRUCTURE (`h`, `h_vec`, `h_ptr`, `d_ptr`), with
     *       `h`: host value, `h_vec`: host vector, `h_ptr`: host pointer,
     *       `d_ptr`: device pointer (pointing to device, stored in host memory)
     * NAME: the variable name
     *
     * EXAMPLES:
     * `h_vec_delays_by_pre` - an array [size = num_pre_post_blocks] of host
     *                         vectors, each storing delay values of a
     *                         (preID, postBlock) pair
     * `h_num_synapses_by_bundle` - a host vector of integers specifying the
     *                              number of synapses in a bundle
     * `d_ptr_synapse_ids` - a device pointer to synapse IDs (all of them)
     */

    // synapse IDs for each (preID, postBlock) pair
    vector_t<int32_t>* h_vec_synapse_ids_by_pre = new vector_t<int32_t>[num_pre_post_blocks];
    // array of synapse IDs in device memory for each (preID, postBlock) pair
    int32_t** d_ptr_synapse_ids_by_pre;
    // number of synapses for each (preID, postBlock) pair
    int* h_num_synapses_by_pre;



    // we need to allocate device memory for synapse IDs independent of delay mode
    int32_t* d_ptr_synapse_ids;
    size_t memory_synapse_ids = sizeof(int32_t) * syn_N;
    CUDA_SAFE_CALL(
            hipMalloc((void**)&d_ptr_synapse_ids, memory_synapse_ids)
            );
    memory_recorder.push_back(std::make_tuple("synapse IDs", memory_synapse_ids, syn_N));


    //fill vectors of connectivity matrix with synapse IDs and delays (in units of simulation time step)
    int max_delay = (int)(_dynamic_array_synapses_1_delay[0] / dt + 0.5);
    for(int syn_id = 0; syn_id < syn_N; syn_id++)  // loop through all synapses
    {


        // Code generation checks
        assert(0 == 0);

        assert(0 == 0);

        // pre/post_neuron_id are integers from 0 to Nsource/Ntarget (from corresponding
        // SynapticPathway) this is relevant only when using Subgroups where they might
        // be NOT equal to the idx in their NeuronGroup
        int32_t pre_neuron_id = _dynamic_array_synapses_1__synaptic_pre[syn_id] - 0;
        int32_t post_neuron_id = _dynamic_array_synapses_1__synaptic_post[syn_id] - 0;


        // each parallel executed cuda block gets an equal part of post neuron IDs
        int post_block_id = (post_neuron_id * num_parallel_blocks) / target_N;
        // we store synapses for each pre neuron and post block
        int pre_post_block_id = pre_neuron_id * num_parallel_blocks + post_block_id;

        h_vec_synapse_ids_by_pre[pre_post_block_id].push_back(syn_id);
    }
    int num_queues = max_delay + 1;  // we also need a current step

    synapses_1_pre_delay = max_delay;
    // Delete delay (in sec) on device, we don't need it
    // TODO: don't copy these delays to the device in first place, see #83
    dev_dynamic_array_synapses_1_delay.clear();
    dev_dynamic_array_synapses_1_delay.shrink_to_fit();
    CUDA_CHECK_MEMORY();
    size_t used_device_memory_after_dealloc = used_device_memory;

    ///////////////////////////////////////////////////////
    // Memory allocations which depend on the delay mode //
    ///////////////////////////////////////////////////////

    {
        h_num_synapses_by_pre = new int[num_pre_post_blocks];
        d_ptr_synapse_ids_by_pre = new int32_t*[num_pre_post_blocks];
    }


    // loop through connectivity matrix [(preID, postBlock) pairs]
    for(int i = 0; i < num_pre_post_blocks; i++)
    {
        // i is pre_post_block_id

        int num_elements = h_vec_synapse_ids_by_pre[i].size();
        size_connectivity_matrix += num_elements;
        if (num_elements > synapses_1_pre_max_size)
            synapses_1_pre_max_size = num_elements;

        {
            // copy the synapse IDs and the number of synapses for this
            // (preID, postBlock) to device and store the device pointer

            h_num_synapses_by_pre[i] = num_elements;

            d_ptr_synapse_ids_by_pre[i] = d_ptr_synapse_ids + sum_num_elements;
            CUDA_SAFE_CALL(
                    hipMemcpy(d_ptr_synapse_ids_by_pre[i],
                        thrust::raw_pointer_cast(&(h_vec_synapse_ids_by_pre[i][0])),
                        sizeof(int32_t) * num_elements,
                        hipMemcpyHostToDevice)
                    );
        }

        sum_num_elements += num_elements;
        updateMeanStd(count_num_elements, mean_num_elements, M2_num_elements, num_elements);
    }  // end for loop through connectivity matrix
    printf("INFO connectivity matrix has size %i, number of (pre neuron ID, post neuron block) pairs is %u\n",
            size_connectivity_matrix, num_pre_post_blocks);

    {
        // synapses size
        COPY_HOST_ARRAY_TO_DEVICE_SYMBOL(h_num_synapses_by_pre,
                synapses_1_pre_num_synapses_by_pre, num_pre_post_blocks,
                "number of synapses per pre/post block");
        // synapses id
        COPY_HOST_ARRAY_TO_DEVICE_SYMBOL(d_ptr_synapse_ids_by_pre,
                synapses_1_pre_synapse_ids_by_pre, num_pre_post_blocks,
                "pointers to synapse IDs");
    }


    ////////////////////////////////////////////////////
    //// PRINT INFORMATION ON MEMORY USAGE AND TIME ////
    ////////////////////////////////////////////////////

    // TODO print statistics!

    // sum all allocated memory
    size_t total_memory = 0;
    int max_string_length = 0;
    for(auto const& tuple: memory_recorder){
        total_memory += std::get<1>(tuple);
        int str_len = std::get<0>(tuple).length();
        if (str_len > max_string_length)
            max_string_length = str_len;
    }
    double total_memory_MB = total_memory * to_MB;
    max_string_length += 5;

    // sort tuples by used memory
    std::sort(begin(memory_recorder), end(memory_recorder),
            [](tuple_t const &t1, tuple_t const &t2) {
            return std::get<1>(t1) > std::get<1>(t2); // or use a custom compare function
            }
            );

    double std_num_elements = getStd(count_num_elements, M2_num_elements);

    // print memory information
    std::cout.precision(1);
    std::cout.setf(std::ios::fixed, std::ios::floatfield);
    std::cout << "INFO: synapse statistics and memory usage for synapses_1_pre:\n"
        << "\tnumber of synapses: " << syn_N << "\n"
        << "\tnumber of pre/post blocks: " << num_pre_post_blocks << "\n"
        << "\tnumber of synapses over all pre/post blocks:\n"
        << "\t\tmean: " << mean_num_elements << "\tstd: "
            << std_num_elements << "\n"
    << "\n\tmemory usage: TOTAL: " << total_memory_MB << " MB (~"
        << total_memory_MB / syn_N * 1024.0 * 1024.0  << " byte per synapse)"
        << std::endl;

    for(auto const& tuple: memory_recorder){
        std::string name;
        size_t bytes;
        int num_elements;
        std::tie(name, bytes, num_elements) = tuple;
        double memory = bytes * to_MB;
        double fraction = memory / total_memory_MB * 100;
        std::cout << "\t\t" << std::setprecision(1) << std::fixed << fraction
            << "%\t" << std::setprecision(3) << std::fixed << memory << " MB\t"
            << name << " [" << num_elements << "]" << std::endl;
    }


    // Create circular eventspaces in no_or_const_delay_mode
    {
        int num_spikespaces = dev_array_neurongroup__spikespace.size();
        if (num_queues > num_spikespaces)
        {
            for (int i = num_spikespaces; i < num_queues; i++)
            {
                int32_t* new_eventspace;
                hipError_t status = hipMalloc((void**)&new_eventspace,
                        sizeof(int32_t)*_num__array_neurongroup__spikespace);
                if (status != hipSuccess)
                {
                    printf("ERROR while allocating momory for dev_array_neurongroup__spikespace[%i] on device: %s %s %d\n",
                            i, hipGetErrorString(status), __FILE__, __LINE__);
                    exit(status);
                }
                dev_array_neurongroup__spikespace.push_back(new_eventspace);
            }
        }
    }

    int num_threads = num_queues;
    if(num_threads >= max_threads_per_block)
    {
        num_threads = max_threads_per_block;
    }
    int num_blocks = 1;

    // check if we have enough ressources to call kernel with given number
    // of blocks and threads
    struct hipFuncAttributes funcAttrib;
    hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(_run_synapses_1_pre_initialise_queue_kernel));
    if (num_threads > funcAttrib.maxThreadsPerBlock)
    {
        // use the max num_threads before launch failure
        num_threads = funcAttrib.maxThreadsPerBlock;
        printf("WARNING Not enough ressources available to call "
               "_run_synapses_1_pre_initialise_queue_kernel "
               "with maximum possible threads per block (%u). "
               "Reducing num_threads to %u. (Kernel needs %i "
               "registers per block, %i bytes of "
               "statically-allocated shared memory per block, %i "
               "bytes of local memory per thread and a total of %i "
               "bytes of user-allocated constant memory)\n",
               max_threads_per_block, num_threads, funcAttrib.numRegs,
               funcAttrib.sharedSizeBytes, funcAttrib.localSizeBytes,
               funcAttrib.constSizeBytes);
    }
    else
    {
        printf("INFO _run_synapses_1_pre_initialise_queue_kernel\n"
               "\t%u blocks\n"
               "\t%u threads\n"
               "\t%i registers per block\n"
               "\t%i bytes statically-allocated shared memory per block\n"
               "\t%i bytes local memory per thread\n"
               "\t%i bytes user-allocated constant memory\n"
               "",
               num_blocks, num_threads, funcAttrib.numRegs,
               funcAttrib.sharedSizeBytes, funcAttrib.localSizeBytes,
               funcAttrib.constSizeBytes);
    }

    _run_synapses_1_pre_initialise_queue_kernel<<<num_blocks, num_threads>>>(
        source_N,
        num_parallel_blocks,
        num_threads,
        dt,
        syn_N,
        num_queues,
        true
    );

    {
        delete [] h_num_synapses_by_pre;
        delete [] d_ptr_synapse_ids_by_pre;
    }

    //delete temp arrays
    delete [] h_vec_synapse_ids_by_pre;

    synapses_1_pre_scalar_delay = true;

    hipError_t status = hipGetLastError();
    if (status != hipSuccess)
    {
        printf("ERROR initialising synapses_1_pre in %s:%d %s\n",
                __FILE__, __LINE__, hipGetErrorString(status));
        _dealloc_arrays();
        exit(status);
    }

    CUDA_CHECK_MEMORY();
    double time_passed = (double)(std::clock() - start_timer) / CLOCKS_PER_SEC;
    std::cout << "INFO: synapses_1_pre initialisation took " <<  time_passed << "s";
    if (used_device_memory_after_dealloc < used_device_memory_start){
        size_t freed_bytes = used_device_memory_start - used_device_memory_after_dealloc;
        std::cout << ", freed " << freed_bytes * to_MB << "MB";
    }
    if (used_device_memory > used_device_memory_start){
        size_t used_bytes = used_device_memory - used_device_memory_start;
        std::cout << " and used " << used_bytes * to_MB << "MB of device memory.";
    }
    std::cout << std::endl;
}

