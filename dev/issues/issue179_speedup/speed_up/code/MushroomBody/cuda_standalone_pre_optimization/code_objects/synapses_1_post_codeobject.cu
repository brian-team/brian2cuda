#include "hip/hip_runtime.h"
#include "objects.h"
#include "code_objects/synapses_1_post_codeobject.h"
#include "brianlib/common_math.h"
#include "brianlib/cuda_utils.h"
#include "brianlib/stdint_compat.h"
#include <cmath>
#include <stdint.h>
#include <ctime>
#include <stdio.h>

#include <stdint.h>
#include "synapses_classes.h"


////// SUPPORT CODE ///////
namespace {
    double _host_rand(const int _vectorisation_idx);
    double _host_randn(const int _vectorisation_idx);
    int32_t _host_poisson(double _lambda, const int _vectorisation_idx);

    ///// block extra_device_helper /////

    ///// support_code_lines /////
        
    inline __host__ __device__
    double _brian_clip(const double value,
                              const double a_min,
                              const double a_max)
    {
        if (value < a_min)
            return a_min;
        if (value > a_max)
            return a_max;
        return value;
    }
    template <typename T>
    __host__ __device__
    double _brian_exp(T value)
    {
    #if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
        return exp((double)value);
    #else
        return exp(value);
    #endif
    }
    inline __host__ __device__
    float _brian_exp(float value)
    {
        return exp(value);
    }
    template < typename T1, typename T2 > struct _higher_type;
    template < > struct _higher_type<int,int> { typedef int type; };
    template < > struct _higher_type<int,long> { typedef long type; };
    template < > struct _higher_type<int,long long> { typedef long long type; };
    template < > struct _higher_type<int,float> { typedef float type; };
    template < > struct _higher_type<int,double> { typedef double type; };
    template < > struct _higher_type<long,int> { typedef long type; };
    template < > struct _higher_type<long,long> { typedef long type; };
    template < > struct _higher_type<long,long long> { typedef long long type; };
    template < > struct _higher_type<long,float> { typedef float type; };
    template < > struct _higher_type<long,double> { typedef double type; };
    template < > struct _higher_type<long long,int> { typedef long long type; };
    template < > struct _higher_type<long long,long> { typedef long long type; };
    template < > struct _higher_type<long long,long long> { typedef long long type; };
    template < > struct _higher_type<long long,float> { typedef float type; };
    template < > struct _higher_type<long long,double> { typedef double type; };
    template < > struct _higher_type<float,int> { typedef float type; };
    template < > struct _higher_type<float,long> { typedef float type; };
    template < > struct _higher_type<float,long long> { typedef float type; };
    template < > struct _higher_type<float,float> { typedef float type; };
    template < > struct _higher_type<float,double> { typedef double type; };
    template < > struct _higher_type<double,int> { typedef double type; };
    template < > struct _higher_type<double,long> { typedef double type; };
    template < > struct _higher_type<double,long long> { typedef double type; };
    template < > struct _higher_type<double,float> { typedef double type; };
    template < > struct _higher_type<double,double> { typedef double type; };
    template < typename T1, typename T2 >
    __host__ __device__ static inline typename _higher_type<T1,T2>::type
    _brian_mod(T1 x, T2 y)
    {{
        return x-y*floor(1.0*x/y);
    }}
    template < typename T1, typename T2 >
    __host__ __device__ static inline typename _higher_type<T1,T2>::type
    _brian_floordiv(T1 x, T2 y)
    {{
        return floor(1.0*x/y);
    }}
    #ifdef _MSC_VER
    #define _brian_pow(x, y) (pow((double)(x), (y)))
    #else
    #define _brian_pow(x, y) (pow((x), (y)))
    #endif
                    inline __device__ int _brian_atomicAdd(int* address, int val)
                    {
            // hardware implementation
            return atomicAdd(address, val);
                    }
                    inline __device__ float _brian_atomicAdd(float* address, float val)
                    {
            // hardware implementation
            return atomicAdd(address, val);
                    }
                    inline __device__ double _brian_atomicAdd(double* address, double val)
                    {
                            #if (__CUDA_ARCH__ >= 600)
            // hardware implementation
            return atomicAdd(address, val);
                            #else
            // software implementation
            unsigned long long int* address_as_int = (unsigned long long int*)address;
            unsigned long long int old = *address_as_int, assumed;
            do {
                assumed = old;
                old = atomicCAS(address_as_int, assumed,
                                __double_as_longlong(val +
                                       __longlong_as_double(assumed)));
            // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
            } while (assumed != old);
            return __longlong_as_double(old);
                            #endif
                    }
                    inline __device__ int _brian_atomicMul(int* address, int val)
                    {
                        // software implementation
                        int old = *address, assumed;
                        do {
                            assumed = old;
                            old = atomicCAS(address, assumed, val * assumed);
                        } while (assumed != old);
                        return old;
                    }
                    inline __device__ float _brian_atomicMul(float* address, float val)
                    {
            // software implementation
            int* address_as_int = (int*)address;
            int old = *address_as_int, assumed;
            do {
                assumed = old;
                old = atomicCAS(address_as_int, assumed,
                                __float_as_int(val *
                                       __int_as_float(assumed)));
            // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
            } while (assumed != old);
            return __int_as_float(old);
                    }
                    inline __device__ double _brian_atomicMul(double* address, double val)
                    {
            // software implementation
            unsigned long long int* address_as_int = (unsigned long long int*)address;
            unsigned long long int old = *address_as_int, assumed;
            do {
                assumed = old;
                old = atomicCAS(address_as_int, assumed,
                                __double_as_longlong(val *
                                       __longlong_as_double(assumed)));
            // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
            } while (assumed != old);
            return __longlong_as_double(old);
                    }
                    inline __device__ int _brian_atomicDiv(int* address, int val)
                    {
                        // software implementation
                        int old = *address, assumed;
                        do {
                            assumed = old;
                            old = atomicCAS(address, assumed, val / assumed);
                        } while (assumed != old);
                        return old;
                    }
                    inline __device__ float _brian_atomicDiv(float* address, float val)
                    {
            // software implementation
            int* address_as_int = (int*)address;
            int old = *address_as_int, assumed;
            do {
                assumed = old;
                old = atomicCAS(address_as_int, assumed,
                                __float_as_int(val /
                                       __int_as_float(assumed)));
            // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
            } while (assumed != old);
            return __int_as_float(old);
                    }
                    inline __device__ double _brian_atomicDiv(double* address, double val)
                    {
            // software implementation
            unsigned long long int* address_as_int = (unsigned long long int*)address;
            unsigned long long int old = *address_as_int, assumed;
            do {
                assumed = old;
                old = atomicCAS(address_as_int, assumed,
                                __double_as_longlong(val /
                                       __longlong_as_double(assumed)));
            // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
            } while (assumed != old);
            return __longlong_as_double(old);
                    }


    // Implement dummy functions such that the host compiled code of binomial
    // functions works. Hacky, hacky ...
    double _host_rand(const int _vectorisation_idx)
    {
        printf("ERROR: Called dummy function `_host_rand` in %s:%d\n", __FILE__,
                __LINE__);
        exit(EXIT_FAILURE);
    }
    double _host_randn(const int _vectorisation_idx)
    {
        printf("ERROR: Called dummy function `_host_rand` in %s:%d\n", __FILE__,
                __LINE__);
        exit(EXIT_FAILURE);
    }
    int32_t _host_poisson(double _lambda, const int _vectorisation_idx)
    {
        printf("ERROR: Called dummy function `_host_poisson` in %s:%d\n", __FILE__,
                __LINE__);
        exit(EXIT_FAILURE);
    }
}

////// hashdefine_lines ///////




__global__ void
kernel_synapses_1_post_codeobject(
    int _N,
    int bid_offset,
    int timestep,
    int THREADS_PER_BLOCK,
    int threads_per_bundle,
    int32_t* eventspace,
    int neurongroup_size,
    ///// KERNEL_PARAMETERS /////
    double* _ptr_array_synapses_1_Apost,
	const int _numApost,
	double* _ptr_array_synapses_1_Apre,
	const int _numApre,
	int32_t* _ptr_array_synapses_1_N,
	int32_t* _ptr_array_synapses_1__synaptic_pre,
	const int _num_synaptic_pre,
	double* _ptr_array_synapses_1_g_raw,
	const int _numg_raw,
	double* _ptr_array_synapses_1_lastupdate,
	const int _numlastupdate,
	const double _value_array_defaultclock_t
    )
{
    using namespace brian;

    assert(THREADS_PER_BLOCK == blockDim.x);

    int tid = threadIdx.x;
    int bid = blockIdx.x + bid_offset;
    //TODO: do we need _idx here? if no, get also rid of scoping after scalar code
    // scalar_code can depend on _idx (e.g. if the state update depends on a
    // subexpression that is the same for all synapses, ?)
    int _idx = bid * THREADS_PER_BLOCK + tid;
    int _vectorisation_idx = _idx;

    ///// KERNEL_CONSTANTS /////
    const int _numN = 1;

    ///// kernel_lines /////
        
    const double* _ptr_array_defaultclock_t = &_value_array_defaultclock_t;



    ///// scalar_code /////
        
    const double _lio_1 = 1.0f*1.0/0.01;
    const double _lio_2 = 1.0f*1.0/0.01;


    {  // _idx is defined in outer and inner scope (for `scalar_code`)
        if (synapses_1_post.no_or_const_delay_mode)
        {
            // TODO: pass as kernel parameter instead?
            int num_parallel_blocks = synapses_1_post.queue->num_blocks;
            int32_t spikes_start = synapses_1_post.spikes_start;
            int32_t spikes_stop = synapses_1_post.spikes_stop;

            // for the first delay timesteps the eventspace is not yet filled
            // note that num_queues is the number of eventspaces, num_queues-1 the delay in timesteps
            if (timestep >= synapses_1_post.queue->num_queues - 1)
            {
                // `spiking_neuron_idx` runs through the eventspace
                // `post_block_idx` runs through the post neuron blocks of the connectivity matrix
                int post_block_idx = bid;
                // loop through neurons in eventspace (indices of event neurons, rest -1)
                for(int spiking_neuron_idx = 0;
                        spiking_neuron_idx < neurongroup_size;
                        spiking_neuron_idx++)
                {

                    // spiking_neuron is index in NeuronGroup
                    int32_t spiking_neuron = eventspace[spiking_neuron_idx];

                    if(spiking_neuron == -1) // end of spiking neurons
                    {
                        assert(spiking_neuron_idx == eventspace[neurongroup_size]);
                        return;
                    }

                    // apply effects if event neuron is in sources of current SynapticPathway
                    if(spikes_start <= spiking_neuron && spiking_neuron < spikes_stop)
                    {
                        int pre_post_block_id = (spiking_neuron - spikes_start) * num_parallel_blocks + post_block_idx;
                        int num_synapses = synapses_1_post_num_synapses_by_pre[pre_post_block_id];
                        int32_t* propagating_synapses = synapses_1_post_synapse_ids_by_pre[pre_post_block_id];
                        for(int j = tid; j < num_synapses; j+=THREADS_PER_BLOCK)
                        {
                            // _idx is the synapse id
                            int32_t _idx = propagating_synapses[j];
                            _vectorisation_idx = j;

                            ///// vector_code /////
                                                        
                            double g_raw = _ptr_array_synapses_1_g_raw[_idx];
                            double Apre = _ptr_array_synapses_1_Apre[_idx];
                            const double t = _ptr_array_defaultclock_t[0];
                            double lastupdate = _ptr_array_synapses_1_lastupdate[_idx];
                            double Apost = _ptr_array_synapses_1_Apost[_idx];
                            const double _Apost = Apost * _brian_exp(_lio_1 * (- (t - lastupdate)));
                            const double _Apre = Apre * _brian_exp(_lio_2 * (- (t - lastupdate)));
                            Apost = _Apost;
                            Apre = _Apre;
                            Apost += (- 1.0000000000000002e-10);
                            g_raw = _brian_clip(g_raw + Apre, 0, 3.7500000000000005e-09);
                            lastupdate = t;
                            _ptr_array_synapses_1_Apost[_idx] = Apost;
                            _ptr_array_synapses_1_g_raw[_idx] = g_raw;
                            _ptr_array_synapses_1_Apre[_idx] = Apre;
                            _ptr_array_synapses_1_lastupdate[_idx] = lastupdate;

                        }
                    }

                    __syncthreads();
                }
            }
        }
        else  // heterogeneous delay mode
        {
            cudaVector<int32_t>* synapses_queue;
            synapses_1_post.queue->peek(&synapses_queue);

            int queue_size = synapses_queue[bid].size();

            // use a fixed number of threads per bundle, i runs through all those threads of all bundles
            // for threads_per_bundle == 1, we have one thread per bundle (parallel)
            for (int i = tid; i < queue_size*threads_per_bundle; i+=THREADS_PER_BLOCK)
            {
                // bundle_idx runs through all bundles
                int bundle_idx = i / threads_per_bundle;
                // syn_in_bundle_idx runs through all threads in a single bundle
                int syn_in_bundle_idx = i % threads_per_bundle;

                int bundle_id = synapses_queue[bid].at(bundle_idx);
                int bundle_size = synapses_1_post_num_synapses_by_bundle[bundle_id];
                int synapses_offset = synapses_1_post_synapses_offset_by_bundle[bundle_id];
                int32_t* synapse_ids = synapses_1_post_synapse_ids;
                int32_t* synapse_bundle = synapse_ids + synapses_offset;

                // loop through synapses of this bundle with all available threads_per_bundle
                // if threads_per_bundle == 1, this is serial
                for (int j = syn_in_bundle_idx; j < bundle_size; j+=threads_per_bundle)
                {
                    int32_t _idx = synapse_bundle[j];


                            ///// vector_code /////
                                                        
                            double g_raw = _ptr_array_synapses_1_g_raw[_idx];
                            double Apre = _ptr_array_synapses_1_Apre[_idx];
                            const double t = _ptr_array_defaultclock_t[0];
                            double lastupdate = _ptr_array_synapses_1_lastupdate[_idx];
                            double Apost = _ptr_array_synapses_1_Apost[_idx];
                            const double _Apost = Apost * _brian_exp(_lio_1 * (- (t - lastupdate)));
                            const double _Apre = Apre * _brian_exp(_lio_2 * (- (t - lastupdate)));
                            Apost = _Apost;
                            Apre = _Apre;
                            Apost += (- 1.0000000000000002e-10);
                            g_raw = _brian_clip(g_raw + Apre, 0, 3.7500000000000005e-09);
                            lastupdate = t;
                            _ptr_array_synapses_1_Apost[_idx] = Apost;
                            _ptr_array_synapses_1_g_raw[_idx] = g_raw;
                            _ptr_array_synapses_1_Apre[_idx] = Apre;
                            _ptr_array_synapses_1_lastupdate[_idx] = lastupdate;

                        }
                    }
                }
            }
        }


void _run_synapses_1_post_codeobject()
{
    using namespace brian;


    const int _N = _array_synapses_1_N[0];

    ///// HOST_CONSTANTS ///////////
    double* const dev_array_synapses_1_Apost = thrust::raw_pointer_cast(&dev_dynamic_array_synapses_1_Apost[0]);
		const int _numApost = dev_dynamic_array_synapses_1_Apost.size();
		double* const dev_array_synapses_1_Apre = thrust::raw_pointer_cast(&dev_dynamic_array_synapses_1_Apre[0]);
		const int _numApre = dev_dynamic_array_synapses_1_Apre.size();
		const int _numN = 1;
		int32_t* const dev_array_synapses_1__synaptic_pre = thrust::raw_pointer_cast(&dev_dynamic_array_synapses_1__synaptic_pre[0]);
		const int _num_synaptic_pre = dev_dynamic_array_synapses_1__synaptic_pre.size();
		double* const dev_array_synapses_1_g_raw = thrust::raw_pointer_cast(&dev_dynamic_array_synapses_1_g_raw[0]);
		const int _numg_raw = dev_dynamic_array_synapses_1_g_raw.size();
		double* const dev_array_synapses_1_lastupdate = thrust::raw_pointer_cast(&dev_dynamic_array_synapses_1_lastupdate[0]);
		const int _numlastupdate = dev_dynamic_array_synapses_1_lastupdate.size();

static int num_threads_per_bundle;
static int num_loops;

    static int num_threads, num_blocks;
    static bool first_run = true;
    if (first_run)
    {
// Synaptic effects modify only synapse variables.
num_blocks = num_parallel_blocks;
num_threads = max_threads_per_block;
// TODO: effect of mean instead of max?
num_threads_per_bundle = synapses_1_post_max_bundle_size;
num_loops = 1;


        // calculate theoretical occupancy
        int max_active_blocks;
        CUDA_SAFE_CALL(
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks,
                    kernel_synapses_1_post_codeobject, num_threads, 0)
                );

        float occupancy = (max_active_blocks * num_threads / num_threads_per_warp) /
                          (float)(max_threads_per_sm / num_threads_per_warp);


        // check if we have enough ressources to call kernel with given number
        // of blocks and threads (can only occur for the else case above as for the
        // first max. occupancy)
        struct hipFuncAttributes funcAttrib;
        CUDA_SAFE_CALL(
                hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(kernel_synapses_1_post_codeobject))
                );
        if (num_threads > funcAttrib.maxThreadsPerBlock)
        {
            // use the max num_threads before launch failure
            num_threads = funcAttrib.maxThreadsPerBlock;
            printf("WARNING Not enough ressources available to call "
                   "kernel_synapses_1_post_codeobject "
                   "with maximum possible threads per block (%u). "
                   "Reducing num_threads to %u. (Kernel needs %i "
                   "registers per block, %i bytes of "
                   "statically-allocated shared memory per block, %i "
                   "bytes of local memory per thread and a total of %i "
                   "bytes of user-allocated constant memory)\n",
                   max_threads_per_block, num_threads, funcAttrib.numRegs,
                   funcAttrib.sharedSizeBytes, funcAttrib.localSizeBytes,
                   funcAttrib.constSizeBytes);

            // calculate theoretical occupancy for new num_threads
            CUDA_SAFE_CALL(
                    hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks,
                        kernel_synapses_1_post_codeobject, num_threads, 0)
                    );

            occupancy = (max_active_blocks * num_threads / num_threads_per_warp) /
                        (float)(max_threads_per_sm / num_threads_per_warp);
        }
else if (synapses_1_post_max_size <= 0)
{
    printf("INFO there are no synapses in the synapses_1_post pathway. Skipping synapses_push and synapses kernels.\n");
}
        else
        {
            printf("INFO kernel_synapses_1_post_codeobject\n"
                   "\t%u blocks\n"
                   "\t%u threads\n"
                   "\t%i registers per block\n"
                   "\t%i bytes statically-allocated shared memory per block\n"
                   "\t%i bytes local memory per thread\n"
                   "\t%i bytes user-allocated constant memory\n"
                   "\t%.3f theoretical occupancy\n",
                   num_blocks, num_threads, funcAttrib.numRegs,
                   funcAttrib.sharedSizeBytes, funcAttrib.localSizeBytes,
                   funcAttrib.constSizeBytes, occupancy);
        }
        first_run = false;
    }


// only call kernel if we have synapses (otherwise we skipped the push kernel)
if (synapses_1_post_max_size > 0)
{
        for(int bid_offset = 0; bid_offset < num_loops; bid_offset++)
        {
            kernel_synapses_1_post_codeobject<<<num_blocks, num_threads>>>(
                _N,
                bid_offset,
                defaultclock.timestep[0],
                num_threads,
                num_threads_per_bundle,
                dev_array_neurongroup_1__spikespace[synapses_1_post_eventspace_idx],
                _num__array_neurongroup_1__spikespace-1,
                ///// HOST_PARAMETERS /////
                dev_array_synapses_1_Apost,
			_numApost,
			dev_array_synapses_1_Apre,
			_numApre,
			dev_array_synapses_1_N,
			dev_array_synapses_1__synaptic_pre,
			_num_synaptic_pre,
			dev_array_synapses_1_g_raw,
			_numg_raw,
			dev_array_synapses_1_lastupdate,
			_numlastupdate,
			_array_defaultclock_t[0]
            );
        }

    CUDA_CHECK_ERROR("kernel_synapses_1_post_codeobject");
}


}

void _debugmsg_synapses_1_post_codeobject()
{
    using namespace brian;
    std::cout << "Number of synapses: " << _array_synapses_1_N[0] << endl;
}

