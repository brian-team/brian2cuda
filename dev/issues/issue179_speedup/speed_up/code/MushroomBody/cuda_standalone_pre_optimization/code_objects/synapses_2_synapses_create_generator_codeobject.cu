#include "hip/hip_runtime.h"
#include "objects.h"
#include "code_objects/synapses_2_synapses_create_generator_codeobject.h"
#include "brianlib/common_math.h"
#include "brianlib/cuda_utils.h"
#include "brianlib/stdint_compat.h"
#include <cmath>
#include <stdint.h>
#include <ctime>
#include <stdio.h>

#include <stdint.h>
#include "synapses_classes.h"

#include<iostream>
#include<hiprand.h>
#include<brianlib/curand_buffer.h>
#include "brianlib/cuda_utils.h"
#include<map>


////// SUPPORT CODE ///////
namespace {
    double _host_rand(const int _vectorisation_idx);
    double _host_randn(const int _vectorisation_idx);
    int32_t _host_poisson(double _lambda, const int _vectorisation_idx);

    ///// block extra_device_helper /////

    ///// support_code_lines /////
        
    #define _rand(vectorisation_idx) (_ptr_array_synapses_2_synapses_create_generator_codeobject_rand[vectorisation_idx])
    template < typename T1, typename T2 > struct _higher_type;
    template < > struct _higher_type<int,int> { typedef int type; };
    template < > struct _higher_type<int,long> { typedef long type; };
    template < > struct _higher_type<int,long long> { typedef long long type; };
    template < > struct _higher_type<int,float> { typedef float type; };
    template < > struct _higher_type<int,double> { typedef double type; };
    template < > struct _higher_type<long,int> { typedef long type; };
    template < > struct _higher_type<long,long> { typedef long type; };
    template < > struct _higher_type<long,long long> { typedef long long type; };
    template < > struct _higher_type<long,float> { typedef float type; };
    template < > struct _higher_type<long,double> { typedef double type; };
    template < > struct _higher_type<long long,int> { typedef long long type; };
    template < > struct _higher_type<long long,long> { typedef long long type; };
    template < > struct _higher_type<long long,long long> { typedef long long type; };
    template < > struct _higher_type<long long,float> { typedef float type; };
    template < > struct _higher_type<long long,double> { typedef double type; };
    template < > struct _higher_type<float,int> { typedef float type; };
    template < > struct _higher_type<float,long> { typedef float type; };
    template < > struct _higher_type<float,long long> { typedef float type; };
    template < > struct _higher_type<float,float> { typedef float type; };
    template < > struct _higher_type<float,double> { typedef double type; };
    template < > struct _higher_type<double,int> { typedef double type; };
    template < > struct _higher_type<double,long> { typedef double type; };
    template < > struct _higher_type<double,long long> { typedef double type; };
    template < > struct _higher_type<double,float> { typedef double type; };
    template < > struct _higher_type<double,double> { typedef double type; };
    template < typename T1, typename T2 >
    __host__ __device__ static inline typename _higher_type<T1,T2>::type
    _brian_mod(T1 x, T2 y)
    {{
        return x-y*floor(1.0*x/y);
    }}
    template < typename T1, typename T2 >
    __host__ __device__ static inline typename _higher_type<T1,T2>::type
    _brian_floordiv(T1 x, T2 y)
    {{
        return floor(1.0*x/y);
    }}
    #ifdef _MSC_VER
    #define _brian_pow(x, y) (pow((double)(x), (y)))
    #else
    #define _brian_pow(x, y) (pow((x), (y)))
    #endif
                    inline __device__ int _brian_atomicAdd(int* address, int val)
                    {
            // hardware implementation
            return atomicAdd(address, val);
                    }
                    inline __device__ float _brian_atomicAdd(float* address, float val)
                    {
            // hardware implementation
            return atomicAdd(address, val);
                    }
                    inline __device__ double _brian_atomicAdd(double* address, double val)
                    {
                            #if (__CUDA_ARCH__ >= 600)
            // hardware implementation
            return atomicAdd(address, val);
                            #else
            // software implementation
            unsigned long long int* address_as_int = (unsigned long long int*)address;
            unsigned long long int old = *address_as_int, assumed;
            do {
                assumed = old;
                old = atomicCAS(address_as_int, assumed,
                                __double_as_longlong(val +
                                       __longlong_as_double(assumed)));
            // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
            } while (assumed != old);
            return __longlong_as_double(old);
                            #endif
                    }
                    inline __device__ int _brian_atomicMul(int* address, int val)
                    {
                        // software implementation
                        int old = *address, assumed;
                        do {
                            assumed = old;
                            old = atomicCAS(address, assumed, val * assumed);
                        } while (assumed != old);
                        return old;
                    }
                    inline __device__ float _brian_atomicMul(float* address, float val)
                    {
            // software implementation
            int* address_as_int = (int*)address;
            int old = *address_as_int, assumed;
            do {
                assumed = old;
                old = atomicCAS(address_as_int, assumed,
                                __float_as_int(val *
                                       __int_as_float(assumed)));
            // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
            } while (assumed != old);
            return __int_as_float(old);
                    }
                    inline __device__ double _brian_atomicMul(double* address, double val)
                    {
            // software implementation
            unsigned long long int* address_as_int = (unsigned long long int*)address;
            unsigned long long int old = *address_as_int, assumed;
            do {
                assumed = old;
                old = atomicCAS(address_as_int, assumed,
                                __double_as_longlong(val *
                                       __longlong_as_double(assumed)));
            // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
            } while (assumed != old);
            return __longlong_as_double(old);
                    }
                    inline __device__ int _brian_atomicDiv(int* address, int val)
                    {
                        // software implementation
                        int old = *address, assumed;
                        do {
                            assumed = old;
                            old = atomicCAS(address, assumed, val / assumed);
                        } while (assumed != old);
                        return old;
                    }
                    inline __device__ float _brian_atomicDiv(float* address, float val)
                    {
            // software implementation
            int* address_as_int = (int*)address;
            int old = *address_as_int, assumed;
            do {
                assumed = old;
                old = atomicCAS(address_as_int, assumed,
                                __float_as_int(val /
                                       __int_as_float(assumed)));
            // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
            } while (assumed != old);
            return __int_as_float(old);
                    }
                    inline __device__ double _brian_atomicDiv(double* address, double val)
                    {
            // software implementation
            unsigned long long int* address_as_int = (unsigned long long int*)address;
            unsigned long long int old = *address_as_int, assumed;
            do {
                assumed = old;
                old = atomicCAS(address_as_int, assumed,
                                __double_as_longlong(val /
                                       __longlong_as_double(assumed)));
            // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
            } while (assumed != old);
            return __longlong_as_double(old);
                    }


// NOTE: _ptr_array_synapses_2_synapses_create_generator_codeobject_rand is NOT an array
// but an instance of CurandBuffer, which overloads the operator[], which then just
// returns the next random number in the buffer, ignoring the argument passed to operator[]
// NOTE: Put buffers into anonymous namespace such that _host_rand/n and rand/n
// in main code have access to it.
// NOTE: _host_rand/n is used in the host compiled implementation of binomial
// functions. Here, it just returns the next element from the CurandBuffer.
CurandBuffer<randomNumber_t> _ptr_array_synapses_2_synapses_create_generator_codeobject_rand(&brian::curand_generator, RAND);
randomNumber_t _host_rand(const int _vectorisation_idx)
{
    return _ptr_array_synapses_2_synapses_create_generator_codeobject_rand[_vectorisation_idx];
}

CurandBuffer<randomNumber_t> _ptr_array_synapses_2_synapses_create_generator_codeobject_randn(&brian::curand_generator, RANDN);
randomNumber_t _host_randn(const int _vectorisation_idx)
{
    return _ptr_array_synapses_2_synapses_create_generator_codeobject_randn[_vectorisation_idx];
}

// This is the C++ Standalone implementation of the poisson function, which we use
double _loggam(double x) {
  double x0, x2, xp, gl, gl0;
  int32_t k, n;

  static double a[10] = {8.333333333333333e-02, -2.777777777777778e-03,
                         7.936507936507937e-04, -5.952380952380952e-04,
                         8.417508417508418e-04, -1.917526917526918e-03,
                         6.410256410256410e-03, -2.955065359477124e-02,
                         1.796443723688307e-01, -1.39243221690590e+00};
  x0 = x;
  n = 0;
  if ((x == 1.0) || (x == 2.0))
    return 0.0;
  else if (x <= 7.0) {
    n = (int32_t)(7 - x);
    x0 = x + n;
  }
  x2 = 1.0 / (x0 * x0);
  xp = 2 * M_PI;
  gl0 = a[9];
  for (k=8; k>=0; k--) {
    gl0 *= x2;
    gl0 += a[k];
  }
  gl = gl0 / x0 + 0.5 * log(xp) + (x0 - 0.5) * log(x0) - x0;
  if (x <= 7.0) {
    for (k=1; k<=n; k++) {
      gl -= log(x0 - 1.0);
      x0 -= 1.0;
    }
  }
  return gl;
}

int32_t _poisson_mult(double lam, int _vectorisation_idx) {
  int32_t X;
  double prod, U, enlam;

  enlam = exp(-lam);
  X = 0;
  prod = 1.0;
  while (1) {
    U = _rand(_vectorisation_idx);
    prod *= U;
    if (prod > enlam)
      X += 1;
    else
      return X;
  }
}

int32_t _poisson_ptrs(double lam, int _vectorisation_idx) {
  int32_t k;
  double U, V, slam, loglam, a, b, invalpha, vr, us;

  slam = sqrt(lam);
  loglam = log(lam);
  b = 0.931 + 2.53 * slam;
  a = -0.059 + 0.02483 * b;
  invalpha = 1.1239 + 1.1328 / (b - 3.4);
  vr = 0.9277 - 3.6224 / (b - 2);

  while (1) {
    U = _rand(_vectorisation_idx) - 0.5;
    V = _rand(_vectorisation_idx);
    us = 0.5 - abs(U);
    k = (int32_t)floor((2 * a / us + b) * U + lam + 0.43);
    if ((us >= 0.07) && (V <= vr))
      return k;
    if ((k < 0) || ((us < 0.013) && (V > us)))
      continue;
    if ((log(V) + log(invalpha) - log(a / (us * us) + b)) <=
        (-lam + k * loglam - _loggam(k + 1)))
      return k;
  }
}

int32_t _host_poisson(double lam, int32_t _idx) {
  if (lam >= 10)
    return _poisson_ptrs(lam, _idx);
  else if (lam == 0)
    return 0;
  else
    return _poisson_mult(lam, _idx);
}
}

////// hashdefine_lines ///////




void _run_synapses_2_synapses_create_generator_codeobject()
{
    using namespace brian;

std::clock_t start_timer = std::clock();

CUDA_CHECK_MEMORY();
size_t used_device_memory_start = used_device_memory;


    ///// HOST_CONSTANTS ///////////
    const int _numN = 1;
		int32_t* const _array_synapses_2_N_incoming = thrust::raw_pointer_cast(&_dynamic_array_synapses_2_N_incoming[0]);
		const int _numN_incoming = _dynamic_array_synapses_2_N_incoming.size();
		int32_t* const _array_synapses_2_N_outgoing = thrust::raw_pointer_cast(&_dynamic_array_synapses_2_N_outgoing[0]);
		const int _numN_outgoing = _dynamic_array_synapses_2_N_outgoing.size();
		int32_t* const _array_synapses_2__synaptic_post = thrust::raw_pointer_cast(&_dynamic_array_synapses_2__synaptic_post[0]);
		const int _num_synaptic_post = _dynamic_array_synapses_2__synaptic_post.size();
		int32_t* const _array_synapses_2__synaptic_pre = thrust::raw_pointer_cast(&_dynamic_array_synapses_2__synaptic_pre[0]);
		const int _num_synaptic_pre = _dynamic_array_synapses_2__synaptic_pre.size();


    ///// pointers_lines /////
        
    int32_t* __restrict  _ptr_array_synapses_2__synaptic_post = _array_synapses_2__synaptic_post;
    int32_t* __restrict  _ptr_array_synapses_2_N_incoming = _array_synapses_2_N_incoming;
    int32_t* __restrict  _ptr_array_synapses_2__synaptic_pre = _array_synapses_2__synaptic_pre;
    int32_t* __restrict  _ptr_array_synapses_2_N_outgoing = _array_synapses_2_N_outgoing;
    int32_t*   _ptr_array_synapses_2_N = _array_synapses_2_N;


    const int _N_pre = 100;
    const int _N_post = 100;
    _dynamic_array_synapses_2_N_incoming.resize(_N_post + 0);
    _dynamic_array_synapses_2_N_outgoing.resize(_N_pre + 0);

    int _raw_pre_idx, _raw_post_idx;
    const int _vectorisation_idx = -1;
    ///// scalar_code['setup_iterator'] /////
        

    ///// scalar_code['create_j'] /////
        

    ///// scalar_code['create_cond'] /////
        

    ///// scalar_code['update_post'] /////
        


    for(int _i = 0; _i < _N_pre; _i++)
    {

        bool __cond, _cond;
        _raw_pre_idx = _i + 0;
        {
            ///// vector_code['create_cond'] /////
                        
            const char _cond = true;

            __cond = _cond;
        }
        _cond = __cond;
        if(!_cond) continue;
        // Some explanation of this hackery. The problem is that we have multiple code blocks.
        // Each code block is generated independently of the others, and they declare variables
        // at the beginning if necessary (including declaring them as const if their values don't
        // change). However, if two code blocks follow each other in the same C++ scope then
        // that causes a redeclaration error. So we solve it by putting each block inside a
        // pair of braces to create a new scope specific to each code block. However, that brings
        // up another problem: we need the values from these code blocks. I don't have a general
        // solution to this problem, but in the case of this particular template, we know which
        // values we need from them so we simply create outer scoped variables to copy the value
        // into. Later on we have a slightly more complicated problem because the original name
        // _j has to be used, so we create two variables __j, _j at the outer scope, copy
        // _j to __j in the inner scope (using the inner scope version of _j), and then
        // __j to _j in the outer scope (to the outer scope version of _j). This outer scope
        // version of _j will then be used in subsequent blocks.
        long _uiter_low;
        long _uiter_high;
        long _uiter_step;
        {
            ///// vector_code['setup_iterator'] /////
                        
            const int32_t _iter_low = 0;
            const int32_t _iter_high = 100;
            const int32_t _iter_step = 1;

            _uiter_low = _iter_low;
            _uiter_high = _iter_high;
            _uiter_step = _iter_step;
        }
        for(int _k=_uiter_low; _k<_uiter_high; _k+=_uiter_step)
        {
            long __j, _j, _pre_idx, __pre_idx;
            {
            ///// vector_code['create_j'] /////
                                
                const int32_t _pre_idx = _raw_pre_idx;
                const int32_t _j = _k;

                __j = _j; // pick up the locally scoped _j and store in __j
                __pre_idx = _pre_idx;
            }
            _j = __j; // make the previously locally scoped _j available
            _pre_idx = __pre_idx;
            _raw_post_idx = _j + 0;


            if(_j<0 || _j>=_N_post)
            {
                cout << "Error: tried to create synapse to neuron j=" << _j << " outside range 0 to " <<
                        _N_post-1 << endl;
                exit(1);
            }

            ///// vector_code['update_post'] /////
                        
            const int32_t _post_idx = _raw_post_idx;
            const int32_t _n = 1;


            for (int _repetition=0; _repetition<_n; _repetition++) {
                _dynamic_array_synapses_2_N_outgoing[_pre_idx] += 1;
                _dynamic_array_synapses_2_N_incoming[_post_idx] += 1;
                _dynamic_array_synapses_2__synaptic_pre.push_back(_pre_idx);
                _dynamic_array_synapses_2__synaptic_post.push_back(_post_idx);
            }
        }
    }

    // now we need to resize all registered variables
    const int32_t newsize = _dynamic_array_synapses_2__synaptic_pre.size();
            THRUST_CHECK_ERROR(
                    dev_dynamic_array_synapses_2__synaptic_post.resize(newsize)
                    );
            _dynamic_array_synapses_2__synaptic_post.resize(newsize);
            THRUST_CHECK_ERROR(
                    dev_dynamic_array_synapses_2__synaptic_pre.resize(newsize)
                    );
            _dynamic_array_synapses_2__synaptic_pre.resize(newsize);

    // update the total number of synapses
    _ptr_array_synapses_2_N[0] = newsize;

    // Check for occurrence of multiple source-target pairs in synapses ("synapse number")
    std::map<std::pair<int32_t, int32_t>, int32_t> source_target_count;
    for (int _i=0; _i<newsize; _i++)
    {
        // Note that source_target_count will create a new entry initialized
        // with 0 when the key does not exist yet
        const std::pair<int32_t, int32_t> source_target = std::pair<int32_t, int32_t>(_dynamic_array_synapses_2__synaptic_pre[_i], _dynamic_array_synapses_2__synaptic_post[_i]);
        source_target_count[source_target]++;
        //printf("source target count = %i\n", source_target_count[source_target]);
        if (source_target_count[source_target] > 1)
        {
            synapses_2_multiple_pre_post = true;
            break;
        }
    }

    // copy changed host data to device
    dev_dynamic_array_synapses_2_N_incoming = _dynamic_array_synapses_2_N_incoming;
    dev_dynamic_array_synapses_2_N_outgoing = _dynamic_array_synapses_2_N_outgoing;
    dev_dynamic_array_synapses_2__synaptic_pre = _dynamic_array_synapses_2__synaptic_pre;
    dev_dynamic_array_synapses_2__synaptic_post = _dynamic_array_synapses_2__synaptic_post;
    CUDA_SAFE_CALL(
            hipMemcpy(dev_array_synapses_2_N,
                _array_synapses_2_N,
                sizeof(int32_t),
                hipMemcpyHostToDevice)
            );




// free memory in CurandBuffers
_ptr_array_synapses_2_synapses_create_generator_codeobject_rand.free_memory();
_ptr_array_synapses_2_synapses_create_generator_codeobject_randn.free_memory();

CUDA_CHECK_MEMORY();
const double to_MB = 1.0 / (1024.0 * 1024.0);
double tot_memory_MB = (used_device_memory - used_device_memory_start) * to_MB;
double time_passed = (double)(std::clock() - start_timer) / CLOCKS_PER_SEC;
std::cout << "INFO: synapses_2 creation took " <<  time_passed << "s";
if (tot_memory_MB > 0)
    std::cout << " and used " << tot_memory_MB << "MB of memory.";
std::cout << std::endl;
}


