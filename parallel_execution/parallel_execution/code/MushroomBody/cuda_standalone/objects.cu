#include "hip/hip_runtime.h"

#include "objects.h"
#include "synapses_classes.h"
#include "brianlib/clocks.h"
#include "brianlib/cuda_utils.h"
#include "network.h"
#include "rand.h"
#include <stdint.h>
#include <iostream>
#include <fstream>
#include <ctime>
#include <utility>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

size_t brian::used_device_memory = 0;

//////////////// clocks ///////////////////
Clock brian::defaultclock;

//////////////// networks /////////////////
Network brian::magicnetwork;

//////////////// arrays ///////////////////
double * brian::_array_defaultclock_dt;
double * brian::dev_array_defaultclock_dt;
__device__ double * brian::d_array_defaultclock_dt;
const int brian::_num__array_defaultclock_dt = 1;

double * brian::_array_defaultclock_t;
double * brian::dev_array_defaultclock_t;
__device__ double * brian::d_array_defaultclock_t;
const int brian::_num__array_defaultclock_t = 1;

int64_t * brian::_array_defaultclock_timestep;
int64_t * brian::dev_array_defaultclock_timestep;
__device__ int64_t * brian::d_array_defaultclock_timestep;
const int brian::_num__array_defaultclock_timestep = 1;

double * brian::_array_neurongroup_1_g_eKC_eKC;
double * brian::dev_array_neurongroup_1_g_eKC_eKC;
__device__ double * brian::d_array_neurongroup_1_g_eKC_eKC;
const int brian::_num__array_neurongroup_1_g_eKC_eKC = 100;

double * brian::_array_neurongroup_1_g_iKC_eKC;
double * brian::dev_array_neurongroup_1_g_iKC_eKC;
__device__ double * brian::d_array_neurongroup_1_g_iKC_eKC;
const int brian::_num__array_neurongroup_1_g_iKC_eKC = 100;

double * brian::_array_neurongroup_1_h;
double * brian::dev_array_neurongroup_1_h;
__device__ double * brian::d_array_neurongroup_1_h;
const int brian::_num__array_neurongroup_1_h = 100;

int32_t * brian::_array_neurongroup_1_i;
int32_t * brian::dev_array_neurongroup_1_i;
__device__ int32_t * brian::d_array_neurongroup_1_i;
const int brian::_num__array_neurongroup_1_i = 100;

double * brian::_array_neurongroup_1_lastspike;
double * brian::dev_array_neurongroup_1_lastspike;
__device__ double * brian::d_array_neurongroup_1_lastspike;
const int brian::_num__array_neurongroup_1_lastspike = 100;

double * brian::_array_neurongroup_1_m;
double * brian::dev_array_neurongroup_1_m;
__device__ double * brian::d_array_neurongroup_1_m;
const int brian::_num__array_neurongroup_1_m = 100;

double * brian::_array_neurongroup_1_n;
double * brian::dev_array_neurongroup_1_n;
__device__ double * brian::d_array_neurongroup_1_n;
const int brian::_num__array_neurongroup_1_n = 100;

char * brian::_array_neurongroup_1_not_refractory;
char * brian::dev_array_neurongroup_1_not_refractory;
__device__ char * brian::d_array_neurongroup_1_not_refractory;
const int brian::_num__array_neurongroup_1_not_refractory = 100;

double * brian::_array_neurongroup_1_V;
double * brian::dev_array_neurongroup_1_V;
__device__ double * brian::d_array_neurongroup_1_V;
const int brian::_num__array_neurongroup_1_V = 100;

double * brian::_array_neurongroup_g_PN_iKC;
double * brian::dev_array_neurongroup_g_PN_iKC;
__device__ double * brian::d_array_neurongroup_g_PN_iKC;
const int brian::_num__array_neurongroup_g_PN_iKC = 2500;

double * brian::_array_neurongroup_h;
double * brian::dev_array_neurongroup_h;
__device__ double * brian::d_array_neurongroup_h;
const int brian::_num__array_neurongroup_h = 2500;

int32_t * brian::_array_neurongroup_i;
int32_t * brian::dev_array_neurongroup_i;
__device__ int32_t * brian::d_array_neurongroup_i;
const int brian::_num__array_neurongroup_i = 2500;

double * brian::_array_neurongroup_lastspike;
double * brian::dev_array_neurongroup_lastspike;
__device__ double * brian::d_array_neurongroup_lastspike;
const int brian::_num__array_neurongroup_lastspike = 2500;

double * brian::_array_neurongroup_m;
double * brian::dev_array_neurongroup_m;
__device__ double * brian::d_array_neurongroup_m;
const int brian::_num__array_neurongroup_m = 2500;

double * brian::_array_neurongroup_n;
double * brian::dev_array_neurongroup_n;
__device__ double * brian::d_array_neurongroup_n;
const int brian::_num__array_neurongroup_n = 2500;

char * brian::_array_neurongroup_not_refractory;
char * brian::dev_array_neurongroup_not_refractory;
__device__ char * brian::d_array_neurongroup_not_refractory;
const int brian::_num__array_neurongroup_not_refractory = 2500;

double * brian::_array_neurongroup_V;
double * brian::dev_array_neurongroup_V;
__device__ double * brian::d_array_neurongroup_V;
const int brian::_num__array_neurongroup_V = 2500;

int32_t * brian::_array_spikegeneratorgroup__lastindex;
int32_t * brian::dev_array_spikegeneratorgroup__lastindex;
__device__ int32_t * brian::d_array_spikegeneratorgroup__lastindex;
const int brian::_num__array_spikegeneratorgroup__lastindex = 1;

int32_t * brian::_array_spikegeneratorgroup__period_bins;
int32_t * brian::dev_array_spikegeneratorgroup__period_bins;
__device__ int32_t * brian::d_array_spikegeneratorgroup__period_bins;
const int brian::_num__array_spikegeneratorgroup__period_bins = 1;

int32_t * brian::_array_spikegeneratorgroup_i;
int32_t * brian::dev_array_spikegeneratorgroup_i;
__device__ int32_t * brian::d_array_spikegeneratorgroup_i;
const int brian::_num__array_spikegeneratorgroup_i = 100;

double * brian::_array_spikegeneratorgroup_period;
double * brian::dev_array_spikegeneratorgroup_period;
__device__ double * brian::d_array_spikegeneratorgroup_period;
const int brian::_num__array_spikegeneratorgroup_period = 1;

int32_t * brian::_array_spikemonitor_1__source_idx;
int32_t * brian::dev_array_spikemonitor_1__source_idx;
__device__ int32_t * brian::d_array_spikemonitor_1__source_idx;
const int brian::_num__array_spikemonitor_1__source_idx = 2500;

int32_t * brian::_array_spikemonitor_1_count;
int32_t * brian::dev_array_spikemonitor_1_count;
__device__ int32_t * brian::d_array_spikemonitor_1_count;
const int brian::_num__array_spikemonitor_1_count = 2500;

int32_t * brian::_array_spikemonitor_1_N;
int32_t * brian::dev_array_spikemonitor_1_N;
__device__ int32_t * brian::d_array_spikemonitor_1_N;
const int brian::_num__array_spikemonitor_1_N = 1;

int32_t * brian::_array_spikemonitor_2__source_idx;
int32_t * brian::dev_array_spikemonitor_2__source_idx;
__device__ int32_t * brian::d_array_spikemonitor_2__source_idx;
const int brian::_num__array_spikemonitor_2__source_idx = 100;

int32_t * brian::_array_spikemonitor_2_count;
int32_t * brian::dev_array_spikemonitor_2_count;
__device__ int32_t * brian::d_array_spikemonitor_2_count;
const int brian::_num__array_spikemonitor_2_count = 100;

int32_t * brian::_array_spikemonitor_2_N;
int32_t * brian::dev_array_spikemonitor_2_N;
__device__ int32_t * brian::d_array_spikemonitor_2_N;
const int brian::_num__array_spikemonitor_2_N = 1;

int32_t * brian::_array_spikemonitor__source_idx;
int32_t * brian::dev_array_spikemonitor__source_idx;
__device__ int32_t * brian::d_array_spikemonitor__source_idx;
const int brian::_num__array_spikemonitor__source_idx = 100;

int32_t * brian::_array_spikemonitor_count;
int32_t * brian::dev_array_spikemonitor_count;
__device__ int32_t * brian::d_array_spikemonitor_count;
const int brian::_num__array_spikemonitor_count = 100;

int32_t * brian::_array_spikemonitor_N;
int32_t * brian::dev_array_spikemonitor_N;
__device__ int32_t * brian::d_array_spikemonitor_N;
const int brian::_num__array_spikemonitor_N = 1;

int32_t * brian::_array_synapses_1_N;
int32_t * brian::dev_array_synapses_1_N;
__device__ int32_t * brian::d_array_synapses_1_N;
const int brian::_num__array_synapses_1_N = 1;

int32_t * brian::_array_synapses_2_N;
int32_t * brian::dev_array_synapses_2_N;
__device__ int32_t * brian::d_array_synapses_2_N;
const int brian::_num__array_synapses_2_N = 1;

int32_t * brian::_array_synapses_N;
int32_t * brian::dev_array_synapses_N;
__device__ int32_t * brian::d_array_synapses_N;
const int brian::_num__array_synapses_N = 1;


//////////////// eventspaces ///////////////
// we dynamically create multiple eventspaces in no_or_const_delay_mode
// for initiating the first spikespace, we need a host pointer
// for choosing the right spikespace, we need a global index variable
int32_t * brian::_array_neurongroup_1__spikespace;
const int brian::_num__array_neurongroup_1__spikespace = 101;
thrust::host_vector<int32_t*> brian::dev_array_neurongroup_1__spikespace(1);
int brian::current_idx_array_neurongroup_1__spikespace = 0;
int32_t * brian::_array_neurongroup__spikespace;
const int brian::_num__array_neurongroup__spikespace = 2501;
thrust::host_vector<int32_t*> brian::dev_array_neurongroup__spikespace(1);
int brian::current_idx_array_neurongroup__spikespace = 0;
int32_t * brian::_array_spikegeneratorgroup__spikespace;
const int brian::_num__array_spikegeneratorgroup__spikespace = 101;
thrust::host_vector<int32_t*> brian::dev_array_spikegeneratorgroup__spikespace(1);
int brian::current_idx_array_spikegeneratorgroup__spikespace = 0;
int brian::previous_idx_array_spikegeneratorgroup__spikespace;

//////////////// dynamic arrays 1d /////////
thrust::host_vector<int32_t> brian::_dynamic_array_spikegeneratorgroup__timebins;
thrust::device_vector<int32_t> brian::dev_dynamic_array_spikegeneratorgroup__timebins;
thrust::host_vector<int32_t> brian::_dynamic_array_spikegeneratorgroup_neuron_index;
thrust::device_vector<int32_t> brian::dev_dynamic_array_spikegeneratorgroup_neuron_index;
thrust::host_vector<int32_t> brian::_dynamic_array_spikegeneratorgroup_spike_number;
thrust::device_vector<int32_t> brian::dev_dynamic_array_spikegeneratorgroup_spike_number;
thrust::host_vector<double> brian::_dynamic_array_spikegeneratorgroup_spike_time;
thrust::device_vector<double> brian::dev_dynamic_array_spikegeneratorgroup_spike_time;
thrust::host_vector<int32_t> brian::_dynamic_array_spikemonitor_1_i;
thrust::device_vector<int32_t> brian::dev_dynamic_array_spikemonitor_1_i;
thrust::host_vector<double> brian::_dynamic_array_spikemonitor_1_t;
thrust::device_vector<double> brian::dev_dynamic_array_spikemonitor_1_t;
thrust::host_vector<int32_t> brian::_dynamic_array_spikemonitor_2_i;
thrust::device_vector<int32_t> brian::dev_dynamic_array_spikemonitor_2_i;
thrust::host_vector<double> brian::_dynamic_array_spikemonitor_2_t;
thrust::device_vector<double> brian::dev_dynamic_array_spikemonitor_2_t;
thrust::host_vector<int32_t> brian::_dynamic_array_spikemonitor_i;
thrust::device_vector<int32_t> brian::dev_dynamic_array_spikemonitor_i;
thrust::host_vector<double> brian::_dynamic_array_spikemonitor_t;
thrust::device_vector<double> brian::dev_dynamic_array_spikemonitor_t;
thrust::host_vector<int32_t> brian::_dynamic_array_synapses_1__synaptic_post;
thrust::device_vector<int32_t> brian::dev_dynamic_array_synapses_1__synaptic_post;
thrust::host_vector<int32_t> brian::_dynamic_array_synapses_1__synaptic_pre;
thrust::device_vector<int32_t> brian::dev_dynamic_array_synapses_1__synaptic_pre;
thrust::host_vector<double> brian::_dynamic_array_synapses_1_Apost;
thrust::device_vector<double> brian::dev_dynamic_array_synapses_1_Apost;
thrust::host_vector<double> brian::_dynamic_array_synapses_1_Apre;
thrust::device_vector<double> brian::dev_dynamic_array_synapses_1_Apre;
thrust::host_vector<double> brian::_dynamic_array_synapses_1_delay;
thrust::device_vector<double> brian::dev_dynamic_array_synapses_1_delay;
thrust::host_vector<double> brian::_dynamic_array_synapses_1_delay_1;
thrust::device_vector<double> brian::dev_dynamic_array_synapses_1_delay_1;
thrust::host_vector<double> brian::_dynamic_array_synapses_1_g_raw;
thrust::device_vector<double> brian::dev_dynamic_array_synapses_1_g_raw;
thrust::host_vector<double> brian::_dynamic_array_synapses_1_lastupdate;
thrust::device_vector<double> brian::dev_dynamic_array_synapses_1_lastupdate;
thrust::host_vector<int32_t> brian::_dynamic_array_synapses_1_N_incoming;
thrust::device_vector<int32_t> brian::dev_dynamic_array_synapses_1_N_incoming;
thrust::host_vector<int32_t> brian::_dynamic_array_synapses_1_N_outgoing;
thrust::device_vector<int32_t> brian::dev_dynamic_array_synapses_1_N_outgoing;
thrust::host_vector<int32_t> brian::_dynamic_array_synapses_2__synaptic_post;
thrust::device_vector<int32_t> brian::dev_dynamic_array_synapses_2__synaptic_post;
thrust::host_vector<int32_t> brian::_dynamic_array_synapses_2__synaptic_pre;
thrust::device_vector<int32_t> brian::dev_dynamic_array_synapses_2__synaptic_pre;
thrust::host_vector<double> brian::_dynamic_array_synapses_2_delay;
thrust::device_vector<double> brian::dev_dynamic_array_synapses_2_delay;
thrust::host_vector<int32_t> brian::_dynamic_array_synapses_2_N_incoming;
thrust::device_vector<int32_t> brian::dev_dynamic_array_synapses_2_N_incoming;
thrust::host_vector<int32_t> brian::_dynamic_array_synapses_2_N_outgoing;
thrust::device_vector<int32_t> brian::dev_dynamic_array_synapses_2_N_outgoing;
thrust::host_vector<int32_t> brian::_dynamic_array_synapses__synaptic_post;
thrust::device_vector<int32_t> brian::dev_dynamic_array_synapses__synaptic_post;
thrust::host_vector<int32_t> brian::_dynamic_array_synapses__synaptic_pre;
thrust::device_vector<int32_t> brian::dev_dynamic_array_synapses__synaptic_pre;
thrust::host_vector<double> brian::_dynamic_array_synapses_delay;
thrust::device_vector<double> brian::dev_dynamic_array_synapses_delay;
thrust::host_vector<int32_t> brian::_dynamic_array_synapses_N_incoming;
thrust::device_vector<int32_t> brian::dev_dynamic_array_synapses_N_incoming;
thrust::host_vector<int32_t> brian::_dynamic_array_synapses_N_outgoing;
thrust::device_vector<int32_t> brian::dev_dynamic_array_synapses_N_outgoing;
thrust::host_vector<double> brian::_dynamic_array_synapses_weight;
thrust::device_vector<double> brian::dev_dynamic_array_synapses_weight;

//////////////// dynamic arrays 2d /////////

/////////////// static arrays /////////////
int32_t * brian::_static_array__dynamic_array_spikegeneratorgroup__timebins;
int32_t * brian::dev_static_array__dynamic_array_spikegeneratorgroup__timebins;
__device__ int32_t * brian::d_static_array__dynamic_array_spikegeneratorgroup__timebins;
const int brian::_num__static_array__dynamic_array_spikegeneratorgroup__timebins = 19676;
int64_t * brian::_static_array__dynamic_array_spikegeneratorgroup_neuron_index;
int64_t * brian::dev_static_array__dynamic_array_spikegeneratorgroup_neuron_index;
__device__ int64_t * brian::d_static_array__dynamic_array_spikegeneratorgroup_neuron_index;
const int brian::_num__static_array__dynamic_array_spikegeneratorgroup_neuron_index = 19676;
int64_t * brian::_static_array__dynamic_array_spikegeneratorgroup_spike_number;
int64_t * brian::dev_static_array__dynamic_array_spikegeneratorgroup_spike_number;
__device__ int64_t * brian::d_static_array__dynamic_array_spikegeneratorgroup_spike_number;
const int brian::_num__static_array__dynamic_array_spikegeneratorgroup_spike_number = 19676;
double * brian::_static_array__dynamic_array_spikegeneratorgroup_spike_time;
double * brian::dev_static_array__dynamic_array_spikegeneratorgroup_spike_time;
__device__ double * brian::d_static_array__dynamic_array_spikegeneratorgroup_spike_time;
const int brian::_num__static_array__dynamic_array_spikegeneratorgroup_spike_time = 19676;

//////////////// synapses /////////////////
// synapses
int32_t synapses_source_start_index;
int32_t synapses_source_stop_index;
bool brian::synapses_multiple_pre_post = false;
// synapses_pre
__device__ int* brian::synapses_pre_num_synapses_by_pre;
__device__ int* brian::synapses_pre_num_synapses_by_bundle;
__device__ int* brian::synapses_pre_unique_delays;
__device__ int* brian::synapses_pre_synapses_offset_by_bundle;
__device__ int* brian::synapses_pre_global_bundle_id_start_by_pre;
int brian::synapses_pre_max_bundle_size = 0;
int brian::synapses_pre_mean_bundle_size = 0;
int brian::synapses_pre_max_size = 0;
__device__ int* brian::synapses_pre_num_unique_delays_by_pre;
int brian::synapses_pre_max_num_unique_delays = 0;
__device__ int32_t** brian::synapses_pre_synapse_ids_by_pre;
__device__ int32_t* brian::synapses_pre_synapse_ids;
__device__ int* brian::synapses_pre_unique_delay_start_idcs;
__device__ int* brian::synapses_pre_unique_delays_offset_by_pre;
__device__ SynapticPathway brian::synapses_pre;
int brian::synapses_pre_eventspace_idx = 0;
int brian::synapses_pre_delay;
bool brian::synapses_pre_scalar_delay;
// synapses_1
int32_t synapses_1_source_start_index;
int32_t synapses_1_source_stop_index;
bool brian::synapses_1_multiple_pre_post = false;
// synapses_1_post
__device__ int* brian::synapses_1_post_num_synapses_by_pre;
__device__ int* brian::synapses_1_post_num_synapses_by_bundle;
__device__ int* brian::synapses_1_post_unique_delays;
__device__ int* brian::synapses_1_post_synapses_offset_by_bundle;
__device__ int* brian::synapses_1_post_global_bundle_id_start_by_pre;
int brian::synapses_1_post_max_bundle_size = 0;
int brian::synapses_1_post_mean_bundle_size = 0;
int brian::synapses_1_post_max_size = 0;
__device__ int* brian::synapses_1_post_num_unique_delays_by_pre;
int brian::synapses_1_post_max_num_unique_delays = 0;
__device__ int32_t** brian::synapses_1_post_synapse_ids_by_pre;
__device__ int32_t* brian::synapses_1_post_synapse_ids;
__device__ int* brian::synapses_1_post_unique_delay_start_idcs;
__device__ int* brian::synapses_1_post_unique_delays_offset_by_pre;
__device__ SynapticPathway brian::synapses_1_post;
int brian::synapses_1_post_eventspace_idx = 0;
int brian::synapses_1_post_delay;
bool brian::synapses_1_post_scalar_delay;
// synapses_1_pre
__device__ int* brian::synapses_1_pre_num_synapses_by_pre;
__device__ int* brian::synapses_1_pre_num_synapses_by_bundle;
__device__ int* brian::synapses_1_pre_unique_delays;
__device__ int* brian::synapses_1_pre_synapses_offset_by_bundle;
__device__ int* brian::synapses_1_pre_global_bundle_id_start_by_pre;
int brian::synapses_1_pre_max_bundle_size = 0;
int brian::synapses_1_pre_mean_bundle_size = 0;
int brian::synapses_1_pre_max_size = 0;
__device__ int* brian::synapses_1_pre_num_unique_delays_by_pre;
int brian::synapses_1_pre_max_num_unique_delays = 0;
__device__ int32_t** brian::synapses_1_pre_synapse_ids_by_pre;
__device__ int32_t* brian::synapses_1_pre_synapse_ids;
__device__ int* brian::synapses_1_pre_unique_delay_start_idcs;
__device__ int* brian::synapses_1_pre_unique_delays_offset_by_pre;
__device__ SynapticPathway brian::synapses_1_pre;
int brian::synapses_1_pre_eventspace_idx = 0;
int brian::synapses_1_pre_delay;
bool brian::synapses_1_pre_scalar_delay;
// synapses_2
int32_t synapses_2_source_start_index;
int32_t synapses_2_source_stop_index;
bool brian::synapses_2_multiple_pre_post = false;
// synapses_2_pre
__device__ int* brian::synapses_2_pre_num_synapses_by_pre;
__device__ int* brian::synapses_2_pre_num_synapses_by_bundle;
__device__ int* brian::synapses_2_pre_unique_delays;
__device__ int* brian::synapses_2_pre_synapses_offset_by_bundle;
__device__ int* brian::synapses_2_pre_global_bundle_id_start_by_pre;
int brian::synapses_2_pre_max_bundle_size = 0;
int brian::synapses_2_pre_mean_bundle_size = 0;
int brian::synapses_2_pre_max_size = 0;
__device__ int* brian::synapses_2_pre_num_unique_delays_by_pre;
int brian::synapses_2_pre_max_num_unique_delays = 0;
__device__ int32_t** brian::synapses_2_pre_synapse_ids_by_pre;
__device__ int32_t* brian::synapses_2_pre_synapse_ids;
__device__ int* brian::synapses_2_pre_unique_delay_start_idcs;
__device__ int* brian::synapses_2_pre_unique_delays_offset_by_pre;
__device__ SynapticPathway brian::synapses_2_pre;
int brian::synapses_2_pre_eventspace_idx = 0;
int brian::synapses_2_pre_delay;
bool brian::synapses_2_pre_scalar_delay;

int brian::num_parallel_blocks;
int brian::max_threads_per_block;
int brian::max_threads_per_sm;
int brian::max_shared_mem_size;
int brian::num_threads_per_warp;

__global__ void synapses_pre_init(
                int32_t* sources,
                int32_t* targets,
                double dt,
                int32_t source_start,
                int32_t source_stop
        )
{
    using namespace brian;

    synapses_pre.init(
            sources,
            targets,
            dt,
            // TODO: called source here, spikes in SynapticPathway (use same name)
            source_start,
            source_stop);
}
__global__ void synapses_1_post_init(
                int32_t* sources,
                int32_t* targets,
                double dt,
                int32_t source_start,
                int32_t source_stop
        )
{
    using namespace brian;

    synapses_1_post.init(
            sources,
            targets,
            dt,
            // TODO: called source here, spikes in SynapticPathway (use same name)
            source_start,
            source_stop);
}
__global__ void synapses_1_pre_init(
                int32_t* sources,
                int32_t* targets,
                double dt,
                int32_t source_start,
                int32_t source_stop
        )
{
    using namespace brian;

    synapses_1_pre.init(
            sources,
            targets,
            dt,
            // TODO: called source here, spikes in SynapticPathway (use same name)
            source_start,
            source_stop);
}
__global__ void synapses_2_pre_init(
                int32_t* sources,
                int32_t* targets,
                double dt,
                int32_t source_start,
                int32_t source_stop
        )
{
    using namespace brian;

    synapses_2_pre.init(
            sources,
            targets,
            dt,
            // TODO: called source here, spikes in SynapticPathway (use same name)
            source_start,
            source_stop);
}

// Profiling information for each code object

//////////////random numbers//////////////////
hiprandGenerator_t brian::curand_generator;
__device__ unsigned long long* brian::d_curand_seed;
unsigned long long* brian::dev_curand_seed;
// dev_{co.name}_{rng_type}_allocator
//      pointer to start of generated random numbers array
//      at each generation cycle this array is refilled
// dev_{co.name}_{rng_type}
//      pointer moving through generated random number array
//      until it is regenerated at the next generation cycle
hiprandState* brian::dev_curand_states;
hipStream_t brian::stream;
hipStream_t brian::stream1;
hipStream_t brian::stream2;
hipStream_t brian::neurongroup_stream1;
hipStream_t brian::neurongroup_stream;
hipStream_t brian::spikegenerator_stream;
hipStream_t brian::spikemonitor_stream1;
hipStream_t brian::spikegenerator_stream2;
hipStream_t brian::spikegenerator_stream;

__device__ hiprandState* brian::d_curand_states;
RandomNumberBuffer brian::random_number_buffer;

void _init_arrays()
{
    using namespace brian;

    std::clock_t start_timer = std::clock();

    CUDA_CHECK_MEMORY();
    size_t used_device_memory_start = used_device_memory;

    hipDeviceProp_t props;
    CUDA_SAFE_CALL(
            hipGetDeviceProperties(&props, 0)
            );

    num_parallel_blocks = props.multiProcessorCount * 1;
    printf("objects cu num par blocks %d\n", num_parallel_blocks);
    max_threads_per_block = props.maxThreadsPerBlock;
    max_threads_per_sm = props.maxThreadsPerMultiProcessor;
    max_shared_mem_size = props.sharedMemPerBlock;
    num_threads_per_warp = props.warpSize;

    // Random seeds might be overwritten in main.cu
    unsigned long long seed = time(0);

    CUDA_SAFE_CALL(
            hipMalloc((void**)&dev_curand_seed,
                sizeof(unsigned long long))
            );

    CUDA_SAFE_CALL(
            hipMemcpyToSymbol(HIP_SYMBOL(d_curand_seed), &dev_curand_seed,
                sizeof(unsigned long long*))
            );

    CUDA_SAFE_CALL(
            hiprandCreateGenerator(&curand_generator, HIPRAND_RNG_PSEUDO_DEFAULT)
            );


    // this sets seed for host and device api RNG
    random_number_buffer.set_seed(seed);
    // initialise neurongroups
    CUDA_SAFE_CALL(hipStreamCreate(&neurongroup_stream1));
    CUDA_SAFE_CALL(hipStreamCreate(&neurongroup_stream));
    
    //spike generator
    CUDA_SAFE_CALL(hipStreamCreate(&spikegenerator_stream));

    //spike monitor
    CUDA_SAFE_CALL(hipStreamCreate(&spikemonitor_stream1));
    CUDA_SAFE_CALL(hipStreamCreate(&spikemonitor_stream));
    CUDA_SAFE_CALL(hipStreamCreate(&spikemonitor_stream2));
    

    CUDA_SAFE_CALL(hipStreamCreate(&stream));

    synapses_pre_init<<<1,1>>>(
            thrust::raw_pointer_cast(&dev_dynamic_array_synapses__synaptic_pre[0]),
            thrust::raw_pointer_cast(&dev_dynamic_array_synapses__synaptic_post[0]),
            0,  //was dt, maybe irrelevant?
            0,
            100
            );
    CUDA_CHECK_ERROR("synapses_pre_init");
    synapses_1_post_init<<<1,1>>>(
            thrust::raw_pointer_cast(&dev_dynamic_array_synapses_1__synaptic_post[0]),
            thrust::raw_pointer_cast(&dev_dynamic_array_synapses_1__synaptic_pre[0]),
            0,  //was dt, maybe irrelevant?
            0,
            100
            );
    CUDA_CHECK_ERROR("synapses_1_post_init");
    CUDA_SAFE_CALL(hipStreamCreate(&stream1));
    synapses_1_pre_init<<<1,1>>>(
            thrust::raw_pointer_cast(&dev_dynamic_array_synapses_1__synaptic_pre[0]),
            thrust::raw_pointer_cast(&dev_dynamic_array_synapses_1__synaptic_post[0]),
            0,  //was dt, maybe irrelevant?
            0,
            2500
            );
    CUDA_CHECK_ERROR("synapses_1_pre_init");
    CUDA_SAFE_CALL(hipStreamCreate(&stream2));
    synapses_2_pre_init<<<1,1>>>(
            thrust::raw_pointer_cast(&dev_dynamic_array_synapses_2__synaptic_pre[0]),
            thrust::raw_pointer_cast(&dev_dynamic_array_synapses_2__synaptic_post[0]),
            0,  //was dt, maybe irrelevant?
            0,
            100
            );
    CUDA_CHECK_ERROR("synapses_2_pre_init");

    // Arrays initialized to 0
            _array_defaultclock_dt = new double[1];
            for(int i=0; i<1; i++) _array_defaultclock_dt[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_defaultclock_dt, sizeof(double)*_num__array_defaultclock_dt)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_defaultclock_dt, _array_defaultclock_dt, sizeof(double)*_num__array_defaultclock_dt, hipMemcpyHostToDevice)
                    );
            _array_defaultclock_t = new double[1];
            for(int i=0; i<1; i++) _array_defaultclock_t[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_defaultclock_t, sizeof(double)*_num__array_defaultclock_t)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_defaultclock_t, _array_defaultclock_t, sizeof(double)*_num__array_defaultclock_t, hipMemcpyHostToDevice)
                    );
            _array_defaultclock_timestep = new int64_t[1];
            for(int i=0; i<1; i++) _array_defaultclock_timestep[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_defaultclock_timestep, sizeof(int64_t)*_num__array_defaultclock_timestep)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_defaultclock_timestep, _array_defaultclock_timestep, sizeof(int64_t)*_num__array_defaultclock_timestep, hipMemcpyHostToDevice)
                    );
            _array_neurongroup_1_g_eKC_eKC = new double[100];
            for(int i=0; i<100; i++) _array_neurongroup_1_g_eKC_eKC[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_neurongroup_1_g_eKC_eKC, sizeof(double)*_num__array_neurongroup_1_g_eKC_eKC)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_neurongroup_1_g_eKC_eKC, _array_neurongroup_1_g_eKC_eKC, sizeof(double)*_num__array_neurongroup_1_g_eKC_eKC, hipMemcpyHostToDevice)
                    );
            _array_neurongroup_1_g_iKC_eKC = new double[100];
            for(int i=0; i<100; i++) _array_neurongroup_1_g_iKC_eKC[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_neurongroup_1_g_iKC_eKC, sizeof(double)*_num__array_neurongroup_1_g_iKC_eKC)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_neurongroup_1_g_iKC_eKC, _array_neurongroup_1_g_iKC_eKC, sizeof(double)*_num__array_neurongroup_1_g_iKC_eKC, hipMemcpyHostToDevice)
                    );
            _array_neurongroup_1_h = new double[100];
            for(int i=0; i<100; i++) _array_neurongroup_1_h[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_neurongroup_1_h, sizeof(double)*_num__array_neurongroup_1_h)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_neurongroup_1_h, _array_neurongroup_1_h, sizeof(double)*_num__array_neurongroup_1_h, hipMemcpyHostToDevice)
                    );
            _array_neurongroup_1_i = new int32_t[100];
            for(int i=0; i<100; i++) _array_neurongroup_1_i[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_neurongroup_1_i, sizeof(int32_t)*_num__array_neurongroup_1_i)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_neurongroup_1_i, _array_neurongroup_1_i, sizeof(int32_t)*_num__array_neurongroup_1_i, hipMemcpyHostToDevice)
                    );
            _array_neurongroup_1_lastspike = new double[100];
            for(int i=0; i<100; i++) _array_neurongroup_1_lastspike[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_neurongroup_1_lastspike, sizeof(double)*_num__array_neurongroup_1_lastspike)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_neurongroup_1_lastspike, _array_neurongroup_1_lastspike, sizeof(double)*_num__array_neurongroup_1_lastspike, hipMemcpyHostToDevice)
                    );
            _array_neurongroup_1_m = new double[100];
            for(int i=0; i<100; i++) _array_neurongroup_1_m[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_neurongroup_1_m, sizeof(double)*_num__array_neurongroup_1_m)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_neurongroup_1_m, _array_neurongroup_1_m, sizeof(double)*_num__array_neurongroup_1_m, hipMemcpyHostToDevice)
                    );
            _array_neurongroup_1_n = new double[100];
            for(int i=0; i<100; i++) _array_neurongroup_1_n[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_neurongroup_1_n, sizeof(double)*_num__array_neurongroup_1_n)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_neurongroup_1_n, _array_neurongroup_1_n, sizeof(double)*_num__array_neurongroup_1_n, hipMemcpyHostToDevice)
                    );
            _array_neurongroup_1_not_refractory = new char[100];
            for(int i=0; i<100; i++) _array_neurongroup_1_not_refractory[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_neurongroup_1_not_refractory, sizeof(char)*_num__array_neurongroup_1_not_refractory)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_neurongroup_1_not_refractory, _array_neurongroup_1_not_refractory, sizeof(char)*_num__array_neurongroup_1_not_refractory, hipMemcpyHostToDevice)
                    );
            _array_neurongroup_1_V = new double[100];
            for(int i=0; i<100; i++) _array_neurongroup_1_V[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_neurongroup_1_V, sizeof(double)*_num__array_neurongroup_1_V)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_neurongroup_1_V, _array_neurongroup_1_V, sizeof(double)*_num__array_neurongroup_1_V, hipMemcpyHostToDevice)
                    );
            _array_neurongroup_g_PN_iKC = new double[2500];
            for(int i=0; i<2500; i++) _array_neurongroup_g_PN_iKC[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_neurongroup_g_PN_iKC, sizeof(double)*_num__array_neurongroup_g_PN_iKC)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_neurongroup_g_PN_iKC, _array_neurongroup_g_PN_iKC, sizeof(double)*_num__array_neurongroup_g_PN_iKC, hipMemcpyHostToDevice)
                    );
            _array_neurongroup_h = new double[2500];
            for(int i=0; i<2500; i++) _array_neurongroup_h[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_neurongroup_h, sizeof(double)*_num__array_neurongroup_h)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_neurongroup_h, _array_neurongroup_h, sizeof(double)*_num__array_neurongroup_h, hipMemcpyHostToDevice)
                    );
            _array_neurongroup_i = new int32_t[2500];
            for(int i=0; i<2500; i++) _array_neurongroup_i[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_neurongroup_i, sizeof(int32_t)*_num__array_neurongroup_i)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_neurongroup_i, _array_neurongroup_i, sizeof(int32_t)*_num__array_neurongroup_i, hipMemcpyHostToDevice)
                    );
            _array_neurongroup_lastspike = new double[2500];
            for(int i=0; i<2500; i++) _array_neurongroup_lastspike[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_neurongroup_lastspike, sizeof(double)*_num__array_neurongroup_lastspike)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_neurongroup_lastspike, _array_neurongroup_lastspike, sizeof(double)*_num__array_neurongroup_lastspike, hipMemcpyHostToDevice)
                    );
            _array_neurongroup_m = new double[2500];
            for(int i=0; i<2500; i++) _array_neurongroup_m[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_neurongroup_m, sizeof(double)*_num__array_neurongroup_m)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_neurongroup_m, _array_neurongroup_m, sizeof(double)*_num__array_neurongroup_m, hipMemcpyHostToDevice)
                    );
            _array_neurongroup_n = new double[2500];
            for(int i=0; i<2500; i++) _array_neurongroup_n[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_neurongroup_n, sizeof(double)*_num__array_neurongroup_n)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_neurongroup_n, _array_neurongroup_n, sizeof(double)*_num__array_neurongroup_n, hipMemcpyHostToDevice)
                    );
            _array_neurongroup_not_refractory = new char[2500];
            for(int i=0; i<2500; i++) _array_neurongroup_not_refractory[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_neurongroup_not_refractory, sizeof(char)*_num__array_neurongroup_not_refractory)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_neurongroup_not_refractory, _array_neurongroup_not_refractory, sizeof(char)*_num__array_neurongroup_not_refractory, hipMemcpyHostToDevice)
                    );
            _array_neurongroup_V = new double[2500];
            for(int i=0; i<2500; i++) _array_neurongroup_V[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_neurongroup_V, sizeof(double)*_num__array_neurongroup_V)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_neurongroup_V, _array_neurongroup_V, sizeof(double)*_num__array_neurongroup_V, hipMemcpyHostToDevice)
                    );
            _array_spikegeneratorgroup__lastindex = new int32_t[1];
            for(int i=0; i<1; i++) _array_spikegeneratorgroup__lastindex[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_spikegeneratorgroup__lastindex, sizeof(int32_t)*_num__array_spikegeneratorgroup__lastindex)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_spikegeneratorgroup__lastindex, _array_spikegeneratorgroup__lastindex, sizeof(int32_t)*_num__array_spikegeneratorgroup__lastindex, hipMemcpyHostToDevice)
                    );
            _array_spikegeneratorgroup__period_bins = new int32_t[1];
            for(int i=0; i<1; i++) _array_spikegeneratorgroup__period_bins[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_spikegeneratorgroup__period_bins, sizeof(int32_t)*_num__array_spikegeneratorgroup__period_bins)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_spikegeneratorgroup__period_bins, _array_spikegeneratorgroup__period_bins, sizeof(int32_t)*_num__array_spikegeneratorgroup__period_bins, hipMemcpyHostToDevice)
                    );
            _array_spikegeneratorgroup_i = new int32_t[100];
            for(int i=0; i<100; i++) _array_spikegeneratorgroup_i[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_spikegeneratorgroup_i, sizeof(int32_t)*_num__array_spikegeneratorgroup_i)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_spikegeneratorgroup_i, _array_spikegeneratorgroup_i, sizeof(int32_t)*_num__array_spikegeneratorgroup_i, hipMemcpyHostToDevice)
                    );
            _array_spikegeneratorgroup_period = new double[1];
            for(int i=0; i<1; i++) _array_spikegeneratorgroup_period[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_spikegeneratorgroup_period, sizeof(double)*_num__array_spikegeneratorgroup_period)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_spikegeneratorgroup_period, _array_spikegeneratorgroup_period, sizeof(double)*_num__array_spikegeneratorgroup_period, hipMemcpyHostToDevice)
                    );
            _array_spikemonitor_1__source_idx = new int32_t[2500];
            for(int i=0; i<2500; i++) _array_spikemonitor_1__source_idx[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_spikemonitor_1__source_idx, sizeof(int32_t)*_num__array_spikemonitor_1__source_idx)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_spikemonitor_1__source_idx, _array_spikemonitor_1__source_idx, sizeof(int32_t)*_num__array_spikemonitor_1__source_idx, hipMemcpyHostToDevice)
                    );
            _array_spikemonitor_1_count = new int32_t[2500];
            for(int i=0; i<2500; i++) _array_spikemonitor_1_count[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_spikemonitor_1_count, sizeof(int32_t)*_num__array_spikemonitor_1_count)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_spikemonitor_1_count, _array_spikemonitor_1_count, sizeof(int32_t)*_num__array_spikemonitor_1_count, hipMemcpyHostToDevice)
                    );
            _array_spikemonitor_1_N = new int32_t[1];
            for(int i=0; i<1; i++) _array_spikemonitor_1_N[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_spikemonitor_1_N, sizeof(int32_t)*_num__array_spikemonitor_1_N)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_spikemonitor_1_N, _array_spikemonitor_1_N, sizeof(int32_t)*_num__array_spikemonitor_1_N, hipMemcpyHostToDevice)
                    );
            _array_spikemonitor_2__source_idx = new int32_t[100];
            for(int i=0; i<100; i++) _array_spikemonitor_2__source_idx[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_spikemonitor_2__source_idx, sizeof(int32_t)*_num__array_spikemonitor_2__source_idx)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_spikemonitor_2__source_idx, _array_spikemonitor_2__source_idx, sizeof(int32_t)*_num__array_spikemonitor_2__source_idx, hipMemcpyHostToDevice)
                    );
            _array_spikemonitor_2_count = new int32_t[100];
            for(int i=0; i<100; i++) _array_spikemonitor_2_count[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_spikemonitor_2_count, sizeof(int32_t)*_num__array_spikemonitor_2_count)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_spikemonitor_2_count, _array_spikemonitor_2_count, sizeof(int32_t)*_num__array_spikemonitor_2_count, hipMemcpyHostToDevice)
                    );
            _array_spikemonitor_2_N = new int32_t[1];
            for(int i=0; i<1; i++) _array_spikemonitor_2_N[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_spikemonitor_2_N, sizeof(int32_t)*_num__array_spikemonitor_2_N)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_spikemonitor_2_N, _array_spikemonitor_2_N, sizeof(int32_t)*_num__array_spikemonitor_2_N, hipMemcpyHostToDevice)
                    );
            _array_spikemonitor__source_idx = new int32_t[100];
            for(int i=0; i<100; i++) _array_spikemonitor__source_idx[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_spikemonitor__source_idx, sizeof(int32_t)*_num__array_spikemonitor__source_idx)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_spikemonitor__source_idx, _array_spikemonitor__source_idx, sizeof(int32_t)*_num__array_spikemonitor__source_idx, hipMemcpyHostToDevice)
                    );
            _array_spikemonitor_count = new int32_t[100];
            for(int i=0; i<100; i++) _array_spikemonitor_count[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_spikemonitor_count, sizeof(int32_t)*_num__array_spikemonitor_count)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_spikemonitor_count, _array_spikemonitor_count, sizeof(int32_t)*_num__array_spikemonitor_count, hipMemcpyHostToDevice)
                    );
            _array_spikemonitor_N = new int32_t[1];
            for(int i=0; i<1; i++) _array_spikemonitor_N[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_spikemonitor_N, sizeof(int32_t)*_num__array_spikemonitor_N)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_spikemonitor_N, _array_spikemonitor_N, sizeof(int32_t)*_num__array_spikemonitor_N, hipMemcpyHostToDevice)
                    );
            _array_synapses_1_N = new int32_t[1];
            for(int i=0; i<1; i++) _array_synapses_1_N[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_synapses_1_N, sizeof(int32_t)*_num__array_synapses_1_N)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_synapses_1_N, _array_synapses_1_N, sizeof(int32_t)*_num__array_synapses_1_N, hipMemcpyHostToDevice)
                    );
            _array_synapses_2_N = new int32_t[1];
            for(int i=0; i<1; i++) _array_synapses_2_N[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_synapses_2_N, sizeof(int32_t)*_num__array_synapses_2_N)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_synapses_2_N, _array_synapses_2_N, sizeof(int32_t)*_num__array_synapses_2_N, hipMemcpyHostToDevice)
                    );
            _array_synapses_N = new int32_t[1];
            for(int i=0; i<1; i++) _array_synapses_N[i] = 0;
            CUDA_SAFE_CALL(
                    hipMalloc((void**)&dev_array_synapses_N, sizeof(int32_t)*_num__array_synapses_N)
                    );
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_array_synapses_N, _array_synapses_N, sizeof(int32_t)*_num__array_synapses_N, hipMemcpyHostToDevice)
                    );
            _dynamic_array_spikegeneratorgroup__timebins.resize(19676);
            THRUST_CHECK_ERROR(dev_dynamic_array_spikegeneratorgroup__timebins.resize(19676));
            for(int i=0; i<19676; i++)
            {
                _dynamic_array_spikegeneratorgroup__timebins[i] = 0;
                dev_dynamic_array_spikegeneratorgroup__timebins[i] = 0;
            }
            _dynamic_array_synapses_1_delay.resize(1);
            THRUST_CHECK_ERROR(dev_dynamic_array_synapses_1_delay.resize(1));
            for(int i=0; i<1; i++)
            {
                _dynamic_array_synapses_1_delay[i] = 0;
                dev_dynamic_array_synapses_1_delay[i] = 0;
            }
            _dynamic_array_synapses_2_delay.resize(1);
            THRUST_CHECK_ERROR(dev_dynamic_array_synapses_2_delay.resize(1));
            for(int i=0; i<1; i++)
            {
                _dynamic_array_synapses_2_delay[i] = 0;
                dev_dynamic_array_synapses_2_delay[i] = 0;
            }

    // Arrays initialized to an "arange"
    _array_neurongroup_1_i = new int32_t[100];
    for(int i=0; i<100; i++) _array_neurongroup_1_i[i] = 0 + i;
    CUDA_SAFE_CALL(
            hipMalloc((void**)&dev_array_neurongroup_1_i, sizeof(int32_t)*_num__array_neurongroup_1_i)
            );

    CUDA_SAFE_CALL(
            hipMemcpy(dev_array_neurongroup_1_i, _array_neurongroup_1_i, sizeof(int32_t)*_num__array_neurongroup_1_i, hipMemcpyHostToDevice)
            );
    _array_neurongroup_i = new int32_t[2500];
    for(int i=0; i<2500; i++) _array_neurongroup_i[i] = 0 + i;
    CUDA_SAFE_CALL(
            hipMalloc((void**)&dev_array_neurongroup_i, sizeof(int32_t)*_num__array_neurongroup_i)
            );

    CUDA_SAFE_CALL(
            hipMemcpy(dev_array_neurongroup_i, _array_neurongroup_i, sizeof(int32_t)*_num__array_neurongroup_i, hipMemcpyHostToDevice)
            );
    _array_spikegeneratorgroup_i = new int32_t[100];
    for(int i=0; i<100; i++) _array_spikegeneratorgroup_i[i] = 0 + i;
    CUDA_SAFE_CALL(
            hipMalloc((void**)&dev_array_spikegeneratorgroup_i, sizeof(int32_t)*_num__array_spikegeneratorgroup_i)
            );

    CUDA_SAFE_CALL(
            hipMemcpy(dev_array_spikegeneratorgroup_i, _array_spikegeneratorgroup_i, sizeof(int32_t)*_num__array_spikegeneratorgroup_i, hipMemcpyHostToDevice)
            );
    _array_spikemonitor_1__source_idx = new int32_t[2500];
    for(int i=0; i<2500; i++) _array_spikemonitor_1__source_idx[i] = 0 + i;
    CUDA_SAFE_CALL(
            hipMalloc((void**)&dev_array_spikemonitor_1__source_idx, sizeof(int32_t)*_num__array_spikemonitor_1__source_idx)
            );

    CUDA_SAFE_CALL(
            hipMemcpy(dev_array_spikemonitor_1__source_idx, _array_spikemonitor_1__source_idx, sizeof(int32_t)*_num__array_spikemonitor_1__source_idx, hipMemcpyHostToDevice)
            );
    _array_spikemonitor_2__source_idx = new int32_t[100];
    for(int i=0; i<100; i++) _array_spikemonitor_2__source_idx[i] = 0 + i;
    CUDA_SAFE_CALL(
            hipMalloc((void**)&dev_array_spikemonitor_2__source_idx, sizeof(int32_t)*_num__array_spikemonitor_2__source_idx)
            );

    CUDA_SAFE_CALL(
            hipMemcpy(dev_array_spikemonitor_2__source_idx, _array_spikemonitor_2__source_idx, sizeof(int32_t)*_num__array_spikemonitor_2__source_idx, hipMemcpyHostToDevice)
            );
    _array_spikemonitor__source_idx = new int32_t[100];
    for(int i=0; i<100; i++) _array_spikemonitor__source_idx[i] = 0 + i;
    CUDA_SAFE_CALL(
            hipMalloc((void**)&dev_array_spikemonitor__source_idx, sizeof(int32_t)*_num__array_spikemonitor__source_idx)
            );

    CUDA_SAFE_CALL(
            hipMemcpy(dev_array_spikemonitor__source_idx, _array_spikemonitor__source_idx, sizeof(int32_t)*_num__array_spikemonitor__source_idx, hipMemcpyHostToDevice)
            );

    // static arrays
    _static_array__dynamic_array_spikegeneratorgroup__timebins = new int32_t[19676];
    CUDA_SAFE_CALL(
            hipMalloc((void**)&dev_static_array__dynamic_array_spikegeneratorgroup__timebins, sizeof(int32_t)*19676)
            );
    CUDA_SAFE_CALL(
            hipMemcpyToSymbol(HIP_SYMBOL(d_static_array__dynamic_array_spikegeneratorgroup__timebins), &dev_static_array__dynamic_array_spikegeneratorgroup__timebins, sizeof(int32_t*))
            );
    _static_array__dynamic_array_spikegeneratorgroup_neuron_index = new int64_t[19676];
    CUDA_SAFE_CALL(
            hipMalloc((void**)&dev_static_array__dynamic_array_spikegeneratorgroup_neuron_index, sizeof(int64_t)*19676)
            );
    CUDA_SAFE_CALL(
            hipMemcpyToSymbol(HIP_SYMBOL(d_static_array__dynamic_array_spikegeneratorgroup_neuron_index), &dev_static_array__dynamic_array_spikegeneratorgroup_neuron_index, sizeof(int64_t*))
            );
    _static_array__dynamic_array_spikegeneratorgroup_spike_number = new int64_t[19676];
    CUDA_SAFE_CALL(
            hipMalloc((void**)&dev_static_array__dynamic_array_spikegeneratorgroup_spike_number, sizeof(int64_t)*19676)
            );
    CUDA_SAFE_CALL(
            hipMemcpyToSymbol(HIP_SYMBOL(d_static_array__dynamic_array_spikegeneratorgroup_spike_number), &dev_static_array__dynamic_array_spikegeneratorgroup_spike_number, sizeof(int64_t*))
            );
    _static_array__dynamic_array_spikegeneratorgroup_spike_time = new double[19676];
    CUDA_SAFE_CALL(
            hipMalloc((void**)&dev_static_array__dynamic_array_spikegeneratorgroup_spike_time, sizeof(double)*19676)
            );
    CUDA_SAFE_CALL(
            hipMemcpyToSymbol(HIP_SYMBOL(d_static_array__dynamic_array_spikegeneratorgroup_spike_time), &dev_static_array__dynamic_array_spikegeneratorgroup_spike_time, sizeof(double*))
            );


    // eventspace_arrays
    CUDA_SAFE_CALL(
            hipMalloc((void**)&dev_array_neurongroup_1__spikespace[0], sizeof(int32_t)*_num__array_neurongroup_1__spikespace)
            );
    _array_neurongroup_1__spikespace = new int32_t[101];
    CUDA_SAFE_CALL(
            hipMalloc((void**)&dev_array_neurongroup__spikespace[0], sizeof(int32_t)*_num__array_neurongroup__spikespace)
            );
    _array_neurongroup__spikespace = new int32_t[2501];
    CUDA_SAFE_CALL(
            hipMalloc((void**)&dev_array_spikegeneratorgroup__spikespace[0], sizeof(int32_t)*_num__array_spikegeneratorgroup__spikespace)
            );
    _array_spikegeneratorgroup__spikespace = new int32_t[101];

    CUDA_CHECK_MEMORY();
    const double to_MB = 1.0 / (1024.0 * 1024.0);
    double tot_memory_MB = (used_device_memory - used_device_memory_start) * to_MB;
    double time_passed = (double)(std::clock() - start_timer) / CLOCKS_PER_SEC;
    std::cout << "INFO: _init_arrays() took " <<  time_passed << "s";
    if (tot_memory_MB > 0)
        std::cout << " and used " << tot_memory_MB << "MB of device memory.";
    std::cout << std::endl;
}

void _load_arrays()
{
    using namespace brian;

    ifstream f_static_array__dynamic_array_spikegeneratorgroup__timebins;
    f_static_array__dynamic_array_spikegeneratorgroup__timebins.open("static_arrays/_static_array__dynamic_array_spikegeneratorgroup__timebins", ios::in | ios::binary);
    if(f_static_array__dynamic_array_spikegeneratorgroup__timebins.is_open())
    {
        f_static_array__dynamic_array_spikegeneratorgroup__timebins.read(reinterpret_cast<char*>(_static_array__dynamic_array_spikegeneratorgroup__timebins), 19676*sizeof(int32_t));
    } else
    {
        std::cout << "Error opening static array _static_array__dynamic_array_spikegeneratorgroup__timebins." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(dev_static_array__dynamic_array_spikegeneratorgroup__timebins, _static_array__dynamic_array_spikegeneratorgroup__timebins, sizeof(int32_t)*19676, hipMemcpyHostToDevice)
            );
    ifstream f_static_array__dynamic_array_spikegeneratorgroup_neuron_index;
    f_static_array__dynamic_array_spikegeneratorgroup_neuron_index.open("static_arrays/_static_array__dynamic_array_spikegeneratorgroup_neuron_index", ios::in | ios::binary);
    if(f_static_array__dynamic_array_spikegeneratorgroup_neuron_index.is_open())
    {
        f_static_array__dynamic_array_spikegeneratorgroup_neuron_index.read(reinterpret_cast<char*>(_static_array__dynamic_array_spikegeneratorgroup_neuron_index), 19676*sizeof(int64_t));
    } else
    {
        std::cout << "Error opening static array _static_array__dynamic_array_spikegeneratorgroup_neuron_index." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(dev_static_array__dynamic_array_spikegeneratorgroup_neuron_index, _static_array__dynamic_array_spikegeneratorgroup_neuron_index, sizeof(int64_t)*19676, hipMemcpyHostToDevice)
            );
    ifstream f_static_array__dynamic_array_spikegeneratorgroup_spike_number;
    f_static_array__dynamic_array_spikegeneratorgroup_spike_number.open("static_arrays/_static_array__dynamic_array_spikegeneratorgroup_spike_number", ios::in | ios::binary);
    if(f_static_array__dynamic_array_spikegeneratorgroup_spike_number.is_open())
    {
        f_static_array__dynamic_array_spikegeneratorgroup_spike_number.read(reinterpret_cast<char*>(_static_array__dynamic_array_spikegeneratorgroup_spike_number), 19676*sizeof(int64_t));
    } else
    {
        std::cout << "Error opening static array _static_array__dynamic_array_spikegeneratorgroup_spike_number." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(dev_static_array__dynamic_array_spikegeneratorgroup_spike_number, _static_array__dynamic_array_spikegeneratorgroup_spike_number, sizeof(int64_t)*19676, hipMemcpyHostToDevice)
            );
    ifstream f_static_array__dynamic_array_spikegeneratorgroup_spike_time;
    f_static_array__dynamic_array_spikegeneratorgroup_spike_time.open("static_arrays/_static_array__dynamic_array_spikegeneratorgroup_spike_time", ios::in | ios::binary);
    if(f_static_array__dynamic_array_spikegeneratorgroup_spike_time.is_open())
    {
        f_static_array__dynamic_array_spikegeneratorgroup_spike_time.read(reinterpret_cast<char*>(_static_array__dynamic_array_spikegeneratorgroup_spike_time), 19676*sizeof(double));
    } else
    {
        std::cout << "Error opening static array _static_array__dynamic_array_spikegeneratorgroup_spike_time." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(dev_static_array__dynamic_array_spikegeneratorgroup_spike_time, _static_array__dynamic_array_spikegeneratorgroup_spike_time, sizeof(double)*19676, hipMemcpyHostToDevice)
            );
}

void _write_arrays()
{
    using namespace brian;

    CUDA_SAFE_CALL(
            hipMemcpy(_array_defaultclock_dt, dev_array_defaultclock_dt, sizeof(double)*_num__array_defaultclock_dt, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_defaultclock_dt;
    outfile__array_defaultclock_dt.open("results/_array_defaultclock_dt_-847410599827917468", ios::binary | ios::out);
    if(outfile__array_defaultclock_dt.is_open())
    {
        outfile__array_defaultclock_dt.write(reinterpret_cast<char*>(_array_defaultclock_dt), 1*sizeof(double));
        outfile__array_defaultclock_dt.close();
    } else
    {
        std::cout << "Error writing output file for _array_defaultclock_dt." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_defaultclock_t, dev_array_defaultclock_t, sizeof(double)*_num__array_defaultclock_t, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_defaultclock_t;
    outfile__array_defaultclock_t.open("results/_array_defaultclock_t_8322660633589888012", ios::binary | ios::out);
    if(outfile__array_defaultclock_t.is_open())
    {
        outfile__array_defaultclock_t.write(reinterpret_cast<char*>(_array_defaultclock_t), 1*sizeof(double));
        outfile__array_defaultclock_t.close();
    } else
    {
        std::cout << "Error writing output file for _array_defaultclock_t." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_defaultclock_timestep, dev_array_defaultclock_timestep, sizeof(int64_t)*_num__array_defaultclock_timestep, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_defaultclock_timestep;
    outfile__array_defaultclock_timestep.open("results/_array_defaultclock_timestep_1352370266667125095", ios::binary | ios::out);
    if(outfile__array_defaultclock_timestep.is_open())
    {
        outfile__array_defaultclock_timestep.write(reinterpret_cast<char*>(_array_defaultclock_timestep), 1*sizeof(int64_t));
        outfile__array_defaultclock_timestep.close();
    } else
    {
        std::cout << "Error writing output file for _array_defaultclock_timestep." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_neurongroup_1_g_eKC_eKC, dev_array_neurongroup_1_g_eKC_eKC, sizeof(double)*_num__array_neurongroup_1_g_eKC_eKC, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_neurongroup_1_g_eKC_eKC;
    outfile__array_neurongroup_1_g_eKC_eKC.open("results/_array_neurongroup_1_g_eKC_eKC_-2719670425652398549", ios::binary | ios::out);
    if(outfile__array_neurongroup_1_g_eKC_eKC.is_open())
    {
        outfile__array_neurongroup_1_g_eKC_eKC.write(reinterpret_cast<char*>(_array_neurongroup_1_g_eKC_eKC), 100*sizeof(double));
        outfile__array_neurongroup_1_g_eKC_eKC.close();
    } else
    {
        std::cout << "Error writing output file for _array_neurongroup_1_g_eKC_eKC." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_neurongroup_1_g_iKC_eKC, dev_array_neurongroup_1_g_iKC_eKC, sizeof(double)*_num__array_neurongroup_1_g_iKC_eKC, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_neurongroup_1_g_iKC_eKC;
    outfile__array_neurongroup_1_g_iKC_eKC.open("results/_array_neurongroup_1_g_iKC_eKC_-6839007311668324058", ios::binary | ios::out);
    if(outfile__array_neurongroup_1_g_iKC_eKC.is_open())
    {
        outfile__array_neurongroup_1_g_iKC_eKC.write(reinterpret_cast<char*>(_array_neurongroup_1_g_iKC_eKC), 100*sizeof(double));
        outfile__array_neurongroup_1_g_iKC_eKC.close();
    } else
    {
        std::cout << "Error writing output file for _array_neurongroup_1_g_iKC_eKC." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_neurongroup_1_h, dev_array_neurongroup_1_h, sizeof(double)*_num__array_neurongroup_1_h, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_neurongroup_1_h;
    outfile__array_neurongroup_1_h.open("results/_array_neurongroup_1_h_1075921236281676937", ios::binary | ios::out);
    if(outfile__array_neurongroup_1_h.is_open())
    {
        outfile__array_neurongroup_1_h.write(reinterpret_cast<char*>(_array_neurongroup_1_h), 100*sizeof(double));
        outfile__array_neurongroup_1_h.close();
    } else
    {
        std::cout << "Error writing output file for _array_neurongroup_1_h." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_neurongroup_1_i, dev_array_neurongroup_1_i, sizeof(int32_t)*_num__array_neurongroup_1_i, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_neurongroup_1_i;
    outfile__array_neurongroup_1_i.open("results/_array_neurongroup_1_i_8994940115406199838", ios::binary | ios::out);
    if(outfile__array_neurongroup_1_i.is_open())
    {
        outfile__array_neurongroup_1_i.write(reinterpret_cast<char*>(_array_neurongroup_1_i), 100*sizeof(int32_t));
        outfile__array_neurongroup_1_i.close();
    } else
    {
        std::cout << "Error writing output file for _array_neurongroup_1_i." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_neurongroup_1_lastspike, dev_array_neurongroup_1_lastspike, sizeof(double)*_num__array_neurongroup_1_lastspike, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_neurongroup_1_lastspike;
    outfile__array_neurongroup_1_lastspike.open("results/_array_neurongroup_1_lastspike_-8689292283566925331", ios::binary | ios::out);
    if(outfile__array_neurongroup_1_lastspike.is_open())
    {
        outfile__array_neurongroup_1_lastspike.write(reinterpret_cast<char*>(_array_neurongroup_1_lastspike), 100*sizeof(double));
        outfile__array_neurongroup_1_lastspike.close();
    } else
    {
        std::cout << "Error writing output file for _array_neurongroup_1_lastspike." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_neurongroup_1_m, dev_array_neurongroup_1_m, sizeof(double)*_num__array_neurongroup_1_m, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_neurongroup_1_m;
    outfile__array_neurongroup_1_m.open("results/_array_neurongroup_1_m_7921550157009594959", ios::binary | ios::out);
    if(outfile__array_neurongroup_1_m.is_open())
    {
        outfile__array_neurongroup_1_m.write(reinterpret_cast<char*>(_array_neurongroup_1_m), 100*sizeof(double));
        outfile__array_neurongroup_1_m.close();
    } else
    {
        std::cout << "Error writing output file for _array_neurongroup_1_m." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_neurongroup_1_n, dev_array_neurongroup_1_n, sizeof(double)*_num__array_neurongroup_1_n, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_neurongroup_1_n;
    outfile__array_neurongroup_1_n.open("results/_array_neurongroup_1_n_-5628489820633515426", ios::binary | ios::out);
    if(outfile__array_neurongroup_1_n.is_open())
    {
        outfile__array_neurongroup_1_n.write(reinterpret_cast<char*>(_array_neurongroup_1_n), 100*sizeof(double));
        outfile__array_neurongroup_1_n.close();
    } else
    {
        std::cout << "Error writing output file for _array_neurongroup_1_n." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_neurongroup_1_not_refractory, dev_array_neurongroup_1_not_refractory, sizeof(char)*_num__array_neurongroup_1_not_refractory, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_neurongroup_1_not_refractory;
    outfile__array_neurongroup_1_not_refractory.open("results/_array_neurongroup_1_not_refractory_-6252862397328651189", ios::binary | ios::out);
    if(outfile__array_neurongroup_1_not_refractory.is_open())
    {
        outfile__array_neurongroup_1_not_refractory.write(reinterpret_cast<char*>(_array_neurongroup_1_not_refractory), 100*sizeof(char));
        outfile__array_neurongroup_1_not_refractory.close();
    } else
    {
        std::cout << "Error writing output file for _array_neurongroup_1_not_refractory." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_neurongroup_1_V, dev_array_neurongroup_1_V, sizeof(double)*_num__array_neurongroup_1_V, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_neurongroup_1_V;
    outfile__array_neurongroup_1_V.open("results/_array_neurongroup_1_V_-1395569865091706992", ios::binary | ios::out);
    if(outfile__array_neurongroup_1_V.is_open())
    {
        outfile__array_neurongroup_1_V.write(reinterpret_cast<char*>(_array_neurongroup_1_V), 100*sizeof(double));
        outfile__array_neurongroup_1_V.close();
    } else
    {
        std::cout << "Error writing output file for _array_neurongroup_1_V." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_neurongroup_g_PN_iKC, dev_array_neurongroup_g_PN_iKC, sizeof(double)*_num__array_neurongroup_g_PN_iKC, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_neurongroup_g_PN_iKC;
    outfile__array_neurongroup_g_PN_iKC.open("results/_array_neurongroup_g_PN_iKC_-4808752820085404947", ios::binary | ios::out);
    if(outfile__array_neurongroup_g_PN_iKC.is_open())
    {
        outfile__array_neurongroup_g_PN_iKC.write(reinterpret_cast<char*>(_array_neurongroup_g_PN_iKC), 2500*sizeof(double));
        outfile__array_neurongroup_g_PN_iKC.close();
    } else
    {
        std::cout << "Error writing output file for _array_neurongroup_g_PN_iKC." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_neurongroup_h, dev_array_neurongroup_h, sizeof(double)*_num__array_neurongroup_h, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_neurongroup_h;
    outfile__array_neurongroup_h.open("results/_array_neurongroup_h_8698551290289247068", ios::binary | ios::out);
    if(outfile__array_neurongroup_h.is_open())
    {
        outfile__array_neurongroup_h.write(reinterpret_cast<char*>(_array_neurongroup_h), 2500*sizeof(double));
        outfile__array_neurongroup_h.close();
    } else
    {
        std::cout << "Error writing output file for _array_neurongroup_h." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_neurongroup_i, dev_array_neurongroup_i, sizeof(int32_t)*_num__array_neurongroup_i, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_neurongroup_i;
    outfile__array_neurongroup_i.open("results/_array_neurongroup_i_8335793832464323850", ios::binary | ios::out);
    if(outfile__array_neurongroup_i.is_open())
    {
        outfile__array_neurongroup_i.write(reinterpret_cast<char*>(_array_neurongroup_i), 2500*sizeof(int32_t));
        outfile__array_neurongroup_i.close();
    } else
    {
        std::cout << "Error writing output file for _array_neurongroup_i." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_neurongroup_lastspike, dev_array_neurongroup_lastspike, sizeof(double)*_num__array_neurongroup_lastspike, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_neurongroup_lastspike;
    outfile__array_neurongroup_lastspike.open("results/_array_neurongroup_lastspike_6427935437904044193", ios::binary | ios::out);
    if(outfile__array_neurongroup_lastspike.is_open())
    {
        outfile__array_neurongroup_lastspike.write(reinterpret_cast<char*>(_array_neurongroup_lastspike), 2500*sizeof(double));
        outfile__array_neurongroup_lastspike.close();
    } else
    {
        std::cout << "Error writing output file for _array_neurongroup_lastspike." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_neurongroup_m, dev_array_neurongroup_m, sizeof(double)*_num__array_neurongroup_m, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_neurongroup_m;
    outfile__array_neurongroup_m.open("results/_array_neurongroup_m_-5621447401784989625", ios::binary | ios::out);
    if(outfile__array_neurongroup_m.is_open())
    {
        outfile__array_neurongroup_m.write(reinterpret_cast<char*>(_array_neurongroup_m), 2500*sizeof(double));
        outfile__array_neurongroup_m.close();
    } else
    {
        std::cout << "Error writing output file for _array_neurongroup_m." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_neurongroup_n, dev_array_neurongroup_n, sizeof(double)*_num__array_neurongroup_n, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_neurongroup_n;
    outfile__array_neurongroup_n.open("results/_array_neurongroup_n_-2546797609979266637", ios::binary | ios::out);
    if(outfile__array_neurongroup_n.is_open())
    {
        outfile__array_neurongroup_n.write(reinterpret_cast<char*>(_array_neurongroup_n), 2500*sizeof(double));
        outfile__array_neurongroup_n.close();
    } else
    {
        std::cout << "Error writing output file for _array_neurongroup_n." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_neurongroup_not_refractory, dev_array_neurongroup_not_refractory, sizeof(char)*_num__array_neurongroup_not_refractory, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_neurongroup_not_refractory;
    outfile__array_neurongroup_not_refractory.open("results/_array_neurongroup_not_refractory_5726736962615233645", ios::binary | ios::out);
    if(outfile__array_neurongroup_not_refractory.is_open())
    {
        outfile__array_neurongroup_not_refractory.write(reinterpret_cast<char*>(_array_neurongroup_not_refractory), 2500*sizeof(char));
        outfile__array_neurongroup_not_refractory.close();
    } else
    {
        std::cout << "Error writing output file for _array_neurongroup_not_refractory." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_neurongroup_V, dev_array_neurongroup_V, sizeof(double)*_num__array_neurongroup_V, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_neurongroup_V;
    outfile__array_neurongroup_V.open("results/_array_neurongroup_V_2686151377283509651", ios::binary | ios::out);
    if(outfile__array_neurongroup_V.is_open())
    {
        outfile__array_neurongroup_V.write(reinterpret_cast<char*>(_array_neurongroup_V), 2500*sizeof(double));
        outfile__array_neurongroup_V.close();
    } else
    {
        std::cout << "Error writing output file for _array_neurongroup_V." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_spikegeneratorgroup__lastindex, dev_array_spikegeneratorgroup__lastindex, sizeof(int32_t)*_num__array_spikegeneratorgroup__lastindex, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_spikegeneratorgroup__lastindex;
    outfile__array_spikegeneratorgroup__lastindex.open("results/_array_spikegeneratorgroup__lastindex_1821964835846880533", ios::binary | ios::out);
    if(outfile__array_spikegeneratorgroup__lastindex.is_open())
    {
        outfile__array_spikegeneratorgroup__lastindex.write(reinterpret_cast<char*>(_array_spikegeneratorgroup__lastindex), 1*sizeof(int32_t));
        outfile__array_spikegeneratorgroup__lastindex.close();
    } else
    {
        std::cout << "Error writing output file for _array_spikegeneratorgroup__lastindex." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_spikegeneratorgroup__period_bins, dev_array_spikegeneratorgroup__period_bins, sizeof(int32_t)*_num__array_spikegeneratorgroup__period_bins, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_spikegeneratorgroup__period_bins;
    outfile__array_spikegeneratorgroup__period_bins.open("results/_array_spikegeneratorgroup__period_bins_-7971398493031931846", ios::binary | ios::out);
    if(outfile__array_spikegeneratorgroup__period_bins.is_open())
    {
        outfile__array_spikegeneratorgroup__period_bins.write(reinterpret_cast<char*>(_array_spikegeneratorgroup__period_bins), 1*sizeof(int32_t));
        outfile__array_spikegeneratorgroup__period_bins.close();
    } else
    {
        std::cout << "Error writing output file for _array_spikegeneratorgroup__period_bins." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_spikegeneratorgroup_i, dev_array_spikegeneratorgroup_i, sizeof(int32_t)*_num__array_spikegeneratorgroup_i, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_spikegeneratorgroup_i;
    outfile__array_spikegeneratorgroup_i.open("results/_array_spikegeneratorgroup_i_-1292482055040653574", ios::binary | ios::out);
    if(outfile__array_spikegeneratorgroup_i.is_open())
    {
        outfile__array_spikegeneratorgroup_i.write(reinterpret_cast<char*>(_array_spikegeneratorgroup_i), 100*sizeof(int32_t));
        outfile__array_spikegeneratorgroup_i.close();
    } else
    {
        std::cout << "Error writing output file for _array_spikegeneratorgroup_i." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_spikegeneratorgroup_period, dev_array_spikegeneratorgroup_period, sizeof(double)*_num__array_spikegeneratorgroup_period, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_spikegeneratorgroup_period;
    outfile__array_spikegeneratorgroup_period.open("results/_array_spikegeneratorgroup_period_-353366131269823746", ios::binary | ios::out);
    if(outfile__array_spikegeneratorgroup_period.is_open())
    {
        outfile__array_spikegeneratorgroup_period.write(reinterpret_cast<char*>(_array_spikegeneratorgroup_period), 1*sizeof(double));
        outfile__array_spikegeneratorgroup_period.close();
    } else
    {
        std::cout << "Error writing output file for _array_spikegeneratorgroup_period." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_spikemonitor_1__source_idx, dev_array_spikemonitor_1__source_idx, sizeof(int32_t)*_num__array_spikemonitor_1__source_idx, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_spikemonitor_1__source_idx;
    outfile__array_spikemonitor_1__source_idx.open("results/_array_spikemonitor_1__source_idx_-50543664629489326", ios::binary | ios::out);
    if(outfile__array_spikemonitor_1__source_idx.is_open())
    {
        outfile__array_spikemonitor_1__source_idx.write(reinterpret_cast<char*>(_array_spikemonitor_1__source_idx), 2500*sizeof(int32_t));
        outfile__array_spikemonitor_1__source_idx.close();
    } else
    {
        std::cout << "Error writing output file for _array_spikemonitor_1__source_idx." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_spikemonitor_1_count, dev_array_spikemonitor_1_count, sizeof(int32_t)*_num__array_spikemonitor_1_count, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_spikemonitor_1_count;
    outfile__array_spikemonitor_1_count.open("results/_array_spikemonitor_1_count_6013008031212298333", ios::binary | ios::out);
    if(outfile__array_spikemonitor_1_count.is_open())
    {
        outfile__array_spikemonitor_1_count.write(reinterpret_cast<char*>(_array_spikemonitor_1_count), 2500*sizeof(int32_t));
        outfile__array_spikemonitor_1_count.close();
    } else
    {
        std::cout << "Error writing output file for _array_spikemonitor_1_count." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_spikemonitor_1_N, dev_array_spikemonitor_1_N, sizeof(int32_t)*_num__array_spikemonitor_1_N, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_spikemonitor_1_N;
    outfile__array_spikemonitor_1_N.open("results/_array_spikemonitor_1_N_3169190033621949867", ios::binary | ios::out);
    if(outfile__array_spikemonitor_1_N.is_open())
    {
        outfile__array_spikemonitor_1_N.write(reinterpret_cast<char*>(_array_spikemonitor_1_N), 1*sizeof(int32_t));
        outfile__array_spikemonitor_1_N.close();
    } else
    {
        std::cout << "Error writing output file for _array_spikemonitor_1_N." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_spikemonitor_2__source_idx, dev_array_spikemonitor_2__source_idx, sizeof(int32_t)*_num__array_spikemonitor_2__source_idx, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_spikemonitor_2__source_idx;
    outfile__array_spikemonitor_2__source_idx.open("results/_array_spikemonitor_2__source_idx_-7925017314742328674", ios::binary | ios::out);
    if(outfile__array_spikemonitor_2__source_idx.is_open())
    {
        outfile__array_spikemonitor_2__source_idx.write(reinterpret_cast<char*>(_array_spikemonitor_2__source_idx), 100*sizeof(int32_t));
        outfile__array_spikemonitor_2__source_idx.close();
    } else
    {
        std::cout << "Error writing output file for _array_spikemonitor_2__source_idx." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_spikemonitor_2_count, dev_array_spikemonitor_2_count, sizeof(int32_t)*_num__array_spikemonitor_2_count, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_spikemonitor_2_count;
    outfile__array_spikemonitor_2_count.open("results/_array_spikemonitor_2_count_7670286378054215486", ios::binary | ios::out);
    if(outfile__array_spikemonitor_2_count.is_open())
    {
        outfile__array_spikemonitor_2_count.write(reinterpret_cast<char*>(_array_spikemonitor_2_count), 100*sizeof(int32_t));
        outfile__array_spikemonitor_2_count.close();
    } else
    {
        std::cout << "Error writing output file for _array_spikemonitor_2_count." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_spikemonitor_2_N, dev_array_spikemonitor_2_N, sizeof(int32_t)*_num__array_spikemonitor_2_N, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_spikemonitor_2_N;
    outfile__array_spikemonitor_2_N.open("results/_array_spikemonitor_2_N_6693733537479841813", ios::binary | ios::out);
    if(outfile__array_spikemonitor_2_N.is_open())
    {
        outfile__array_spikemonitor_2_N.write(reinterpret_cast<char*>(_array_spikemonitor_2_N), 1*sizeof(int32_t));
        outfile__array_spikemonitor_2_N.close();
    } else
    {
        std::cout << "Error writing output file for _array_spikemonitor_2_N." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_spikemonitor__source_idx, dev_array_spikemonitor__source_idx, sizeof(int32_t)*_num__array_spikemonitor__source_idx, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_spikemonitor__source_idx;
    outfile__array_spikemonitor__source_idx.open("results/_array_spikemonitor__source_idx_-8117872864355079535", ios::binary | ios::out);
    if(outfile__array_spikemonitor__source_idx.is_open())
    {
        outfile__array_spikemonitor__source_idx.write(reinterpret_cast<char*>(_array_spikemonitor__source_idx), 100*sizeof(int32_t));
        outfile__array_spikemonitor__source_idx.close();
    } else
    {
        std::cout << "Error writing output file for _array_spikemonitor__source_idx." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_spikemonitor_count, dev_array_spikemonitor_count, sizeof(int32_t)*_num__array_spikemonitor_count, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_spikemonitor_count;
    outfile__array_spikemonitor_count.open("results/_array_spikemonitor_count_2626824674132290633", ios::binary | ios::out);
    if(outfile__array_spikemonitor_count.is_open())
    {
        outfile__array_spikemonitor_count.write(reinterpret_cast<char*>(_array_spikemonitor_count), 100*sizeof(int32_t));
        outfile__array_spikemonitor_count.close();
    } else
    {
        std::cout << "Error writing output file for _array_spikemonitor_count." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_spikemonitor_N, dev_array_spikemonitor_N, sizeof(int32_t)*_num__array_spikemonitor_N, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_spikemonitor_N;
    outfile__array_spikemonitor_N.open("results/_array_spikemonitor_N_6263166261093207124", ios::binary | ios::out);
    if(outfile__array_spikemonitor_N.is_open())
    {
        outfile__array_spikemonitor_N.write(reinterpret_cast<char*>(_array_spikemonitor_N), 1*sizeof(int32_t));
        outfile__array_spikemonitor_N.close();
    } else
    {
        std::cout << "Error writing output file for _array_spikemonitor_N." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_synapses_1_N, dev_array_synapses_1_N, sizeof(int32_t)*_num__array_synapses_1_N, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_synapses_1_N;
    outfile__array_synapses_1_N.open("results/_array_synapses_1_N_-5388579170602877692", ios::binary | ios::out);
    if(outfile__array_synapses_1_N.is_open())
    {
        outfile__array_synapses_1_N.write(reinterpret_cast<char*>(_array_synapses_1_N), 1*sizeof(int32_t));
        outfile__array_synapses_1_N.close();
    } else
    {
        std::cout << "Error writing output file for _array_synapses_1_N." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_synapses_2_N, dev_array_synapses_2_N, sizeof(int32_t)*_num__array_synapses_2_N, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_synapses_2_N;
    outfile__array_synapses_2_N.open("results/_array_synapses_2_N_5269920966642024342", ios::binary | ios::out);
    if(outfile__array_synapses_2_N.is_open())
    {
        outfile__array_synapses_2_N.write(reinterpret_cast<char*>(_array_synapses_2_N), 1*sizeof(int32_t));
        outfile__array_synapses_2_N.close();
    } else
    {
        std::cout << "Error writing output file for _array_synapses_2_N." << endl;
    }
    CUDA_SAFE_CALL(
            hipMemcpy(_array_synapses_N, dev_array_synapses_N, sizeof(int32_t)*_num__array_synapses_N, hipMemcpyDeviceToHost)
            );
    ofstream outfile__array_synapses_N;
    outfile__array_synapses_N.open("results/_array_synapses_N_-2482695578908200934", ios::binary | ios::out);
    if(outfile__array_synapses_N.is_open())
    {
        outfile__array_synapses_N.write(reinterpret_cast<char*>(_array_synapses_N), 1*sizeof(int32_t));
        outfile__array_synapses_N.close();
    } else
    {
        std::cout << "Error writing output file for _array_synapses_N." << endl;
    }

    _dynamic_array_spikegeneratorgroup__timebins = dev_dynamic_array_spikegeneratorgroup__timebins;
    ofstream outfile__dynamic_array_spikegeneratorgroup__timebins;
    outfile__dynamic_array_spikegeneratorgroup__timebins.open("results/_dynamic_array_spikegeneratorgroup__timebins_8131810897310887393", ios::binary | ios::out);
    if(outfile__dynamic_array_spikegeneratorgroup__timebins.is_open())
    {
        outfile__dynamic_array_spikegeneratorgroup__timebins.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_spikegeneratorgroup__timebins[0])), _dynamic_array_spikegeneratorgroup__timebins.size()*sizeof(int32_t));
        outfile__dynamic_array_spikegeneratorgroup__timebins.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_spikegeneratorgroup__timebins." << endl;
    }
    _dynamic_array_spikegeneratorgroup_neuron_index = dev_dynamic_array_spikegeneratorgroup_neuron_index;
    ofstream outfile__dynamic_array_spikegeneratorgroup_neuron_index;
    outfile__dynamic_array_spikegeneratorgroup_neuron_index.open("results/_dynamic_array_spikegeneratorgroup_neuron_index_-7594505304508306195", ios::binary | ios::out);
    if(outfile__dynamic_array_spikegeneratorgroup_neuron_index.is_open())
    {
        outfile__dynamic_array_spikegeneratorgroup_neuron_index.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_spikegeneratorgroup_neuron_index[0])), _dynamic_array_spikegeneratorgroup_neuron_index.size()*sizeof(int32_t));
        outfile__dynamic_array_spikegeneratorgroup_neuron_index.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_spikegeneratorgroup_neuron_index." << endl;
    }
    _dynamic_array_spikegeneratorgroup_spike_number = dev_dynamic_array_spikegeneratorgroup_spike_number;
    ofstream outfile__dynamic_array_spikegeneratorgroup_spike_number;
    outfile__dynamic_array_spikegeneratorgroup_spike_number.open("results/_dynamic_array_spikegeneratorgroup_spike_number_-4815301131874600719", ios::binary | ios::out);
    if(outfile__dynamic_array_spikegeneratorgroup_spike_number.is_open())
    {
        outfile__dynamic_array_spikegeneratorgroup_spike_number.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_spikegeneratorgroup_spike_number[0])), _dynamic_array_spikegeneratorgroup_spike_number.size()*sizeof(int32_t));
        outfile__dynamic_array_spikegeneratorgroup_spike_number.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_spikegeneratorgroup_spike_number." << endl;
    }
    _dynamic_array_spikegeneratorgroup_spike_time = dev_dynamic_array_spikegeneratorgroup_spike_time;
    ofstream outfile__dynamic_array_spikegeneratorgroup_spike_time;
    outfile__dynamic_array_spikegeneratorgroup_spike_time.open("results/_dynamic_array_spikegeneratorgroup_spike_time_6567911360708844700", ios::binary | ios::out);
    if(outfile__dynamic_array_spikegeneratorgroup_spike_time.is_open())
    {
        outfile__dynamic_array_spikegeneratorgroup_spike_time.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_spikegeneratorgroup_spike_time[0])), _dynamic_array_spikegeneratorgroup_spike_time.size()*sizeof(double));
        outfile__dynamic_array_spikegeneratorgroup_spike_time.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_spikegeneratorgroup_spike_time." << endl;
    }
    _dynamic_array_spikemonitor_1_i = dev_dynamic_array_spikemonitor_1_i;
    ofstream outfile__dynamic_array_spikemonitor_1_i;
    outfile__dynamic_array_spikemonitor_1_i.open("results/_dynamic_array_spikemonitor_1_i_-2190502851196353835", ios::binary | ios::out);
    if(outfile__dynamic_array_spikemonitor_1_i.is_open())
    {
        outfile__dynamic_array_spikemonitor_1_i.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_spikemonitor_1_i[0])), _dynamic_array_spikemonitor_1_i.size()*sizeof(int32_t));
        outfile__dynamic_array_spikemonitor_1_i.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_spikemonitor_1_i." << endl;
    }
    _dynamic_array_spikemonitor_1_t = dev_dynamic_array_spikemonitor_1_t;
    ofstream outfile__dynamic_array_spikemonitor_1_t;
    outfile__dynamic_array_spikemonitor_1_t.open("results/_dynamic_array_spikemonitor_1_t_-841006843677588084", ios::binary | ios::out);
    if(outfile__dynamic_array_spikemonitor_1_t.is_open())
    {
        outfile__dynamic_array_spikemonitor_1_t.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_spikemonitor_1_t[0])), _dynamic_array_spikemonitor_1_t.size()*sizeof(double));
        outfile__dynamic_array_spikemonitor_1_t.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_spikemonitor_1_t." << endl;
    }
    _dynamic_array_spikemonitor_2_i = dev_dynamic_array_spikemonitor_2_i;
    ofstream outfile__dynamic_array_spikemonitor_2_i;
    outfile__dynamic_array_spikemonitor_2_i.open("results/_dynamic_array_spikemonitor_2_i_-7452697810678630303", ios::binary | ios::out);
    if(outfile__dynamic_array_spikemonitor_2_i.is_open())
    {
        outfile__dynamic_array_spikemonitor_2_i.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_spikemonitor_2_i[0])), _dynamic_array_spikemonitor_2_i.size()*sizeof(int32_t));
        outfile__dynamic_array_spikemonitor_2_i.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_spikemonitor_2_i." << endl;
    }
    _dynamic_array_spikemonitor_2_t = dev_dynamic_array_spikemonitor_2_t;
    ofstream outfile__dynamic_array_spikemonitor_2_t;
    outfile__dynamic_array_spikemonitor_2_t.open("results/_dynamic_array_spikemonitor_2_t_-2066051122613997313", ios::binary | ios::out);
    if(outfile__dynamic_array_spikemonitor_2_t.is_open())
    {
        outfile__dynamic_array_spikemonitor_2_t.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_spikemonitor_2_t[0])), _dynamic_array_spikemonitor_2_t.size()*sizeof(double));
        outfile__dynamic_array_spikemonitor_2_t.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_spikemonitor_2_t." << endl;
    }
    _dynamic_array_spikemonitor_i = dev_dynamic_array_spikemonitor_i;
    ofstream outfile__dynamic_array_spikemonitor_i;
    outfile__dynamic_array_spikemonitor_i.open("results/_dynamic_array_spikemonitor_i_2878104665717261157", ios::binary | ios::out);
    if(outfile__dynamic_array_spikemonitor_i.is_open())
    {
        outfile__dynamic_array_spikemonitor_i.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_spikemonitor_i[0])), _dynamic_array_spikemonitor_i.size()*sizeof(int32_t));
        outfile__dynamic_array_spikemonitor_i.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_spikemonitor_i." << endl;
    }
    _dynamic_array_spikemonitor_t = dev_dynamic_array_spikemonitor_t;
    ofstream outfile__dynamic_array_spikemonitor_t;
    outfile__dynamic_array_spikemonitor_t.open("results/_dynamic_array_spikemonitor_t_7865095316440674513", ios::binary | ios::out);
    if(outfile__dynamic_array_spikemonitor_t.is_open())
    {
        outfile__dynamic_array_spikemonitor_t.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_spikemonitor_t[0])), _dynamic_array_spikemonitor_t.size()*sizeof(double));
        outfile__dynamic_array_spikemonitor_t.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_spikemonitor_t." << endl;
    }
    ofstream outfile__dynamic_array_synapses_1__synaptic_post;
    outfile__dynamic_array_synapses_1__synaptic_post.open("results/_dynamic_array_synapses_1__synaptic_post_-7537747434503640794", ios::binary | ios::out);
    if(outfile__dynamic_array_synapses_1__synaptic_post.is_open())
    {
        outfile__dynamic_array_synapses_1__synaptic_post.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_synapses_1__synaptic_post[0])), _dynamic_array_synapses_1__synaptic_post.size()*sizeof(int32_t));
        outfile__dynamic_array_synapses_1__synaptic_post.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_synapses_1__synaptic_post." << endl;
    }
    ofstream outfile__dynamic_array_synapses_1__synaptic_pre;
    outfile__dynamic_array_synapses_1__synaptic_pre.open("results/_dynamic_array_synapses_1__synaptic_pre_-8170898951251124790", ios::binary | ios::out);
    if(outfile__dynamic_array_synapses_1__synaptic_pre.is_open())
    {
        outfile__dynamic_array_synapses_1__synaptic_pre.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_synapses_1__synaptic_pre[0])), _dynamic_array_synapses_1__synaptic_pre.size()*sizeof(int32_t));
        outfile__dynamic_array_synapses_1__synaptic_pre.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_synapses_1__synaptic_pre." << endl;
    }
    _dynamic_array_synapses_1_Apost = dev_dynamic_array_synapses_1_Apost;
    ofstream outfile__dynamic_array_synapses_1_Apost;
    outfile__dynamic_array_synapses_1_Apost.open("results/_dynamic_array_synapses_1_Apost_6485379228718605548", ios::binary | ios::out);
    if(outfile__dynamic_array_synapses_1_Apost.is_open())
    {
        outfile__dynamic_array_synapses_1_Apost.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_synapses_1_Apost[0])), _dynamic_array_synapses_1_Apost.size()*sizeof(double));
        outfile__dynamic_array_synapses_1_Apost.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_synapses_1_Apost." << endl;
    }
    _dynamic_array_synapses_1_Apre = dev_dynamic_array_synapses_1_Apre;
    ofstream outfile__dynamic_array_synapses_1_Apre;
    outfile__dynamic_array_synapses_1_Apre.open("results/_dynamic_array_synapses_1_Apre_1158801600114762896", ios::binary | ios::out);
    if(outfile__dynamic_array_synapses_1_Apre.is_open())
    {
        outfile__dynamic_array_synapses_1_Apre.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_synapses_1_Apre[0])), _dynamic_array_synapses_1_Apre.size()*sizeof(double));
        outfile__dynamic_array_synapses_1_Apre.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_synapses_1_Apre." << endl;
    }
    ofstream outfile__dynamic_array_synapses_1_delay;
    outfile__dynamic_array_synapses_1_delay.open("results/_dynamic_array_synapses_1_delay_-2566178675962201282", ios::binary | ios::out);
    if(outfile__dynamic_array_synapses_1_delay.is_open())
    {
        outfile__dynamic_array_synapses_1_delay.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_synapses_1_delay[0])), _dynamic_array_synapses_1_delay.size()*sizeof(double));
        outfile__dynamic_array_synapses_1_delay.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_synapses_1_delay." << endl;
    }
    ofstream outfile__dynamic_array_synapses_1_delay_1;
    outfile__dynamic_array_synapses_1_delay_1.open("results/_dynamic_array_synapses_1_delay_1_-2293552668042484320", ios::binary | ios::out);
    if(outfile__dynamic_array_synapses_1_delay_1.is_open())
    {
        outfile__dynamic_array_synapses_1_delay_1.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_synapses_1_delay_1[0])), _dynamic_array_synapses_1_delay_1.size()*sizeof(double));
        outfile__dynamic_array_synapses_1_delay_1.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_synapses_1_delay_1." << endl;
    }
    _dynamic_array_synapses_1_g_raw = dev_dynamic_array_synapses_1_g_raw;
    ofstream outfile__dynamic_array_synapses_1_g_raw;
    outfile__dynamic_array_synapses_1_g_raw.open("results/_dynamic_array_synapses_1_g_raw_-296211884898250956", ios::binary | ios::out);
    if(outfile__dynamic_array_synapses_1_g_raw.is_open())
    {
        outfile__dynamic_array_synapses_1_g_raw.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_synapses_1_g_raw[0])), _dynamic_array_synapses_1_g_raw.size()*sizeof(double));
        outfile__dynamic_array_synapses_1_g_raw.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_synapses_1_g_raw." << endl;
    }
    _dynamic_array_synapses_1_lastupdate = dev_dynamic_array_synapses_1_lastupdate;
    ofstream outfile__dynamic_array_synapses_1_lastupdate;
    outfile__dynamic_array_synapses_1_lastupdate.open("results/_dynamic_array_synapses_1_lastupdate_-4620983009986066308", ios::binary | ios::out);
    if(outfile__dynamic_array_synapses_1_lastupdate.is_open())
    {
        outfile__dynamic_array_synapses_1_lastupdate.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_synapses_1_lastupdate[0])), _dynamic_array_synapses_1_lastupdate.size()*sizeof(double));
        outfile__dynamic_array_synapses_1_lastupdate.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_synapses_1_lastupdate." << endl;
    }
    _dynamic_array_synapses_1_N_incoming = dev_dynamic_array_synapses_1_N_incoming;
    ofstream outfile__dynamic_array_synapses_1_N_incoming;
    outfile__dynamic_array_synapses_1_N_incoming.open("results/_dynamic_array_synapses_1_N_incoming_-5416286353695559554", ios::binary | ios::out);
    if(outfile__dynamic_array_synapses_1_N_incoming.is_open())
    {
        outfile__dynamic_array_synapses_1_N_incoming.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_synapses_1_N_incoming[0])), _dynamic_array_synapses_1_N_incoming.size()*sizeof(int32_t));
        outfile__dynamic_array_synapses_1_N_incoming.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_synapses_1_N_incoming." << endl;
    }
    _dynamic_array_synapses_1_N_outgoing = dev_dynamic_array_synapses_1_N_outgoing;
    ofstream outfile__dynamic_array_synapses_1_N_outgoing;
    outfile__dynamic_array_synapses_1_N_outgoing.open("results/_dynamic_array_synapses_1_N_outgoing_5769272226699040095", ios::binary | ios::out);
    if(outfile__dynamic_array_synapses_1_N_outgoing.is_open())
    {
        outfile__dynamic_array_synapses_1_N_outgoing.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_synapses_1_N_outgoing[0])), _dynamic_array_synapses_1_N_outgoing.size()*sizeof(int32_t));
        outfile__dynamic_array_synapses_1_N_outgoing.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_synapses_1_N_outgoing." << endl;
    }
    ofstream outfile__dynamic_array_synapses_2__synaptic_post;
    outfile__dynamic_array_synapses_2__synaptic_post.open("results/_dynamic_array_synapses_2__synaptic_post_-8504964520201554399", ios::binary | ios::out);
    if(outfile__dynamic_array_synapses_2__synaptic_post.is_open())
    {
        outfile__dynamic_array_synapses_2__synaptic_post.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_synapses_2__synaptic_post[0])), _dynamic_array_synapses_2__synaptic_post.size()*sizeof(int32_t));
        outfile__dynamic_array_synapses_2__synaptic_post.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_synapses_2__synaptic_post." << endl;
    }
    ofstream outfile__dynamic_array_synapses_2__synaptic_pre;
    outfile__dynamic_array_synapses_2__synaptic_pre.open("results/_dynamic_array_synapses_2__synaptic_pre_-5492879376519788356", ios::binary | ios::out);
    if(outfile__dynamic_array_synapses_2__synaptic_pre.is_open())
    {
        outfile__dynamic_array_synapses_2__synaptic_pre.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_synapses_2__synaptic_pre[0])), _dynamic_array_synapses_2__synaptic_pre.size()*sizeof(int32_t));
        outfile__dynamic_array_synapses_2__synaptic_pre.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_synapses_2__synaptic_pre." << endl;
    }
    ofstream outfile__dynamic_array_synapses_2_delay;
    outfile__dynamic_array_synapses_2_delay.open("results/_dynamic_array_synapses_2_delay_-785530481191211215", ios::binary | ios::out);
    if(outfile__dynamic_array_synapses_2_delay.is_open())
    {
        outfile__dynamic_array_synapses_2_delay.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_synapses_2_delay[0])), _dynamic_array_synapses_2_delay.size()*sizeof(double));
        outfile__dynamic_array_synapses_2_delay.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_synapses_2_delay." << endl;
    }
    _dynamic_array_synapses_2_N_incoming = dev_dynamic_array_synapses_2_N_incoming;
    ofstream outfile__dynamic_array_synapses_2_N_incoming;
    outfile__dynamic_array_synapses_2_N_incoming.open("results/_dynamic_array_synapses_2_N_incoming_-2633956166385116811", ios::binary | ios::out);
    if(outfile__dynamic_array_synapses_2_N_incoming.is_open())
    {
        outfile__dynamic_array_synapses_2_N_incoming.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_synapses_2_N_incoming[0])), _dynamic_array_synapses_2_N_incoming.size()*sizeof(int32_t));
        outfile__dynamic_array_synapses_2_N_incoming.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_synapses_2_N_incoming." << endl;
    }
    _dynamic_array_synapses_2_N_outgoing = dev_dynamic_array_synapses_2_N_outgoing;
    ofstream outfile__dynamic_array_synapses_2_N_outgoing;
    outfile__dynamic_array_synapses_2_N_outgoing.open("results/_dynamic_array_synapses_2_N_outgoing_-8330418898748964037", ios::binary | ios::out);
    if(outfile__dynamic_array_synapses_2_N_outgoing.is_open())
    {
        outfile__dynamic_array_synapses_2_N_outgoing.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_synapses_2_N_outgoing[0])), _dynamic_array_synapses_2_N_outgoing.size()*sizeof(int32_t));
        outfile__dynamic_array_synapses_2_N_outgoing.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_synapses_2_N_outgoing." << endl;
    }
    ofstream outfile__dynamic_array_synapses__synaptic_post;
    outfile__dynamic_array_synapses__synaptic_post.open("results/_dynamic_array_synapses__synaptic_post_6330116830759336919", ios::binary | ios::out);
    if(outfile__dynamic_array_synapses__synaptic_post.is_open())
    {
        outfile__dynamic_array_synapses__synaptic_post.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_synapses__synaptic_post[0])), _dynamic_array_synapses__synaptic_post.size()*sizeof(int32_t));
        outfile__dynamic_array_synapses__synaptic_post.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_synapses__synaptic_post." << endl;
    }
    ofstream outfile__dynamic_array_synapses__synaptic_pre;
    outfile__dynamic_array_synapses__synaptic_pre.open("results/_dynamic_array_synapses__synaptic_pre_2137649452266235309", ios::binary | ios::out);
    if(outfile__dynamic_array_synapses__synaptic_pre.is_open())
    {
        outfile__dynamic_array_synapses__synaptic_pre.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_synapses__synaptic_pre[0])), _dynamic_array_synapses__synaptic_pre.size()*sizeof(int32_t));
        outfile__dynamic_array_synapses__synaptic_pre.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_synapses__synaptic_pre." << endl;
    }
    ofstream outfile__dynamic_array_synapses_delay;
    outfile__dynamic_array_synapses_delay.open("results/_dynamic_array_synapses_delay_6546873993127671381", ios::binary | ios::out);
    if(outfile__dynamic_array_synapses_delay.is_open())
    {
        outfile__dynamic_array_synapses_delay.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_synapses_delay[0])), _dynamic_array_synapses_delay.size()*sizeof(double));
        outfile__dynamic_array_synapses_delay.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_synapses_delay." << endl;
    }
    _dynamic_array_synapses_N_incoming = dev_dynamic_array_synapses_N_incoming;
    ofstream outfile__dynamic_array_synapses_N_incoming;
    outfile__dynamic_array_synapses_N_incoming.open("results/_dynamic_array_synapses_N_incoming_2854242842403593343", ios::binary | ios::out);
    if(outfile__dynamic_array_synapses_N_incoming.is_open())
    {
        outfile__dynamic_array_synapses_N_incoming.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_synapses_N_incoming[0])), _dynamic_array_synapses_N_incoming.size()*sizeof(int32_t));
        outfile__dynamic_array_synapses_N_incoming.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_synapses_N_incoming." << endl;
    }
    _dynamic_array_synapses_N_outgoing = dev_dynamic_array_synapses_N_outgoing;
    ofstream outfile__dynamic_array_synapses_N_outgoing;
    outfile__dynamic_array_synapses_N_outgoing.open("results/_dynamic_array_synapses_N_outgoing_-6705529799763348580", ios::binary | ios::out);
    if(outfile__dynamic_array_synapses_N_outgoing.is_open())
    {
        outfile__dynamic_array_synapses_N_outgoing.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_synapses_N_outgoing[0])), _dynamic_array_synapses_N_outgoing.size()*sizeof(int32_t));
        outfile__dynamic_array_synapses_N_outgoing.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_synapses_N_outgoing." << endl;
    }
    _dynamic_array_synapses_weight = dev_dynamic_array_synapses_weight;
    ofstream outfile__dynamic_array_synapses_weight;
    outfile__dynamic_array_synapses_weight.open("results/_dynamic_array_synapses_weight_-4970804317082307398", ios::binary | ios::out);
    if(outfile__dynamic_array_synapses_weight.is_open())
    {
        outfile__dynamic_array_synapses_weight.write(reinterpret_cast<char*>(thrust::raw_pointer_cast(&_dynamic_array_synapses_weight[0])), _dynamic_array_synapses_weight.size()*sizeof(double));
        outfile__dynamic_array_synapses_weight.close();
    } else
    {
        std::cout << "Error writing output file for _dynamic_array_synapses_weight." << endl;
    }


    // Write last run info to disk
    ofstream outfile_last_run_info;
    outfile_last_run_info.open("results/last_run_info.txt", ios::out);
    if(outfile_last_run_info.is_open())
    {
        outfile_last_run_info << (Network::_last_run_time) << " " << (Network::_last_run_completed_fraction) << std::endl;
        outfile_last_run_info.close();
    } else
    {
        std::cout << "Error writing last run info to file." << std::endl;
    }
}

__global__ void synapses_pre_destroy()
{
    using namespace brian;

    synapses_pre.destroy();
}
__global__ void synapses_1_post_destroy()
{
    using namespace brian;

    synapses_1_post.destroy();
}
__global__ void synapses_1_pre_destroy()
{
    using namespace brian;

    synapses_1_pre.destroy();
}
__global__ void synapses_2_pre_destroy()
{
    using namespace brian;

    synapses_2_pre.destroy();
}

void _dealloc_arrays()
{
    using namespace brian;


    CUDA_SAFE_CALL(
            hiprandDestroyGenerator(curand_generator)
            );

    synapses_pre_destroy<<<1,1>>>();
    CUDA_CHECK_ERROR("synapses_pre_destroy");
    synapses_1_post_destroy<<<1,1>>>();
    CUDA_CHECK_ERROR("synapses_1_post_destroy");
    synapses_1_pre_destroy<<<1,1>>>();
    CUDA_CHECK_ERROR("synapses_1_pre_destroy");
    synapses_2_pre_destroy<<<1,1>>>();
    CUDA_CHECK_ERROR("synapses_2_pre_destroy");

    dev_dynamic_array_spikegeneratorgroup__timebins.clear();
    thrust::device_vector<int32_t>().swap(dev_dynamic_array_spikegeneratorgroup__timebins);
    _dynamic_array_spikegeneratorgroup__timebins.clear();
    thrust::host_vector<int32_t>().swap(_dynamic_array_spikegeneratorgroup__timebins);
    dev_dynamic_array_spikegeneratorgroup_neuron_index.clear();
    thrust::device_vector<int32_t>().swap(dev_dynamic_array_spikegeneratorgroup_neuron_index);
    _dynamic_array_spikegeneratorgroup_neuron_index.clear();
    thrust::host_vector<int32_t>().swap(_dynamic_array_spikegeneratorgroup_neuron_index);
    dev_dynamic_array_spikegeneratorgroup_spike_number.clear();
    thrust::device_vector<int32_t>().swap(dev_dynamic_array_spikegeneratorgroup_spike_number);
    _dynamic_array_spikegeneratorgroup_spike_number.clear();
    thrust::host_vector<int32_t>().swap(_dynamic_array_spikegeneratorgroup_spike_number);
    dev_dynamic_array_spikegeneratorgroup_spike_time.clear();
    thrust::device_vector<double>().swap(dev_dynamic_array_spikegeneratorgroup_spike_time);
    _dynamic_array_spikegeneratorgroup_spike_time.clear();
    thrust::host_vector<double>().swap(_dynamic_array_spikegeneratorgroup_spike_time);
    dev_dynamic_array_spikemonitor_1_i.clear();
    thrust::device_vector<int32_t>().swap(dev_dynamic_array_spikemonitor_1_i);
    _dynamic_array_spikemonitor_1_i.clear();
    thrust::host_vector<int32_t>().swap(_dynamic_array_spikemonitor_1_i);
    dev_dynamic_array_spikemonitor_1_t.clear();
    thrust::device_vector<double>().swap(dev_dynamic_array_spikemonitor_1_t);
    _dynamic_array_spikemonitor_1_t.clear();
    thrust::host_vector<double>().swap(_dynamic_array_spikemonitor_1_t);
    dev_dynamic_array_spikemonitor_2_i.clear();
    thrust::device_vector<int32_t>().swap(dev_dynamic_array_spikemonitor_2_i);
    _dynamic_array_spikemonitor_2_i.clear();
    thrust::host_vector<int32_t>().swap(_dynamic_array_spikemonitor_2_i);
    dev_dynamic_array_spikemonitor_2_t.clear();
    thrust::device_vector<double>().swap(dev_dynamic_array_spikemonitor_2_t);
    _dynamic_array_spikemonitor_2_t.clear();
    thrust::host_vector<double>().swap(_dynamic_array_spikemonitor_2_t);
    dev_dynamic_array_spikemonitor_i.clear();
    thrust::device_vector<int32_t>().swap(dev_dynamic_array_spikemonitor_i);
    _dynamic_array_spikemonitor_i.clear();
    thrust::host_vector<int32_t>().swap(_dynamic_array_spikemonitor_i);
    dev_dynamic_array_spikemonitor_t.clear();
    thrust::device_vector<double>().swap(dev_dynamic_array_spikemonitor_t);
    _dynamic_array_spikemonitor_t.clear();
    thrust::host_vector<double>().swap(_dynamic_array_spikemonitor_t);
    dev_dynamic_array_synapses_1__synaptic_post.clear();
    thrust::device_vector<int32_t>().swap(dev_dynamic_array_synapses_1__synaptic_post);
    _dynamic_array_synapses_1__synaptic_post.clear();
    thrust::host_vector<int32_t>().swap(_dynamic_array_synapses_1__synaptic_post);
    dev_dynamic_array_synapses_1__synaptic_pre.clear();
    thrust::device_vector<int32_t>().swap(dev_dynamic_array_synapses_1__synaptic_pre);
    _dynamic_array_synapses_1__synaptic_pre.clear();
    thrust::host_vector<int32_t>().swap(_dynamic_array_synapses_1__synaptic_pre);
    dev_dynamic_array_synapses_1_Apost.clear();
    thrust::device_vector<double>().swap(dev_dynamic_array_synapses_1_Apost);
    _dynamic_array_synapses_1_Apost.clear();
    thrust::host_vector<double>().swap(_dynamic_array_synapses_1_Apost);
    dev_dynamic_array_synapses_1_Apre.clear();
    thrust::device_vector<double>().swap(dev_dynamic_array_synapses_1_Apre);
    _dynamic_array_synapses_1_Apre.clear();
    thrust::host_vector<double>().swap(_dynamic_array_synapses_1_Apre);
    dev_dynamic_array_synapses_1_delay.clear();
    thrust::device_vector<double>().swap(dev_dynamic_array_synapses_1_delay);
    _dynamic_array_synapses_1_delay.clear();
    thrust::host_vector<double>().swap(_dynamic_array_synapses_1_delay);
    dev_dynamic_array_synapses_1_delay_1.clear();
    thrust::device_vector<double>().swap(dev_dynamic_array_synapses_1_delay_1);
    _dynamic_array_synapses_1_delay_1.clear();
    thrust::host_vector<double>().swap(_dynamic_array_synapses_1_delay_1);
    dev_dynamic_array_synapses_1_g_raw.clear();
    thrust::device_vector<double>().swap(dev_dynamic_array_synapses_1_g_raw);
    _dynamic_array_synapses_1_g_raw.clear();
    thrust::host_vector<double>().swap(_dynamic_array_synapses_1_g_raw);
    dev_dynamic_array_synapses_1_lastupdate.clear();
    thrust::device_vector<double>().swap(dev_dynamic_array_synapses_1_lastupdate);
    _dynamic_array_synapses_1_lastupdate.clear();
    thrust::host_vector<double>().swap(_dynamic_array_synapses_1_lastupdate);
    dev_dynamic_array_synapses_1_N_incoming.clear();
    thrust::device_vector<int32_t>().swap(dev_dynamic_array_synapses_1_N_incoming);
    _dynamic_array_synapses_1_N_incoming.clear();
    thrust::host_vector<int32_t>().swap(_dynamic_array_synapses_1_N_incoming);
    dev_dynamic_array_synapses_1_N_outgoing.clear();
    thrust::device_vector<int32_t>().swap(dev_dynamic_array_synapses_1_N_outgoing);
    _dynamic_array_synapses_1_N_outgoing.clear();
    thrust::host_vector<int32_t>().swap(_dynamic_array_synapses_1_N_outgoing);
    dev_dynamic_array_synapses_2__synaptic_post.clear();
    thrust::device_vector<int32_t>().swap(dev_dynamic_array_synapses_2__synaptic_post);
    _dynamic_array_synapses_2__synaptic_post.clear();
    thrust::host_vector<int32_t>().swap(_dynamic_array_synapses_2__synaptic_post);
    dev_dynamic_array_synapses_2__synaptic_pre.clear();
    thrust::device_vector<int32_t>().swap(dev_dynamic_array_synapses_2__synaptic_pre);
    _dynamic_array_synapses_2__synaptic_pre.clear();
    thrust::host_vector<int32_t>().swap(_dynamic_array_synapses_2__synaptic_pre);
    dev_dynamic_array_synapses_2_delay.clear();
    thrust::device_vector<double>().swap(dev_dynamic_array_synapses_2_delay);
    _dynamic_array_synapses_2_delay.clear();
    thrust::host_vector<double>().swap(_dynamic_array_synapses_2_delay);
    dev_dynamic_array_synapses_2_N_incoming.clear();
    thrust::device_vector<int32_t>().swap(dev_dynamic_array_synapses_2_N_incoming);
    _dynamic_array_synapses_2_N_incoming.clear();
    thrust::host_vector<int32_t>().swap(_dynamic_array_synapses_2_N_incoming);
    dev_dynamic_array_synapses_2_N_outgoing.clear();
    thrust::device_vector<int32_t>().swap(dev_dynamic_array_synapses_2_N_outgoing);
    _dynamic_array_synapses_2_N_outgoing.clear();
    thrust::host_vector<int32_t>().swap(_dynamic_array_synapses_2_N_outgoing);
    dev_dynamic_array_synapses__synaptic_post.clear();
    thrust::device_vector<int32_t>().swap(dev_dynamic_array_synapses__synaptic_post);
    _dynamic_array_synapses__synaptic_post.clear();
    thrust::host_vector<int32_t>().swap(_dynamic_array_synapses__synaptic_post);
    dev_dynamic_array_synapses__synaptic_pre.clear();
    thrust::device_vector<int32_t>().swap(dev_dynamic_array_synapses__synaptic_pre);
    _dynamic_array_synapses__synaptic_pre.clear();
    thrust::host_vector<int32_t>().swap(_dynamic_array_synapses__synaptic_pre);
    dev_dynamic_array_synapses_delay.clear();
    thrust::device_vector<double>().swap(dev_dynamic_array_synapses_delay);
    _dynamic_array_synapses_delay.clear();
    thrust::host_vector<double>().swap(_dynamic_array_synapses_delay);
    dev_dynamic_array_synapses_N_incoming.clear();
    thrust::device_vector<int32_t>().swap(dev_dynamic_array_synapses_N_incoming);
    _dynamic_array_synapses_N_incoming.clear();
    thrust::host_vector<int32_t>().swap(_dynamic_array_synapses_N_incoming);
    dev_dynamic_array_synapses_N_outgoing.clear();
    thrust::device_vector<int32_t>().swap(dev_dynamic_array_synapses_N_outgoing);
    _dynamic_array_synapses_N_outgoing.clear();
    thrust::host_vector<int32_t>().swap(_dynamic_array_synapses_N_outgoing);
    dev_dynamic_array_synapses_weight.clear();
    thrust::device_vector<double>().swap(dev_dynamic_array_synapses_weight);
    _dynamic_array_synapses_weight.clear();
    thrust::host_vector<double>().swap(_dynamic_array_synapses_weight);

    if(_array_defaultclock_dt!=0)
    {
        delete [] _array_defaultclock_dt;
        _array_defaultclock_dt = 0;
    }
    if(dev_array_defaultclock_dt!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_defaultclock_dt)
                );
        dev_array_defaultclock_dt = 0;
    }
    if(_array_defaultclock_t!=0)
    {
        delete [] _array_defaultclock_t;
        _array_defaultclock_t = 0;
    }
    if(dev_array_defaultclock_t!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_defaultclock_t)
                );
        dev_array_defaultclock_t = 0;
    }
    if(_array_defaultclock_timestep!=0)
    {
        delete [] _array_defaultclock_timestep;
        _array_defaultclock_timestep = 0;
    }
    if(dev_array_defaultclock_timestep!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_defaultclock_timestep)
                );
        dev_array_defaultclock_timestep = 0;
    }
    if(_array_neurongroup_1_g_eKC_eKC!=0)
    {
        delete [] _array_neurongroup_1_g_eKC_eKC;
        _array_neurongroup_1_g_eKC_eKC = 0;
    }
    if(dev_array_neurongroup_1_g_eKC_eKC!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_neurongroup_1_g_eKC_eKC)
                );
        dev_array_neurongroup_1_g_eKC_eKC = 0;
    }
    if(_array_neurongroup_1_g_iKC_eKC!=0)
    {
        delete [] _array_neurongroup_1_g_iKC_eKC;
        _array_neurongroup_1_g_iKC_eKC = 0;
    }
    if(dev_array_neurongroup_1_g_iKC_eKC!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_neurongroup_1_g_iKC_eKC)
                );
        dev_array_neurongroup_1_g_iKC_eKC = 0;
    }
    if(_array_neurongroup_1_h!=0)
    {
        delete [] _array_neurongroup_1_h;
        _array_neurongroup_1_h = 0;
    }
    if(dev_array_neurongroup_1_h!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_neurongroup_1_h)
                );
        dev_array_neurongroup_1_h = 0;
    }
    if(_array_neurongroup_1_i!=0)
    {
        delete [] _array_neurongroup_1_i;
        _array_neurongroup_1_i = 0;
    }
    if(dev_array_neurongroup_1_i!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_neurongroup_1_i)
                );
        dev_array_neurongroup_1_i = 0;
    }
    if(_array_neurongroup_1_lastspike!=0)
    {
        delete [] _array_neurongroup_1_lastspike;
        _array_neurongroup_1_lastspike = 0;
    }
    if(dev_array_neurongroup_1_lastspike!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_neurongroup_1_lastspike)
                );
        dev_array_neurongroup_1_lastspike = 0;
    }
    if(_array_neurongroup_1_m!=0)
    {
        delete [] _array_neurongroup_1_m;
        _array_neurongroup_1_m = 0;
    }
    if(dev_array_neurongroup_1_m!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_neurongroup_1_m)
                );
        dev_array_neurongroup_1_m = 0;
    }
    if(_array_neurongroup_1_n!=0)
    {
        delete [] _array_neurongroup_1_n;
        _array_neurongroup_1_n = 0;
    }
    if(dev_array_neurongroup_1_n!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_neurongroup_1_n)
                );
        dev_array_neurongroup_1_n = 0;
    }
    if(_array_neurongroup_1_not_refractory!=0)
    {
        delete [] _array_neurongroup_1_not_refractory;
        _array_neurongroup_1_not_refractory = 0;
    }
    if(dev_array_neurongroup_1_not_refractory!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_neurongroup_1_not_refractory)
                );
        dev_array_neurongroup_1_not_refractory = 0;
    }
    if(_array_neurongroup_1_V!=0)
    {
        delete [] _array_neurongroup_1_V;
        _array_neurongroup_1_V = 0;
    }
    if(dev_array_neurongroup_1_V!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_neurongroup_1_V)
                );
        dev_array_neurongroup_1_V = 0;
    }
    if(_array_neurongroup_g_PN_iKC!=0)
    {
        delete [] _array_neurongroup_g_PN_iKC;
        _array_neurongroup_g_PN_iKC = 0;
    }
    if(dev_array_neurongroup_g_PN_iKC!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_neurongroup_g_PN_iKC)
                );
        dev_array_neurongroup_g_PN_iKC = 0;
    }
    if(_array_neurongroup_h!=0)
    {
        delete [] _array_neurongroup_h;
        _array_neurongroup_h = 0;
    }
    if(dev_array_neurongroup_h!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_neurongroup_h)
                );
        dev_array_neurongroup_h = 0;
    }
    if(_array_neurongroup_i!=0)
    {
        delete [] _array_neurongroup_i;
        _array_neurongroup_i = 0;
    }
    if(dev_array_neurongroup_i!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_neurongroup_i)
                );
        dev_array_neurongroup_i = 0;
    }
    if(_array_neurongroup_lastspike!=0)
    {
        delete [] _array_neurongroup_lastspike;
        _array_neurongroup_lastspike = 0;
    }
    if(dev_array_neurongroup_lastspike!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_neurongroup_lastspike)
                );
        dev_array_neurongroup_lastspike = 0;
    }
    if(_array_neurongroup_m!=0)
    {
        delete [] _array_neurongroup_m;
        _array_neurongroup_m = 0;
    }
    if(dev_array_neurongroup_m!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_neurongroup_m)
                );
        dev_array_neurongroup_m = 0;
    }
    if(_array_neurongroup_n!=0)
    {
        delete [] _array_neurongroup_n;
        _array_neurongroup_n = 0;
    }
    if(dev_array_neurongroup_n!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_neurongroup_n)
                );
        dev_array_neurongroup_n = 0;
    }
    if(_array_neurongroup_not_refractory!=0)
    {
        delete [] _array_neurongroup_not_refractory;
        _array_neurongroup_not_refractory = 0;
    }
    if(dev_array_neurongroup_not_refractory!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_neurongroup_not_refractory)
                );
        dev_array_neurongroup_not_refractory = 0;
    }
    if(_array_neurongroup_V!=0)
    {
        delete [] _array_neurongroup_V;
        _array_neurongroup_V = 0;
    }
    if(dev_array_neurongroup_V!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_neurongroup_V)
                );
        dev_array_neurongroup_V = 0;
    }
    if(_array_spikegeneratorgroup__lastindex!=0)
    {
        delete [] _array_spikegeneratorgroup__lastindex;
        _array_spikegeneratorgroup__lastindex = 0;
    }
    if(dev_array_spikegeneratorgroup__lastindex!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_spikegeneratorgroup__lastindex)
                );
        dev_array_spikegeneratorgroup__lastindex = 0;
    }
    if(_array_spikegeneratorgroup__period_bins!=0)
    {
        delete [] _array_spikegeneratorgroup__period_bins;
        _array_spikegeneratorgroup__period_bins = 0;
    }
    if(dev_array_spikegeneratorgroup__period_bins!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_spikegeneratorgroup__period_bins)
                );
        dev_array_spikegeneratorgroup__period_bins = 0;
    }
    if(_array_spikegeneratorgroup_i!=0)
    {
        delete [] _array_spikegeneratorgroup_i;
        _array_spikegeneratorgroup_i = 0;
    }
    if(dev_array_spikegeneratorgroup_i!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_spikegeneratorgroup_i)
                );
        dev_array_spikegeneratorgroup_i = 0;
    }
    if(_array_spikegeneratorgroup_period!=0)
    {
        delete [] _array_spikegeneratorgroup_period;
        _array_spikegeneratorgroup_period = 0;
    }
    if(dev_array_spikegeneratorgroup_period!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_spikegeneratorgroup_period)
                );
        dev_array_spikegeneratorgroup_period = 0;
    }
    if(_array_spikemonitor_1__source_idx!=0)
    {
        delete [] _array_spikemonitor_1__source_idx;
        _array_spikemonitor_1__source_idx = 0;
    }
    if(dev_array_spikemonitor_1__source_idx!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_spikemonitor_1__source_idx)
                );
        dev_array_spikemonitor_1__source_idx = 0;
    }
    if(_array_spikemonitor_1_count!=0)
    {
        delete [] _array_spikemonitor_1_count;
        _array_spikemonitor_1_count = 0;
    }
    if(dev_array_spikemonitor_1_count!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_spikemonitor_1_count)
                );
        dev_array_spikemonitor_1_count = 0;
    }
    if(_array_spikemonitor_1_N!=0)
    {
        delete [] _array_spikemonitor_1_N;
        _array_spikemonitor_1_N = 0;
    }
    if(dev_array_spikemonitor_1_N!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_spikemonitor_1_N)
                );
        dev_array_spikemonitor_1_N = 0;
    }
    if(_array_spikemonitor_2__source_idx!=0)
    {
        delete [] _array_spikemonitor_2__source_idx;
        _array_spikemonitor_2__source_idx = 0;
    }
    if(dev_array_spikemonitor_2__source_idx!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_spikemonitor_2__source_idx)
                );
        dev_array_spikemonitor_2__source_idx = 0;
    }
    if(_array_spikemonitor_2_count!=0)
    {
        delete [] _array_spikemonitor_2_count;
        _array_spikemonitor_2_count = 0;
    }
    if(dev_array_spikemonitor_2_count!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_spikemonitor_2_count)
                );
        dev_array_spikemonitor_2_count = 0;
    }
    if(_array_spikemonitor_2_N!=0)
    {
        delete [] _array_spikemonitor_2_N;
        _array_spikemonitor_2_N = 0;
    }
    if(dev_array_spikemonitor_2_N!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_spikemonitor_2_N)
                );
        dev_array_spikemonitor_2_N = 0;
    }
    if(_array_spikemonitor__source_idx!=0)
    {
        delete [] _array_spikemonitor__source_idx;
        _array_spikemonitor__source_idx = 0;
    }
    if(dev_array_spikemonitor__source_idx!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_spikemonitor__source_idx)
                );
        dev_array_spikemonitor__source_idx = 0;
    }
    if(_array_spikemonitor_count!=0)
    {
        delete [] _array_spikemonitor_count;
        _array_spikemonitor_count = 0;
    }
    if(dev_array_spikemonitor_count!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_spikemonitor_count)
                );
        dev_array_spikemonitor_count = 0;
    }
    if(_array_spikemonitor_N!=0)
    {
        delete [] _array_spikemonitor_N;
        _array_spikemonitor_N = 0;
    }
    if(dev_array_spikemonitor_N!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_spikemonitor_N)
                );
        dev_array_spikemonitor_N = 0;
    }
    if(_array_synapses_1_N!=0)
    {
        delete [] _array_synapses_1_N;
        _array_synapses_1_N = 0;
    }
    if(dev_array_synapses_1_N!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_synapses_1_N)
                );
        dev_array_synapses_1_N = 0;
    }
    if(_array_synapses_2_N!=0)
    {
        delete [] _array_synapses_2_N;
        _array_synapses_2_N = 0;
    }
    if(dev_array_synapses_2_N!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_synapses_2_N)
                );
        dev_array_synapses_2_N = 0;
    }
    if(_array_synapses_N!=0)
    {
        delete [] _array_synapses_N;
        _array_synapses_N = 0;
    }
    if(dev_array_synapses_N!=0)
    {
        CUDA_SAFE_CALL(
                hipFree(dev_array_synapses_N)
                );
        dev_array_synapses_N = 0;
    }


    // static arrays
    if(_static_array__dynamic_array_spikegeneratorgroup__timebins!=0)
    {
        delete [] _static_array__dynamic_array_spikegeneratorgroup__timebins;
        _static_array__dynamic_array_spikegeneratorgroup__timebins = 0;
    }
    if(_static_array__dynamic_array_spikegeneratorgroup_neuron_index!=0)
    {
        delete [] _static_array__dynamic_array_spikegeneratorgroup_neuron_index;
        _static_array__dynamic_array_spikegeneratorgroup_neuron_index = 0;
    }
    if(_static_array__dynamic_array_spikegeneratorgroup_spike_number!=0)
    {
        delete [] _static_array__dynamic_array_spikegeneratorgroup_spike_number;
        _static_array__dynamic_array_spikegeneratorgroup_spike_number = 0;
    }
    if(_static_array__dynamic_array_spikegeneratorgroup_spike_time!=0)
    {
        delete [] _static_array__dynamic_array_spikegeneratorgroup_spike_time;
        _static_array__dynamic_array_spikegeneratorgroup_spike_time = 0;
    }

}

