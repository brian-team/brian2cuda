#include "hip/hip_runtime.h"

#include "brianlib/cuda_utils.h"
#include "objects.h"
#include "network.h"
#include <stdlib.h>
#include <iostream>
#include <ctime>
#include <utility>
#include <stdio.h>
#include <assert.h>

#define Clock_epsilon 1e-14

double Network::_last_run_time = 0.0;
double Network::_last_run_completed_fraction = 0.0;

Network::Network()
{
    t = 0.0;
}

void Network::clear()
{
    objects.clear();
}

void Network::add(Clock *clock, codeobj_func func)
{
#if defined(_MSC_VER) && (_MSC_VER>=1700)
    objects.push_back(std::make_pair(std::move(clock), std::move(func)));
#else
    objects.push_back(std::make_pair(clock, func));
#endif
}

void Network::run(const double duration, void (*report_func)(const double, const double, const double, const double), const double report_period)
{
    std::clock_t start, current;
    const double t_start = t;
    const double t_end = t + duration;
    double next_report_time = report_period;
    // compute the set of clocks
    compute_clocks();
    // set interval for all clocks

    for(std::set<Clock*>::iterator i=clocks.begin(); i!=clocks.end(); i++)
        (*i)->set_interval(t, t_end);

    start = std::clock();
    if (report_func)
    {
        report_func(0.0, 0.0, t_start, duration);
    }

    Clock* clock = next_clocks();
    double elapsed_realtime;
    bool did_break_early = false;

    while(clock && clock->running())
    {
        t = clock->t[0];

        for(int i=0; i<objects.size(); i++)
        {
            if (report_func)
            {
                current = std::clock();
                const double elapsed = (double)(current - start) / CLOCKS_PER_SEC;
                if (elapsed > next_report_time)
                {
                    report_func(elapsed, (clock->t[0]-t_start)/duration, t_start, duration);
                    next_report_time += report_period;
                }
            }
            Clock *obj_clock = objects[i].first;
            // Only execute the object if it uses the right clock for this step
            if (curclocks.find(obj_clock) != curclocks.end())
            {
                codeobj_func func = objects[i].second;
                if (func)  // code objects can be NULL in cases where we store just the clock
                {
                    func();
                }
            }
        }
        for(std::set<Clock*>::iterator i=curclocks.begin(); i!=curclocks.end(); i++)
            (*i)->tick();
        clock = next_clocks();

        // Advance index for circular eventspace vector (for no_or_const_delay_mode)
        brian::current_idx_array_neurongroup_1__spikespace = (brian::current_idx_array_neurongroup_1__spikespace + 1) % brian::dev_array_neurongroup_1__spikespace.size();
        brian::current_idx_array_neurongroup__spikespace = (brian::current_idx_array_neurongroup__spikespace + 1) % brian::dev_array_neurongroup__spikespace.size();
        brian::previous_idx_array_spikegeneratorgroup__spikespace = brian::current_idx_array_spikegeneratorgroup__spikespace;
        brian::current_idx_array_spikegeneratorgroup__spikespace = (brian::current_idx_array_spikegeneratorgroup__spikespace + 1) % brian::dev_array_spikegeneratorgroup__spikespace.size();

        current = std::clock();
        elapsed_realtime = (double)(current - start)/CLOCKS_PER_SEC;


    }

    if(!did_break_early) t = t_end;

    _last_run_time = elapsed_realtime;
    if(duration>0)
    {
        _last_run_completed_fraction = (t-t_start)/duration;
    } else {
        _last_run_completed_fraction = 1.0;
    }
    if (report_func)
    {
        report_func(elapsed_realtime, 1.0, t_start, duration);
    }
}

void Network::compute_clocks()
{
    clocks.clear();
    for(int i=0; i<objects.size(); i++)
    {
        Clock *clock = objects[i].first;
        clocks.insert(clock);
    }
}

Clock* Network::next_clocks()
{
    // find minclock, clock with smallest t value
    Clock *minclock = *clocks.begin();
    if (!minclock) // empty list of clocks
        return NULL;

    for(std::set<Clock*>::iterator i=clocks.begin(); i!=clocks.end(); i++)
    {
        Clock *clock = *i;
        if(clock->t[0]<minclock->t[0])
            minclock = clock;
    }
    // find set of equal clocks
    curclocks.clear();

    double t = minclock->t[0];
    for(std::set<Clock*>::iterator i=clocks.begin(); i!=clocks.end(); i++)
    {
        Clock *clock = *i;
        double s = clock->t[0];
        if(s==t || fabs(s-t)<=Clock_epsilon)
            curclocks.insert(clock);
    }
    return minclock;
}

