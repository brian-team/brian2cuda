#include "hip/hip_runtime.h"
{% macro cu_file() %}

#include "objects.h"
#include "rand.h"
#include "synapses_classes.h"
#include "brianlib/clocks.h"
#include "brianlib/cuda_utils.h"
#include "network.h"
#include <hiprand.h>
#include <ctime>
#include <hiprand/hiprand_kernel.h>

// XXX: for some documentation on random number generation, check out our wiki:
//      https://github.com/brian-team/brian2cuda/wiki/Random-number-generation

using namespace brian;

// TODO make this a class member function
// TODO don't call one kernel per codeobject but instead on kernel which takes
//      care of all codeobjects, preferably called with as many threads/blocks
//      as necessary for all states and initializing in parallel with warp
//      level divergence [needs changing set_curand_device_api_states()]
namespace {

    __global__ void init_curand_states(int N, int sequence_offset)
    {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (idx < N)
        {
            // Each thread gets the same seed, a different sequence number and
            // no offset
            // TODO: different seed and 0 sequence number is much faster, with
            // less security for independent sequences, add option as
            // preference!
            //hiprand_init(curand_seed + idx, 0, 0,
            hiprand_init(
                    *d_curand_seed,          // seed
                    sequence_offset + idx,   // sequence number
                    0,                       // offset
                    &d_curand_states[idx]);
        }
    }
}


// need a function pointer for Network::add(), can't pass a pointer to a class
// method, which is of different type. Random number buffer runs in default
// stream always, the `stream` parameter is not used.
void _run_random_number_buffer(hipStream_t stream)
{
    // random_number_buffer is a RandomNumberBuffer instance, declared in objects.cu
    random_number_buffer.next_time_step();
}


void RandomNumberBuffer::init()
{
    // check that we have enough memory available
    size_t free_byte;
    size_t total_byte;
    CUDA_SAFE_CALL(
            hipMemGetInfo(&free_byte, &total_byte)
            );
    // TODO: This assumes all random number have randomNumber_t type, but poisson
    //       objects have different type
    size_t num_free_floats = free_byte / sizeof(randomNumber_t);

    {% for run_i in range(number_run_calls) %}
    if (run_counter == {{run_i}})
    {
        // number of time steps each codeobject is executed during current Network::run() call
        // XXX: we are assuming here that this function is only run in the first time step of a Network::run()
        {# Loop over all codeobjects in {{run_i}} which use random numbers. Therefore,
           concatenate the lists of codeobjects for each RNG type together (sum), make
           the list unique and sort it by codeobject name #}
        {% for co in codeobjects_with_rng_per_run[run_i].values()
            | sum(start=[])
            | unique(attribute='name')
            | sort(attribute='name') %}
        int64_t num_steps_this_run_{{co.name}} = {{co.owner.clock.name}}.i_end - *{{co.owner.clock.name}}.timestep;
        {% endfor %}

        {% set rng_types = codeobjects_with_rng_per_run[run_i].keys() %}
        {% for type in rng_types %}
        {# no binomial, it doesn't use a buffer but on the fly rng #}

        {% for co in codeobjects_with_rng_per_run[run_i][type] | sort(attribute='name') %}
        {# TODO: pass isinstance to Jinja template to make it available here #}
        {% if co.owner.__class__.__name__ == 'Synapses' %}
        {% set N = '_array_' + co.owner.name + '_N[0]' %}
        {% else %}
        {% set N = co.owner._N %}
        {% endif %}
        // Get the number of needed random numbers per clock cycle, the generation interval, and the number generated per hiprand call.
        num_per_cycle_{{type}}_{{co.name}} = {{N}} * {{co.rng_calls[type]}};
        {{type}}_floats_per_obj_{{co.name}} = floats_per_obj;
        if (floats_per_obj < num_per_cycle_{{type}}_{{co.name}})
            {{type}}_floats_per_obj_{{co.name}} = num_per_cycle_{{type}}_{{co.name}};
        {{type}}_interval_{{co.name}} = (int)({{type}}_floats_per_obj_{{co.name}} / num_per_cycle_{{type}}_{{co.name}});
        num_per_gen_{{type}}_{{co.name}} = num_per_cycle_{{type}}_{{co.name}} * {{type}}_interval_{{co.name}};
        idx_{{type}}_{{co.name}} = {{type}}_interval_{{co.name}};

        // create max as many random numbers as will be needed during the current Network.run() call
        if ((int64_t){{type}}_interval_{{co.name}} > num_steps_this_run_{{co.name}})
        {
            // NOTE: if the conditional is true, we can savely cast num_steps_this_run_{{co.name}} to int
            num_per_gen_{{type}}_{{co.name}} = num_per_cycle_{{type}}_{{co.name}} * (int)num_steps_this_run_{{co.name}};
            assert((int64_t)num_per_cycle_{{type}}_{{co.name}} * num_steps_this_run_{{co.name}} == num_per_gen_{{type}}_{{co.name}});
            {{type}}_interval_{{co.name}} = (int)num_steps_this_run_{{co.name}};
            // set this for buffer to be refilled at first next_time_step() call
            idx_{{type}}_{{co.name}} = {{type}}_interval_{{co.name}};
        }

        // hiprandGenerateNormal requires an even number for pseudorandom generators
        if (num_per_gen_{{type}}_{{co.name}} % 2 != 0)
        {
            num_per_gen_{{type}}_{{co.name}} = num_per_gen_{{type}}_{{co.name}} + 1;
        }

        // make sure that we don't use more memory then available
        // this checks per codeobject the number of generated floats against total available floats
        while (num_free_floats < num_per_gen_{{type}}_{{co.name}})
        {
            printf("INFO not enough memory available to generate %i random numbers for {{co.name}}, reducing the buffer size\n", num_free_floats);
            if (num_per_gen_{{type}}_{{co.name}} < num_per_cycle_{{type}}_{{co.name}})
            {
                if (num_free_floats < num_per_cycle_{{type}}_{{co.name}})
                {
                    printf("ERROR not enough memory to generate random numbers for {{co.name}} %s:%d\n", __FILE__, __LINE__);
                    _dealloc_arrays();
                    exit(1);
                }
                else
                {
                    num_per_gen_{{type}}_{{co.name}} = num_per_cycle_{{type}}_{{co.name}};
                    break;
                }
            }
            num_per_gen_{{type}}_{{co.name}} /= 2;
        }
        printf("INFO generating %i {{type}} every %i clock cycles for {{co.name}}\n", num_per_gen_{{type}}_{{co.name}}, {{type}}_interval_{{co.name}});

        {% if type in ['rand', 'randn'] %}
        {% set dtype = "randomNumber_t" %}
        {% else %}
        {% set dtype = "unsigned int" %}
        {% endif %}

        CUDA_SAFE_CALL(
                hipMalloc((void**)&dev_{{co.name}}_{{type}}_allocator, sizeof({{dtype}})*num_per_gen_{{type}}_{{co.name}})
                );
        {% endfor %}{# for co in codeobjects_with_rng_per_run[run_i][type] #}
        {% endfor %}{# for type in rng_types #}

        // now check if the total number of generated floats fit into available memory
        int total_num_generated_floats = 0;
        {% for type in rng_types %}
        {% for co in codeobjects_with_rng_per_run[run_i][type] | sort(attribute='name') %}
        total_num_generated_floats += num_per_gen_{{type}}_{{co.name}};
        {% endfor %}
        {% endfor %}
        if (num_free_floats < total_num_generated_floats)
        {
            // TODO: find a way to deal with this? E.g. looping over buffers sorted
            // by buffer size and reducing them until it fits.
            printf("MEMORY ERROR: Trying to generate more random numbers than fit "
                   "into available memory. Please report this as an issue on "
                   "GitHub: https://github.com/brian-team/brian2cuda/issues/new");
            _dealloc_arrays();
            exit(1);
        }

    } // if (run_counter == {{run_i}})
    {% endfor %}{# run_i #}

    // init hiprand states only in first run
    if (run_counter == 0)
    {

        // Update hiprand device api states once before anything is run. At this
        // point all N's (also from probabilistically generated synapses) are
        // known. This might update the number of needed hiprand states.
        ensure_enough_curand_states();
    }

}


void RandomNumberBuffer::allocate_device_curand_states()
{
    // allocate globabl memory for hiprand device api states
    CUDA_SAFE_CALL(
            hipMalloc((void**)&dev_curand_states,
                sizeof(hiprandState) * num_curand_states)
            );
    CUDA_SAFE_CALL(
            hipMemcpyToSymbol(HIP_SYMBOL(d_curand_states),
                &dev_curand_states, sizeof(hiprandState*))
            );
}



void RandomNumberBuffer::update_needed_number_curand_states()
{
    // Find the maximum number of threads generating random numbers in parallel
    // using the cuRAND device API. For synapses objects, the number of
    // synapses might not be known yet. This is the case when the first random
    // seed is set and for any seed() call before the synapses creation.
    {% for co_name, (N_ptr, N_value) in needed_number_curand_states.items() %}

    // codeobject with binomial or poisson with variable lambda: {{co_name}}
    {# Only test for null pointer if N_ptr is a pointer (for Synapses) #}
    {% if N_ptr %}
    // test if synapses are already created (else this is a NULL pointer)
    if ({{N_ptr}})
    {
    {% endif %}{# if N_ptr #}

        if (num_curand_states < {{N_value}})
            num_curand_states = {{N_value}};

    {% if N_ptr %}
    }
    {% endif %}{# if N_ptr #}

    {% endfor %}{# for co_name, (N_ptr, N_value) #}
    num_threads_curand_init = max_threads_per_block;
    num_blocks_curand_init = num_curand_states / max_threads_per_block + 1;
    if (num_curand_states < num_threads_curand_init)
        num_threads_curand_init = num_curand_states;
}


void RandomNumberBuffer::set_curand_device_api_states(bool reset_seed)
{
    int sequence_offset = 0;
    int num_curand_states_old = num_curand_states;
    // Whenever hiprand states are set, check if enough states where
    // initialized. This will generate states the first time the seed is set.
    // But it can be that the seed is set before all network objects' N are
    // available (e.g. synapses not created yet) and before the network is
    // run. In such a case, once the network is run, missing hiprand states are
    // generated here. If the seed was not reset inbetween, the pervious states
    // should not be reinitialized (achieved by the `sequence_offset`
    // parameter). If the seed was reset, then all states should be
    // reinitialized.
    update_needed_number_curand_states();

    // number of hiprand states that need to be initialized
    int num_curand_states_to_init;

    if (reset_seed)
    {
        // initialize all hiprand states
        num_curand_states_to_init = num_curand_states;
        sequence_offset = 0;
    }
    else
    {
        // don't initialize existing hiprand states, only the new ones
        num_curand_states_to_init = num_curand_states - num_curand_states_old;
        sequence_offset = num_curand_states_old;
    }

    if (num_curand_states_old < num_curand_states)
    {
        // copy hiprand states to new array of updated size
        hiprandState* dev_curand_states_old = dev_curand_states;
        // allocate memory for new number of hiprand states
        allocate_device_curand_states();

        if ((!reset_seed) && (num_curand_states_old > 0))
        {
            // copy old states to new memory address on device
            CUDA_SAFE_CALL(
                    hipMemcpy(dev_curand_states, dev_curand_states_old,
                        sizeof(hiprandState) * num_curand_states_old,
                        hipMemcpyDeviceToDevice)
                    );
        }
    }

    if (num_curand_states_to_init > 0)
    {
        init_curand_states<<<num_blocks_curand_init, num_threads_curand_init>>>(
                num_curand_states_to_init,
                sequence_offset);
    }
}


void RandomNumberBuffer::ensure_enough_curand_states()
{
    // Separate public function needed for synapses codeobjects that are run
    // only once before the network
    // The N of synapses will not be known when setting the seed and needs to
    // be updated before using random numbers per synapse. This occurs e.g.
    // when initializing synaptic variables (synapses_group_conditional_....)
    bool reset_seed = false;
    set_curand_device_api_states(reset_seed);
}


void RandomNumberBuffer::run_finished()
{
    needs_init = true;
    run_counter += 1;
}


void RandomNumberBuffer::set_seed(unsigned long long seed)
{
    CUDA_SAFE_CALL(
            hiprandSetPseudoRandomGeneratorSeed(curand_generator, seed)
            );

    // generator offset needs to be reset to its default (=0)
    CUDA_SAFE_CALL(
            hiprandSetGeneratorOffset(curand_generator, 0ULL)
            );

    // set seed for hiprand device api calls
    // don't set the same seed for host api and device api random states, just in case
    unsigned long long curand_seed = seed + 1;
    CUDA_SAFE_CALL(
            hipMemcpy(dev_curand_seed, &curand_seed,
                sizeof(unsigned long long), hipMemcpyHostToDevice)
            );

    bool reset_seed = true;
    set_curand_device_api_states(reset_seed);
    // We set all device api states for codeobjects run outside the network
    // since we don't know when they will be used.
    //set_curand_device_api_states_for_separate_calls();
    // Curand device api states for binomials during network runs will be set
    // only for the current run in init(), once the network starts.
}


void RandomNumberBuffer::refill_uniform_numbers(
        randomNumber_t* dev_rand_allocator,
        randomNumber_t* &dev_rand,
        int num_per_gen_rand,
        int &idx_rand)
{
    // generate uniform distributed random numbers and reset buffer index

    {% if curand_float_type == 'float' %}
    hiprandGenerateUniform(curand_generator, dev_rand_allocator, num_per_gen_rand);
    {% else %}
    hiprandGenerateUniformDouble(curand_generator, dev_rand_allocator, num_per_gen_rand);
    {% endif %}
    // before: XXX dev_rand = &dev_rand_allocator[0];
    dev_rand = dev_rand_allocator;
    idx_rand = 1;
}


void RandomNumberBuffer::refill_normal_numbers(
        randomNumber_t* dev_randn_allocator,
        randomNumber_t* &dev_randn,
        int num_per_gen_randn,
        int &idx_randn)
{
    // generate normal distributed random numbers and reset buffer index

    {% if curand_float_type == 'float' %}
    hiprandGenerateNormal(curand_generator, dev_randn_allocator, num_per_gen_randn, 0, 1);
    {% else %}
    hiprandGenerateNormalDouble(curand_generator, dev_randn_allocator, num_per_gen_randn, 0, 1);
    {% endif %}
    // before: XXX dev_randn = &dev_randn_allocator[0];
    dev_randn = dev_randn_allocator;
    idx_randn = 1;
}


void RandomNumberBuffer::refill_poisson_numbers(
        double lambda,
        unsigned int* dev_poisson_allocator,
        unsigned int* &dev_poisson,
        int num_per_gen_poisson,
        int &idx_poisson)
{
    // generate poisson distributed random numbers and reset buffer index

    printf("num_per_gen_poisson %d, lambda %f\n", num_per_gen_poisson, lambda);
    CUDA_SAFE_CALL(
            hiprandGeneratePoisson(curand_generator, dev_poisson_allocator, num_per_gen_poisson, lambda)
            );
    dev_poisson = dev_poisson_allocator;
    idx_poisson = 1;
}

void RandomNumberBuffer::next_time_step()
{
    // init buffers at fist time step of each run call
    if (needs_init)
    {
        // free device memory for random numbers used during last run call
        if (run_counter > 0)
        {
            {% for run_i in range(number_run_calls) %}
            {% set rng_types = codeobjects_with_rng_per_run[run_i].keys() %}
            {% set need_cleanup = False %}
            {% for type in rng_types %}
            {% set need_cleanup = need_cleanup or codeobjects_with_rng_per_run[run_i][type] %}
            {% endfor %}
            {% if need_cleanup %}
            if (run_counter == {{run_i}})
            {
                {% for type in rng_types %}
                {% for co in codeobjects_with_rng_per_run[run_i][type] | sort(attribute='name') %}
                CUDA_SAFE_CALL(
                        hipFree(dev_{{co.name}}_{{type}}_allocator)
                        );
                {% endfor %}
                {% endfor %}
            } // run_counter == {{run_i}}
            {% endif %}{# need_cleanup #}
            {% endfor %}{# run_i #}
        }

        // init random number buffers
        init();
        needs_init = false;
    }

    {% for run_i in range(number_run_calls) %}
    if (run_counter == {{run_i}})
    {
        {% set rng_types = codeobjects_with_rng_per_run[run_i].keys() %}
        {% for type in rng_types %}
        {% for co in codeobjects_with_rng_per_run[run_i][type] | sort(attribute='name') %}

        // random numbers ({{type}}) for {{co.name}}
        if (idx_{{type}}_{{co.name}} == {{type}}_interval_{{co.name}})
        {
            {% if type == 'rand' %}
            refill_uniform_numbers(
            {% elif type == 'randn' %}
            refill_normal_numbers(
            {% else %}{# poisson distributions #}
            refill_poisson_numbers(
                    {{all_poisson_lamdas[co.name][type]}},
            {% endif %}
                    dev_{{co.name}}_{{type}}_allocator,
                    dev_{{co.name}}_{{type}},
                    num_per_gen_{{type}}_{{co.name}},
                    idx_{{type}}_{{co.name}});
        }
        else
        {
            // move device pointer to next numbers
            dev_{{co.name}}_{{type}} += num_per_cycle_{{type}}_{{co.name}};
            idx_{{type}}_{{co.name}} += 1;
        }
        assert(dev_{{co.name}}_{{type}} < dev_{{co.name}}_{{type}}_allocator + num_per_gen_{{type}}_{{co.name}});
        {% endfor %}{# type #}
        {% endfor %}{# co #}
    }// run_counter == {{run_i}}
    {% endfor %}{# run_i #}
}
{% endmacro %}

/////////////////////////////////////////////////////////////////////////////////////////////////////

{% macro h_file() %}

#ifndef _BRIAN_RAND_H
#define _BRIAN_RAND_H

#include <hiprand.h>

void _run_random_number_buffer(hipStream_t);

class RandomNumberBuffer
{
    // TODO let all random number pointers be class members of this class ->
    //      check which ones are needed as global variables, maybe have both,
    //      global and member variables? or change parameters in codeobjects?

    // before each run, buffers need to be reinitialized
    bool needs_init = true;
    // how many 'run' calls have finished
    int run_counter = 0;
    // number of needed cuRAND states
    int num_curand_states = 0;
    // number of threads and blocks to set hiprand states
    int num_threads_curand_init, num_blocks_curand_init;

    // how many random numbers we want to create at once (tradeoff memory usage <-> generation overhead)
    double mb_per_obj = 50;  // MB per codeobject and rand / randn
    // TODO: This assumes all random number have randomNumber_t type, but poisson
    //       objects have different type
    int floats_per_obj = (mb_per_obj * 1024.0 * 1024.0) / sizeof(randomNumber_t);

    // The number of needed random numbers per clock cycle, the generation interval, and the number generated per hiprand call.
    //
    // needed random numbers per clock cycle
    // int num_per_cycle_rand_{};
    //
    // number of time steps after which buffer needs to be refilled
    // int rand_interval_{};
    //
    // buffer size
    // int num_per_gen_rand_{};
    //
    // number of time steps since last buffer refill
    // int idx_rand_{};
    //
    // maximum number of random numbers fitting given allocated memory
    // int rand_floats_per_obj_{};

    // For each call of brians `run`, a new set of codeobjects (with different
    // suffixes) is generated. The following are variables for all codeobjects
    // for all runs that need random numbers.

    {% for run_i in range(number_run_calls) %}
    ////// run {{run_i}}

    {% set rng_types = codeobjects_with_rng_per_run[run_i].keys() %}
    {% for type in rng_types %}

    {% if codeobjects_with_rng_per_run[run_i][type]|length > 0 %}
    {% if type == 'rand' %}
    //// uniform distributed random numbers (rand)
    {% elif type == 'randn' %}
    //// normal distributed random numbers (randn)
    {% else %}
    //// poisson distributed random number ({{type}})
    {% endif %}
    {% endif %}

    {% for co in codeobjects_with_rng_per_run[run_i][type] | sort(attribute='name') %}

    // {{co.name}}
    int num_per_cycle_{{type}}_{{co.name}};
    int {{type}}_interval_{{co.name}};
    int num_per_gen_{{type}}_{{co.name}};
    int idx_{{type}}_{{co.name}};
    int {{type}}_floats_per_obj_{{co.name}};

    {% endfor %}{# co #}
    {% endfor %}{# type #}
    {% endfor %}{# run_i #}

    void init();
    void allocate_device_curand_states();
    void update_needed_number_curand_states();
    void set_curand_device_api_states(bool);
    void refill_uniform_numbers(randomNumber_t*, randomNumber_t*&, int, int&);
    void refill_normal_numbers(randomNumber_t*, randomNumber_t*&, int, int&);
    void refill_poisson_numbers(double lambda, unsigned int*, unsigned int*&, int, int&);

public:
    void next_time_step();
    void set_seed(unsigned long long);
    void run_finished();
    void ensure_enough_curand_states();
};

#endif

{% endmacro %}