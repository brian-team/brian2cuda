#include "hip/hip_runtime.h"
{# USES_VARIABLES { Cm, dt, v, N, Ic, Ri,
                  _ab_star0, _ab_star1, _ab_star2, _b_plus, _b_minus,
                  _v_star, _u_plus, _u_minus,
                  _v_previous,
                  _gtot_all, _I0_all,
                  _c,
                  _P_diag, _P_parent, _P_children,
                  _B, _morph_parent_i, _starts, _ends,
                  _morph_children, _morph_children_num, _morph_idxchild,
                  _invr0, _invrn, _invr,
                  r_length_1, r_length_2, area } #}


{% extends 'common_group.cu' %}


{### BEFORE RUN ###}
{% block before_run_host_maincode %}

    ///// HOST_CONSTANTS ///////////
    %HOST_CONSTANTS%

    // Number of group units in stateupdate is always N (no subgroups)
    const int _N = N;

    ///// ADDITIONAL_HOST_CODE /////
    %ADDITIONAL_HOST_CODE%

    {# needed to translate _array... to _ptr_array... #}
    ///// pointers_lines /////
    {{pointers_lines|autoindent}}

    // The following code is simply copied from spatialneuron_prepare.cpp
    // of the cpp_standalone device (except for copying to GPU memory at bottom of file)

    const double _Ri = {{Ri}};  // Ri is a shared variable

    // Inverse axial resistance
    {# {{ openmp_pragma('parallel-static') }} #}
    for (int _i=1; _i<_N; _i++)
        {{_invr}}[_i] = 1.0/(_Ri*(1/{{r_length_2}}[_i-1] + 1/{{r_length_1}}[_i]));
    // Cut sections
    {# {{ openmp_pragma('parallel-static') }} #}
    for (int _i=0; _i<_num_starts; _i++)
        {{_invr}}[{{_starts}}[_i]] = 0;

    // Linear systems
    // The particular solution
    // a[i,j]=ab[u+i-j,j]   --  u is the number of upper diagonals = 1
    {# {{ openmp_pragma('parallel-static') }} #}
    for (int _i=0; _i<_N; _i++)
        {{_ab_star1}}[_i] = (-({{Cm}}[_i] / {{dt}}) - {{_invr}}[_i] / {{area}}[_i]);
    {# {{ openmp_pragma('parallel-static') }} #}
    for (int _i=1; _i<_N; _i++)
    {
        {{_ab_star0}}[_i] = {{_invr}}[_i] / {{area}}[_i-1];
        {{_ab_star2}}[_i-1] = {{_invr}}[_i] / {{area}}[_i];
        {{_ab_star1}}[_i-1] -= {{_invr}}[_i] / {{area}}[_i-1];
    }

    // Set the boundary conditions
    for (int _counter=0; _counter<_num_starts; _counter++)
    {
        const int _first = {{_starts}}[_counter];
        const int _last = {{_ends}}[_counter] - 1;  // the compartment indices are in the interval [starts, ends[
        // Inverse axial resistances at the ends: r0 and rn
        const double _invr0 = {{r_length_1}}[_first]/_Ri;
        const double _invrn = {{r_length_2}}[_last]/_Ri;
        {{_invr0}}[_counter] = _invr0;
        {{_invrn}}[_counter] = _invrn;
        // Correction for boundary conditions
        {{_ab_star1}}[_first] -= (_invr0 / {{area}}[_first]);
        {{_ab_star1}}[_last] -= (_invrn / {{area}}[_last]);
        // RHS for homogeneous solutions
        {{_b_plus}}[_last] = -(_invrn / {{area}}[_last]);
        {{_b_minus}}[_first] = -(_invr0 / {{area}}[_first]);
    }

    // Copy prepared arrays to GPU
    {% for var in ['_invr', 'Ri', 'Cm', 'dt', 'area', 'r_length_1',
                       'r_length_2', '_ab_star0', '_ab_star1', '_ab_star2',
                       '_starts', '_ends', '_invr0', '_invrn', '_b_plus',
                       '_b_minus'] %}
    {% set varname = get_array_name(variables[var], access_data=False) %}

    // {{var}}
    CUDA_SAFE_CALL(
            hipMemcpy(dev{{varname}}, {{varname}},
                sizeof({{c_data_type(variables[var].dtype)}})*_num_{{varname}},
                hipMemcpyHostToDevice)
            );
    {% endfor %}
{% endblock before_run_host_maincode %}


{### RUN ###}
/////////////////////////////////////////////////////
/////////////////////////////////////////////////////
// FIRST: KERNEL DEFINITIONS
/////////////////////////////////////////////////////
/////////////////////////////////////////////////////

/////////////////////////////////////////////////////
// kernel 1: compute g_total and I_0
// (independent: everything, i.e., compartments and branches)
// remark: for this kernel we use the common_group.cu to have the machinery of optimal thread
//         no calculation machinery
{% block extra_vector_code %}
        {{_gtot_all}}[_idx] = _gtot;
        {{_I0_all}}[_idx] = _I0;

        {{_v_previous}}[_idx] = {{v}}[_idx];
{% endblock %}

// additional kernels (linear systems and solution combination)
{% block extra_device_helper %}

/////////////////////////////////////////////////////
// kernel 2: solve three tridiagonal system (one matrix of size compartment with three right hand sides)
// (independent: branches)
// remark: here we apply over the branches in parallel the Thomas algorithm
//         (i.e., Gaussian elimination for a tridiagonal system) which has a
//         runtime complexity O(compartments) but is inherently sequential
//         => run no as many blocks as branches with one thread each
//         => trivial optimization possible by using three threads (one per rhs)
//         => optimization possible e.g. by using cyclic reduction [more parallel]

__global__ void _tridiagsolve_kernel_{{codeobj_name}}(
    int _N,
    ///// KERNEL_PARAMETERS /////
    %KERNEL_PARAMETERS%
    )
{
    using namespace brian;

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int _idx = bid * blockDim.x + tid;

    ///// KERNEL_CONSTANTS /////
    %KERNEL_CONSTANTS%

    ///// kernel_lines /////
    {{kernel_lines|autoindent}}

    // we need to run the kernel with 1 thread per block (to be changed by optimization)
    assert(tid == 0 && bid == _idx);

    // each thread processes the tridiagsystem of one branch
    const int _i = _idx;

    // below all the code is simply copied from spatialstateupdate.cpp

    // first and last index of the i-th section
    const int _j_start = {{_starts}}[_i];
    const int _j_end = {{_ends}}[_i];

    double _ai, _bi, _m; // helper variables

    // upper triangularization of tridiagonal system for _v_star, _u_plus, and _u_minus
    for(int _j=_j_start; _j<_j_end; _j++)
    {
        {{_v_star}}[_j]=-({{Cm}}[_j]/{{dt}}*{{v}}[_j])-{{_I0_all}}[_j]; // RHS -> _v_star (solution)
        {{_u_plus}}[_j]={{_b_plus}}[_j]; // RHS -> _u_plus (solution)
        {{_u_minus}}[_j]={{_b_minus}}[_j]; // RHS -> _u_minus (solution)
        _bi={{_ab_star1}}[_j]-{{_gtot_all}}[_j]; // main diagonal
        if (_j<_N-1)
            {{_c}}[_j]={{_ab_star0}}[_j+1]; // superdiagonal
        if (_j>0)
        {
            _ai={{_ab_star2}}[_j-1]; // subdiagonal
            _m=1.0/(_bi-_ai*{{_c}}[_j-1]);
            {{_c}}[_j]={{_c}}[_j]*_m;
            {{_v_star}}[_j]=({{_v_star}}[_j] - _ai*{{_v_star}}[_j-1])*_m;
            {{_u_plus}}[_j]=({{_u_plus}}[_j] - _ai*{{_u_plus}}[_j-1])*_m;
            {{_u_minus}}[_j]=({{_u_minus}}[_j] - _ai*{{_u_minus}}[_j-1])*_m;
        } else
        {
            {{_c}}[0]={{_c}}[0]/_bi;
            {{_v_star}}[0]={{_v_star}}[0]/_bi;
            {{_u_plus}}[0]={{_u_plus}}[0]/_bi;
            {{_u_minus}}[0]={{_u_minus}}[0]/_bi;
        }
    }
    // backwards substituation of the upper triangularized system for _v_star
    for(int _j=_j_end-2; _j>=_j_start; _j--)
    {
        {{_v_star}}[_j]={{_v_star}}[_j] - {{_c}}[_j]*{{_v_star}}[_j+1];
        {{_u_plus}}[_j]={{_u_plus}}[_j] - {{_c}}[_j]*{{_u_plus}}[_j+1];
        {{_u_minus}}[_j]={{_u_minus}}[_j] - {{_c}}[_j]*{{_u_minus}}[_j+1];
    }
}



/////////////////////////////////////////////////////
// kernel 3: solve the coupling system (one matrix of size branches)
// (no independence)
// remark: applies the Hines algorithm having O(branches) complexity
//         => run with one block one thread

__global__ void _coupling_kernel_{{codeobj_name}}(
    int _N,
    ///// KERNEL_PARAMETERS /////
    %KERNEL_PARAMETERS%
    )
{
    using namespace brian;

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int _idx = bid * blockDim.x + tid;

    ///// KERNEL_CONSTANTS /////
    %KERNEL_CONSTANTS%

    ///// kernel_lines /////
    {{kernel_lines|autoindent}}

    // we need to run the kernel with 1 thread, 1 block
    assert(_idx == 0);

    // below all the code is simply copied from spatialstateupdate.cpp

        // indexing for _P_children which contains the elements above the diagonal of the coupling matrix _P
    const int _children_rowlength = _num_morph_children/_num_morph_children_num;
    #define _IDX_C(idx_row,idx_col) _children_rowlength * idx_row + idx_col

    // step a) construct the coupling system with matrix _P in sparse form. s.t.
    // _P_diag contains the diagonal elements
    // _P_children contains the super diagonal entries
    // _P_parent contains the single sub diagonal entry for each row
    // _B contains the right hand side
    for (int _i=0; _i<_num_B - 1; _i++)
    {
        const int _i_parent = {{_morph_parent_i}}[_i];
        const int _i_childind = {{_morph_idxchild}}[_i];
        const int _first = {{_starts}}[_i];
        const int _last = {{_ends}}[_i] - 1;  // the compartment indices are in the interval [starts, ends[
        const double _invr0 = {{_invr0}}[_i];
        const double _invrn = {{_invrn}}[_i];

        // Towards parent
        if (_i == 0) // first section, sealed end
        {
            // sparse matrix version
            {{_P_diag}}[0] = {{_u_minus}}[_first] - 1;
            {{_P_children}}[_IDX_C(0,0)] = {{_u_plus}}[_first];

            // RHS
            {{_B}}[0] = -{{_v_star}}[_first];
        }
        else
        {
            // sparse matrix version
            {{_P_diag}}[_i_parent] += (1 - {{_u_minus}}[_first]) * _invr0;
            {{_P_children}}[_IDX_C(_i_parent, _i_childind)] = -{{_u_plus}}[_first] * _invr0;

            // RHS
            {{_B}}[_i_parent] += {{_v_star}}[_first] * _invr0;
        }

        // Towards children

        // sparse matrix version
        {{_P_diag}}[_i+1] = (1 - {{_u_plus}}[_last]) * _invrn;
        {{_P_parent}}[_i] = -{{_u_minus}}[_last] * _invrn;

        // RHS
        {{_B}}[_i+1] = {{_v_star}}[_last] * _invrn;
    }


    // step b) solve the linear system (the result will be stored in the former rhs _B in the end)
    // use efficient O(n) solution of the sparse linear system (structure-specific Gaussian elemination)

    // part 1: lower triangularization
    for (int _i=_num_B-1; _i>=0; _i--) {
        const int _num_children = {{_morph_children_num}}[_i];

        // for every child eliminate the corresponding matrix element of row i
        for (int _k=0; _k<_num_children; _k++) {
            int _j = {{_morph_children}}[_IDX_C(_i,_k)]; // child index

            // subtracting _subfac times the j-th from the i-th row
            double _subfac = {{_P_children}}[_IDX_C(_i,_k)] / {{_P_diag}}[_j]; // element i,j appears only here

            // the following commented (superdiagonal) element is not used in the following anymore since
            // it is 0 by definition of (lower) triangularization; we keep it here for algorithmic clarity
            //{{_P_children}}[_IDX_C(_i,_k)] = {{_P_children}}[_IDX_C(_i,_k)]  - _subfac * {{_P_diag}}[_j]; // = 0;

            {{_P_diag}}[_i] = {{_P_diag}}[_i]  - _subfac * {{_P_parent}}[_j-1]; // note: element j,i is only used here
            {{_B}}[_i] = {{_B}}[_i] - _subfac * {{_B}}[_j];

        }
    }

    // part 2: forwards substitution
    {{_B}}[0] = {{_B}}[0] / {{_P_diag}}[0]; // the first section does not have a parent
    for (int _i=1; _i<_num_B; _i++) {
        const int _j = {{_morph_parent_i}}[_i-1]; // parent index
        {{_B}}[_i] = {{_B}}[_i] - {{_P_parent}}[_i-1] * {{_B}}[_j];
        {{_B}}[_i] = {{_B}}[_i] / {{_P_diag}}[_i];

    }

}


/////////////////////////////////////////////////////
// kernel 4: for each section compute the final solution by linear
//           combination of the general solution
// (independent: everything, i.e., compartments and branches)
// remark: branch granularity in implementation used since parents/children are combined for each branch

__global__ void _combine_kernel_{{codeobj_name}}(
    int _N,
    ///// KERNEL_PARAMETERS /////
    %KERNEL_PARAMETERS%
    )
{
    using namespace brian;

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int _idx = bid * blockDim.x + tid;

    ///// KERNEL_CONSTANTS /////
    %KERNEL_CONSTANTS%

    ///// kernel_lines /////
    {{kernel_lines|autoindent}}

    // we need to run the kernel with 1 thread per block (to be changed by optimization)
    assert(tid == 0 && bid == _idx);

    // each thread combines the tridiagsystem of one branch
    const int _i = _idx;

    // below all the code is simply copied from spatialstateupdate.cpp

    const int _i_parent = {{_morph_parent_i}}[_i];
    const int _j_start = {{_starts}}[_i];
    const int _j_end = {{_ends}}[_i];
    for (int _j=_j_start; _j<_j_end; _j++)
        if (_j < _numv)  // don't go beyond the last element
            {{v}}[_j] = {{_v_star}}[_j] + {{_B}}[_i_parent] * {{_u_minus}}[_j]
                                       + {{_B}}[_i+1] * {{_u_plus}}[_j];


}



/////////////////////////////////////////////////////
// kernel 5: update currents
// (independent: everything, i.e., compartments and branches)

__global__ void _currents_kernel_{{codeobj_name}}(
    int _N,
    ///// KERNEL_PARAMETERS /////
    %KERNEL_PARAMETERS%
    )
{
    using namespace brian;

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int _idx = bid * blockDim.x + tid;

    ///// KERNEL_CONSTANTS /////
    %KERNEL_CONSTANTS%

    ///// kernel_lines /////
    {{kernel_lines|autoindent}}

    if(_idx >= _N)
    {
        return;
    }

    // each thread processes the tridiagsystem of one branch
    const int _i = _idx;

    {{Ic}}[_i] = {{Cm}}[_i]*({{v}}[_i] - {{_v_previous}}[_i])/{{dt}};

}



{% endblock extra_device_helper %}


/////////////////////////////////////////////////////
/////////////////////////////////////////////////////
// SECOND/LAST: KERNEL EXECUTIONS
/////////////////////////////////////////////////////
/////////////////////////////////////////////////////

{% block extra_kernel_call_post %}

    // kernel 1 is automatically run (via common_group.cu), particularly with full occupancy
{#
    {% if profiled %}
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    {{codeobj_name}}_kernel_integration_profiling_info += (double)(std::clock() -_start_time)/CLOCKS_PER_SEC;
    {% endif %}
#}

    // run kernel 2 (tridiag solve): branches many blocks with one thread each
{#
    {% if profiled %}
    std::clock_t _start_time_tridiagsolve = std::clock();
    {% endif %}
#}
    int num_blocks_tridiagsolve = _num_B-1;
    int num_threads_tridiagsolve = 1;
    _tridiagsolve_kernel_{{codeobj_name}}<<<num_blocks_tridiagsolve, num_threads_tridiagsolve>>>(
            _N,
            ///// HOST_PARAMETERS /////
            %HOST_PARAMETERS%
        );
    CUDA_CHECK_ERROR("_tridiagsolve_kernel_{{codeobj_name}}");
{#
    {% if profiled %}
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    {{codeobj_name}}_kernel_tridiagsolve_profiling_info += (double)(std::clock() -_start_time_tridiagsolve)/CLOCKS_PER_SEC;
    {% endif %}
#}

    // kernel 3 (coupling): one block one thread
{#
    {% if profiled %}
    std::clock_t _start_time_coupling = std::clock();
    {% endif %}
#}
    int num_blocks_coupling = 1;
    int num_threads_coupling = 1;
    _coupling_kernel_{{codeobj_name}}<<<num_blocks_coupling, num_threads_coupling>>>(
            _N,
            ///// HOST_PARAMETERS /////
            %HOST_PARAMETERS%
        );
    CUDA_CHECK_ERROR("_coupling_kernel_{{codeobj_name}}");
{#
    {% if profiled %}
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    {{codeobj_name}}_kernel_coupling_profiling_info += (double)(std::clock() -_start_time_coupling)/CLOCKS_PER_SEC;
    {% endif %}
#}

    // kernel 4 (combine): branches many blocks with one thread each
{#
    {% if profiled %}
    std::clock_t _start_time_combine = std::clock();
    {% endif %}
#}
    int num_blocks_combine = _num_B-1;
    int num_threads_combine = 1;
    _combine_kernel_{{codeobj_name}}<<<num_blocks_combine, num_threads_combine>>>(
            _N,
            ///// HOST_PARAMETERS /////
            %HOST_PARAMETERS%
        );
    CUDA_CHECK_ERROR("_combine_kernel_{{codeobj_name}}");
{#
    {% if profiled %}
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    {{codeobj_name}}_kernel_combine_profiling_info += (double)(std::clock() -_start_time_combine)/CLOCKS_PER_SEC;
    {% endif %}
#}

    // kernel 5 (final currents): max. occupancy

    // calculate max. occupancy => num_threads, num_blocks
    // first try to use it again

        static int num_threads_currents, num_blocks_currents;
        static bool first_run_custom = true;
        if (first_run_custom)
        {

            // calculate number of threads that maximize occupancy
            // and also the corresponding number of blocks
            // the code below is adapted from common_group.cu
            int min_num_threads_currents; // The minimum grid size needed to achieve the
                                 // maximum occupancy for a full device launch

            CUDA_SAFE_CALL(
                    hipOccupancyMaxPotentialBlockSize(&min_num_threads_currents, &num_threads_currents,
                        _currents_kernel_{{codeobj_name}}, 0, 0)  // last args: dynamicSMemSize, blockSizeLimit
                    );

            // Round up according to array size
            num_blocks_currents = (_N + num_threads_currents - 1) / num_threads_currents;
            // ensure our grid is executable
            struct hipFuncAttributes funcAttrib_currents;
            CUDA_SAFE_CALL(
                    hipFuncGetAttributes(&funcAttrib_currents, reinterpret_cast<const void*>(_currents_kernel_{{codeobj_name}}))
                    );
            assert(num_threads_currents <= funcAttrib_currents.maxThreadsPerBlock);

            // kernel properties
            int max_active_blocks_currents;
            CUDA_SAFE_CALL(
                    hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks_currents,
                        _currents_kernel_{{codeobj_name}}, num_threads_currents, 0)
                    );

            float occupancy_currents = (max_active_blocks_currents * num_threads_currents / num_threads_per_warp) /
                              (float)(max_threads_per_sm / num_threads_per_warp);

            printf("INFO _currents\n_kernel_{{codeobj_name}}"
                       "\t%u blocks\n"
                       "\t%u threads\n"
                       "\t%i registers per block\n"
                       "\t%i bytes statically-allocated shared memory per block\n"
                       "\t%i bytes local memory per thread\n"
                       "\t%i bytes user-allocated constant memory\n"
                       "\t%.3f theoretical occupancy\n",
                       num_blocks_currents, num_threads_currents, funcAttrib_currents.numRegs,
                       funcAttrib_currents.sharedSizeBytes, funcAttrib_currents.localSizeBytes,
                       funcAttrib_currents.constSizeBytes, occupancy_currents);


            first_run_custom = false; // now we have set up the grid

        }

{#
        {% if profiled %}
        std::clock_t _start_time_currents = std::clock();
        {% endif %}
#}
        // run kernel 5
        _currents_kernel_{{codeobj_name}}<<<num_blocks_currents, num_threads_currents>>>(
                _N,
                ///// HOST_PARAMETERS /////
                %HOST_PARAMETERS%
            );
        CUDA_CHECK_ERROR("_currents_kernel_{{codeobj_name}}");

{#
    {% if profiled %}
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    {{codeobj_name}}_kernel_currents_profiling_info += (double)(std::clock() -_start_time_currents)/CLOCKS_PER_SEC;
    {% endif %}
#}


{% endblock extra_kernel_call_post %}
