#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "objects.h"
#include <ctime>
#include <time.h>
#include "run.h"
#include "brianlib/common_math.h"
#include "brianlib/cuda_utils.h"
#include "rand.h"

{% for codeobj in code_objects %}
#include "code_objects/{{codeobj.name}}.h"
{% for block in codeobj.before_after_blocks %}
#include "code_objects/{{block}}_{{codeobj.name}}.h"
{% endfor %}
{% endfor %}

{% for name in user_headers %}
#include {{name}}
{% endfor %}

#include <iostream>
#include <fstream>
#include <string>
#include "hip/hip_runtime_api.h"

{{report_func|autoindent}}

int main(int argc, char **argv)
{
    {% if helpful %}
    LOG_INFO("%s", "Initializing standalone simulation...\n");
    {% else %}
    LOG_DEBUG("%s", "Initializing standalone simulation...\n");
    {{'\n'.join(code_lines['before_start'])|autoindent}}
    {% endif %}

    // seed variable set in Python through brian2.seed() calls can use this
    // variable (see device.py CUDAStandaloneDevice.generate_main_source())
    unsigned long long seed;

    CUDA_SAFE_CALL(
            hipSetDevice({{gpu_id}})
            );

    hipDeviceProp_t props;
    CUDA_SAFE_CALL(
            hipGetDeviceProperties(&props, {{gpu_id}})
            );
    size_t limit = {{gpu_heap_size}} * 1024 * 1024;
    CUDA_SAFE_CALL(
            hipDeviceSetLimit(hipLimitMallocHeapSize, limit)
            );
    CUDA_SAFE_CALL(
            hipDeviceSynchronize()
            );

    brian_start();

    {{'\n'.join(code_lines['after_start'])|autoindent}}

    //const std::clock_t _start_time3 = std::clock();
    {
        using namespace brian;

        {{main_lines|autoindent}}
    }

    {{'\n'.join(code_lines['before_end'])|autoindent}}
    brian_end();
    {{'\n'.join(code_lines['after_end'])|autoindent}}

    return 0;
}
