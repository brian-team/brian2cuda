#include "hip/hip_runtime.h"
{# TODO: get rid of the variables we don't actually use #}
{# USES_VARIABLES { _synaptic_pre, _synaptic_post, rand,
                    N_incoming, N_outgoing, N,
                    N_pre, N_post, _source_offset, _target_offset } #}

{# WRITES_TO_READ_ONLY_VARIABLES { _synaptic_pre, _synaptic_post,
                                   N_incoming, N_outgoing, N} #}
{% extends 'common_synapses.cu' %}

{% block extra_headers %}
{{ super() }}
#include<iostream>
#include<hiprand.h>
#include<brianlib/curand_buffer.h>
#include "brianlib/cuda_utils.h"
#include<map>
{% endblock extra_headers %}

{% block random_functions %}
// NOTE: _ptr_array_%CODEOBJ_NAME%_rand is NOT an array
// but an instance of CurandBuffer, which overloads the operator[], which then just
// returns the next random number in the buffer, ignoring the argument passed to operator[]
// NOTE: Put buffers into anonymous namespace such that _host_rand/n and rand/n
// in main code have access to it.
// NOTE: _host_rand/n is used in the host compiled implementation of binomial
// functions. Here, it just returns the next element from the CurandBuffer.
CurandBuffer<randomNumber_t> _ptr_array_%CODEOBJ_NAME%_rand(&brian::curand_generator, RAND);
randomNumber_t _host_rand(const int _vectorisation_idx)
{
    return _ptr_array_%CODEOBJ_NAME%_rand[_vectorisation_idx];
}

CurandBuffer<randomNumber_t> _ptr_array_%CODEOBJ_NAME%_randn(&brian::curand_generator, RANDN);
randomNumber_t _host_randn(const int _vectorisation_idx)
{
    return _ptr_array_%CODEOBJ_NAME%_randn[_vectorisation_idx];
}

// This is the C++ Standalone implementation of the poisson function, which we use
double _loggam(double x) {
  double x0, x2, xp, gl, gl0;
  int32_t k, n;

  static double a[10] = {8.333333333333333e-02, -2.777777777777778e-03,
                         7.936507936507937e-04, -5.952380952380952e-04,
                         8.417508417508418e-04, -1.917526917526918e-03,
                         6.410256410256410e-03, -2.955065359477124e-02,
                         1.796443723688307e-01, -1.39243221690590e+00};
  x0 = x;
  n = 0;
  if ((x == 1.0) || (x == 2.0))
    return 0.0;
  else if (x <= 7.0) {
    n = (int32_t)(7 - x);
    x0 = x + n;
  }
  x2 = 1.0 / (x0 * x0);
  xp = 2 * M_PI;
  gl0 = a[9];
  for (k=8; k>=0; k--) {
    gl0 *= x2;
    gl0 += a[k];
  }
  gl = gl0 / x0 + 0.5 * log(xp) + (x0 - 0.5) * log(x0) - x0;
  if (x <= 7.0) {
    for (k=1; k<=n; k++) {
      gl -= log(x0 - 1.0);
      x0 -= 1.0;
    }
  }
  return gl;
}

int32_t _poisson_mult(double lam, int _vectorisation_idx) {
  int32_t X;
  double prod, U, enlam;

  enlam = exp(-lam);
  X = 0;
  prod = 1.0;
  while (1) {
    U = _rand(_vectorisation_idx);
    prod *= U;
    if (prod > enlam)
      X += 1;
    else
      return X;
  }
}

int32_t _poisson_ptrs(double lam, int _vectorisation_idx) {
  int32_t k;
  double U, V, slam, loglam, a, b, invalpha, vr, us;

  slam = sqrt(lam);
  loglam = log(lam);
  b = 0.931 + 2.53 * slam;
  a = -0.059 + 0.02483 * b;
  invalpha = 1.1239 + 1.1328 / (b - 3.4);
  vr = 0.9277 - 3.6224 / (b - 2);

  while (1) {
    U = _rand(_vectorisation_idx) - 0.5;
    V = _rand(_vectorisation_idx);
    us = 0.5 - abs(U);
    k = (int32_t)floor((2 * a / us + b) * U + lam + 0.43);
    if ((us >= 0.07) && (V <= vr))
      return k;
    if ((k < 0) || ((us < 0.013) && (V > us)))
      continue;
    if ((log(V) + log(invalpha) - log(a / (us * us) + b)) <=
        (-lam + k * loglam - _loggam(k + 1)))
      return k;
  }
}

int32_t _host_poisson(double lam, int32_t _idx) {
  if (lam >= 10)
    return _poisson_ptrs(lam, _idx);
  else if (lam == 0)
    return 0;
  else
    return _poisson_mult(lam, _idx);
}
{% endblock random_functions %}


{% block kernel %}
{% endblock %}

{% block kernel_call %}
{% endblock %}

{% block prepare_kernel %}
{% endblock %}

{% block occupancy %}
{% endblock %}

{% block update_occupancy %}
{% endblock %}

{% block kernel_info %}
{% endblock %}

{% block profiling_start %}
std::clock_t start_timer = std::clock();

CUDA_CHECK_MEMORY();
size_t used_device_memory_start = used_device_memory;
{% endblock %}

{% block profiling_stop %}
CUDA_CHECK_MEMORY();
const double to_MB = 1.0 / (1024.0 * 1024.0);
double tot_memory_MB = (used_device_memory - used_device_memory_start) * to_MB;
double time_passed = (double)(std::clock() - start_timer) / CLOCKS_PER_SEC;
std::cout << "INFO: {{owner.name}} creation took " <<  time_passed << "s";
if (tot_memory_MB > 0)
    std::cout << " and used " << tot_memory_MB << "MB of memory.";
std::cout << std::endl;
{% endblock %}

{% block host_maincode %}

    ///// pointers_lines /////
    {{pointers_lines|autoindent}}

    {# Get N_post and N_pre in the correct way, regardless of whether they are
    constants or scalar arrays#}
    const size_t _N_pre = {{constant_or_scalar('N_pre', variables['N_pre'])}};
    const size_t _N_post = {{constant_or_scalar('N_post', variables['N_post'])}};
    {{_dynamic_N_incoming}}.resize(_N_post + _target_offset);
    {{_dynamic_N_outgoing}}.resize(_N_pre + _source_offset);

    size_t _raw_pre_idx, _raw_post_idx;
    {# For a connect call j='k+i for k in range(0, N_post, 2) if k+i < N_post'
    "j" is called the "result index" (and "_post_idx" the "result index array", etc.)
    "i" is called the "outer index" (and "_pre_idx" the "outer index array", etc.)
    "k" is called the inner variable #}

    const size_t _vectorisation_idx = -1;
    ///// scalar_code['setup_iterator'] /////
    {{scalar_code['setup_iterator']|autoindent}}
    ///// scalar_code['generator_expr'] /////
    {{scalar_code['generator_expr']|autoindent}}
    ///// scalar_code['create_cond'] /////
    {{scalar_code['create_cond']|autoindent}}
    ///// scalar_code['update'] /////
    {{scalar_code['update']|autoindent}}

    for(size_t _{{outer_index}}=0; _{{outer_index}}<_{{outer_index_size}}; _{{outer_index}}++)
    {

        bool __cond, _cond;
        _raw{{outer_index_array}} = _{{outer_index}} + {{outer_index_offset}};
        {% if not result_index_condition %}
        {
            ///// vector_code['create_cond'] /////
            {{vector_code['create_cond']|autoindent}}
            __cond = _cond;
        }
        _cond = __cond;
        if(!_cond) continue;
        {% endif %}
        // Some explanation of this hackery. The problem is that we have multiple code blocks.
        // Each code block is generated independently of the others, and they declare variables
        // at the beginning if necessary (including declaring them as const if their values don't
        // change). However, if two code blocks follow each other in the same C++ scope then
        // that causes a redeclaration error. So we solve it by putting each block inside a
        // pair of braces to create a new scope specific to each code block. However, that brings
        // up another problem: we need the values from these code blocks. I don't have a general
        // solution to this problem, but in the case of this particular template, we know which
        // values we need from them so we simply create outer scoped variables to copy the value
        // into. Later on we have a slightly more complicated problem because the original name
        // _j has to be used, so we create two variables __j, _j at the outer scope, copy
        // _j to __j in the inner scope (using the inner scope version of _j), and then
        // __j to _j in the outer scope (to the outer scope version of _j). This outer scope
        // version of _j will then be used in subsequent blocks.
        long _uiter_low;
        long _uiter_high;
        long _uiter_step;
        {% if iterator_func=='sample' %}
        long _uiter_size;
        double _uiter_p;
        {% endif %}
        {
            ///// vector_code['setup_iterator'] /////
            {{vector_code['setup_iterator']|autoindent}}
            _uiter_low = _iter_low;
            _uiter_high = _iter_high;
            _uiter_step = _iter_step;
            {% if iterator_func=='sample' %}
            {% if iterator_kwds['sample_size'] == 'fixed' %}
            _uiter_size = _iter_size;
            {% else %}
            _uiter_p = _iter_p;
            {% endif %}
            {% endif %}
        }
        {% if iterator_func=='range' %}
        for(long {{inner_variable}}=_uiter_low; {{inner_variable}}<_uiter_high; {{inner_variable}}+=_uiter_step)
        {
        {% elif iterator_func=='sample' %}
        const int _iter_sign = _uiter_step > 0 ? 1 : -1;
        {% if iterator_kwds['sample_size'] == 'fixed' %}
        std::set<int> _selected_set = std::set<int>();
        std::set<int>::iterator _selected_it;
        int _n_selected = 0;
        int _n_dealt_with = 0;
        int _n_total;
        if (_uiter_step > 0)
            _n_total = (_uiter_high - _uiter_low - 1) / _uiter_step + 1;
        else
            _n_total = (_uiter_low - _uiter_high - 1) / -_uiter_step + 1;
        // Value determined by benchmarking, see github PR #1280
        const bool _selection_algo = 1.0*_uiter_size / _n_total > 0.06;
        if (_uiter_size > _n_total)
        {
            {% if skip_if_invalid %}
            _uiter_size = _n_total;
            {% else %}
            cout << "Error: Requested sample size " << _uiter_size << " is bigger than the " <<
                    "population size " << _n_total << "." << endl;
            exit(1);
            {% endif %}
        } else if (_uiter_size < 0)
        {
            {% if skip_if_invalid %}
            continue;
            {% else %}
            cout << "Error: Requested sample size " << _uiter_size << " is negative." << endl;
            exit(1);
            {% endif %}
        } else if (_uiter_size == 0)
            continue;
        long {{inner_variable}};

        if (_selection_algo)
        {
            {{inner_variable}} = _uiter_low - _uiter_step;
        } else
        {
            // For the tracking algorithm, we have to first create all values
            // to make sure they will be iterated in sorted order
            _selected_set.clear();
            while (_n_selected < _uiter_size)
            {
                int _r = (int)(_rand(_vectorisation_idx) * _n_total);
                while (! _selected_set.insert(_r).second)
                    _r = (int)(_rand(_vectorisation_idx) * _n_total);
                _n_selected++;
            }
            _n_selected = 0;
            _selected_it = _selected_set.begin();
        }
        while (_n_selected < _uiter_size)
        {
            if (_selection_algo)
            {
                // Selection sampling technique
                // See section 3.4.2 of Donald E. Knuth, AOCP, Vol 2, Seminumerical Algorithms
                {{inner_variable}} += _uiter_step;
                _n_dealt_with++;
                const double _U = _rand(_vectorisation_idx);
                if ((_n_total - _n_dealt_with) * _U >= _uiter_size - _n_selected)
                    continue;
            } else
            {
                {{inner_variable}} = _uiter_low + (*_selected_it)*_uiter_step;
                _selected_it++;
            }
            _n_selected++;
        {% else %}
        if(_uiter_p==0) continue;
        const bool _jump_algo = _uiter_p<0.25;
        double _log1p;
        if(_jump_algo)
            _log1p = log(1-_uiter_p);
        else
            _log1p = 1.0; // will be ignored
        const double _pconst = 1.0/log(1-_uiter_p);
        for(long {{inner_variable}}=_uiter_low; _iter_sign*{{inner_variable}}<_iter_sign*_uiter_high; {{inner_variable}} += _uiter_step)
        {
            if(_jump_algo) {
                const double _r = _rand(_vectorisation_idx);
                if(_r==0.0) break;
                const int _jump = floor(log(_r)*_pconst)*_uiter_step;
                {{inner_variable}} += _jump;
                if (_iter_sign*{{inner_variable}} >= _iter_sign * _uiter_high) continue;
            } else {
                if (_rand(_vectorisation_idx)>=_uiter_p) continue;
            }
        {% endif %}
        {% endif %}
            long __{{result_index}}, _{{result_index}}, {{outer_index_array}}, _{{outer_index_array}};
            {
                ///// vector_code['generator_expr'] /////
                {{vector_code['generator_expr']|autoindent}}
                __{{result_index}} = _{{result_index}}; // pick up the locally scoped var and store in outer var
                _{{outer_index_array}} = {{outer_index_array}};
            }
            _{{result_index}} = __{{result_index}}; // make the previously locally scoped var available
            {{outer_index_array}} = _{{outer_index_array}};
            _raw{{result_index_array}} = _{{result_index}} + {{result_index_offset}};
            {% if result_index_condition %}
            {
                {% if result_index_used %}
                {# The condition could index outside of array range #}
                if(_{{result_index}}<0 || _{{result_index}}>=_{{result_index_size}})
                {
                    {% if skip_if_invalid %}
                    continue;
                    {% else %}
                    cout << "Error: tried to create synapse to neuron {{result_index}}=" << _{{result_index}} << " outside range 0 to " <<
                                            _{{result_index_size}}-1 << endl;
                    exit(1);
                    {% endif %}
                }
                {% endif %}
                ///// vector_code['create_cond'] /////
                {{vector_code['create_cond']|autoindent}}
                __cond = _cond;
            }
            _cond = __cond;
            {% endif %}

            {% if if_expression!='True' %}
            if(!_cond) continue;
            {% endif %}

            {% if not result_index_used %}
            {# Otherwise, we already checked before #}
            if(_{{result_index}}<0 || _{{result_index}}>=_{{result_index_size}})
            {
                {% if skip_if_invalid %}
                continue;
                {% else %}
                cout << "Error: tried to create synapse to neuron {{result_index}}=" << _{{result_index}} <<
                        " outside range 0 to " << _{{result_index_size}}-1 << endl;
                exit(1);
                {% endif %}
            }
            {% endif %}

            ///// vector_code['update'] /////
            {{vector_code['update']|autoindent}}

            for (size_t _repetition=0; _repetition<_n; _repetition++) {
                {{_dynamic_N_outgoing}}[_pre_idx] += 1;
                {{_dynamic_N_incoming}}[_post_idx] += 1;
                {{_dynamic__synaptic_pre}}.push_back(_pre_idx);
                {{_dynamic__synaptic_post}}.push_back(_post_idx);
            }
        }
    }

    // now we need to resize all registered variables
    const int32_t newsize = {{_dynamic__synaptic_pre}}.size();
    {% for variable in owner._registered_variables | sort(attribute='name') %}
        {% set varname = get_array_name(variable, access_data=False) %}
        {% if variable.name == 'delay' and no_or_const_delay_mode %}
            assert(dev{{varname}}.size() <= 1);
            THRUST_CHECK_ERROR(
                    dev{{varname}}.resize(1)
                    );
            {# //TODO: do we actually need to resize varname? #}
            {{varname}}.resize(1);
        {% elif variable.name == '_synaptic_pre' and no_pre_references %}
        // prefs['devices.cuda_standalone.no_pre_references'] was set,
        // skipping synaptic_pre resize
        {% elif variable.name == '_synaptic_post' and no_post_references %}
        // prefs['devices.cuda_standalone.no_post_references'] was set,
        // skipping synaptic_post resize
        {% else %}
            {% if not multisynaptic_index or not variable == multisynaptic_idx_var %}
            THRUST_CHECK_ERROR(
                    dev{{varname}}.resize(newsize)
                    );
            {% endif %}
            {# //TODO: do we actually need to resize varname? #}
            {{varname}}.resize(newsize);
        {% endif %}
    {% endfor %}
    // Also update the total number of synapses
    {{N}} = newsize;

    // Check for occurrence of multiple source-target pairs in synapses ("synapse number")
    std::map<std::pair<int32_t, int32_t>, int32_t> source_target_count;
    for (int _i=0; _i<newsize; _i++)
    {
        // Note that source_target_count will create a new entry initialized
        // with 0 when the key does not exist yet
        const std::pair<int32_t, int32_t> source_target = std::pair<int32_t, int32_t>({{_dynamic__synaptic_pre}}[_i], {{_dynamic__synaptic_post}}[_i]);
        {% if multisynaptic_index %}
        // Save the "synapse number"
        {% set dynamic_multisynaptic_idx = get_array_name(multisynaptic_idx_var, access_data=False) %}
        {{dynamic_multisynaptic_idx}}[_i] = source_target_count[source_target];
        {% endif %}
        source_target_count[source_target]++;
        //printf("source target count = %i\n", source_target_count[source_target]);
        if (source_target_count[source_target] > 1)
        {
            {{owner.name}}_multiple_pre_post = true;
            {% if not multisynaptic_index %}
            break;
            {% endif %}
        }
    }

    // copy changed host data to device
    dev{{_dynamic_N_incoming}} = {{_dynamic_N_incoming}};
    dev{{_dynamic_N_outgoing}} = {{_dynamic_N_outgoing}};
    dev{{_dynamic__synaptic_pre}} = {{_dynamic__synaptic_pre}};
    dev{{_dynamic__synaptic_post}} = {{_dynamic__synaptic_post}};
    {% if multisynaptic_index %}
    dev{{dynamic_multisynaptic_idx}} = {{dynamic_multisynaptic_idx}};
    {% endif %}
    CUDA_SAFE_CALL(
            hipMemcpy(dev{{get_array_name(variables['N'], access_data=False)}},
                {{get_array_name(variables['N'], access_data=False)}},
                sizeof({{c_data_type(variables['N'].dtype)}}),
                hipMemcpyHostToDevice)
            );
{% endblock host_maincode %}

{% block extra_kernel_call_post %}
// free memory in CurandBuffers
_ptr_array_%CODEOBJ_NAME%_rand.free_memory();
_ptr_array_%CODEOBJ_NAME%_randn.free_memory();
{% endblock extra_kernel_call_post %}
